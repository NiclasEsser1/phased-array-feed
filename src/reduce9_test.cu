#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <inttypes.h>
#include "cudautil.cuh"
#include "kernel.cuh"

#include "constants.h"

#define NBYTE_RT   8

extern "C" void usage ()
{
  fprintf (stdout,
	   "reduce9_test - Test the reduce9 kernel \n"
	   "\n"
	   "Usage: reduce9_test [options]\n"
	   " -a  Grid size in X\n"
	   " -b  Grid size in Y\n"
	   " -c  Block size in X\n"
	   " -d  Number of samples to accumulate in each block\n"
	   " -e  Number of streams in use\n"
	   " -h  show help\n");
}

// ./reduce9_test -a 512 -b 1 -c 512 -d 1024 -e 2 -f 1.0
int main(int argc, char *argv[])
{
  int i, j, k;
  int arg, nstream;
  int grid_x, grid_y, block_x;
  uint64_t n_accumulate;
  uint64_t len_in, len_out, idx;
  dim3 gridsize_reduce9, blocksize_reduce9;
  float h_total = 0, g_total = 0;
  hipfftComplex *data = NULL, *g_in = NULL, *g_out = NULL, *g_result = NULL, *h_result = NULL;
  float scl_ndim;
  
  /* Read in parameters, the arguments here have the same name  */
  while((arg=getopt(argc,argv,"a:b:hc:d:e:f:")) != -1)
    {
      switch(arg)
	{
	case 'h':
	  usage();
	  exit(EXIT_FAILURE);	  

	case 'a':	  
	  if (sscanf (optarg, "%d", &grid_x) != 1)
	    {
	      fprintf (stderr, "Does not get grid_x, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	      exit(EXIT_FAILURE);
	    }
	  break;
	  
	case 'b':	  
	  if (sscanf (optarg, "%d", &grid_y) != 1)
	    {
	      fprintf (stderr, "Does not get grid_y, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	      exit(EXIT_FAILURE);
	    }
	  break;
	  
	case 'c':	  
	  if (sscanf (optarg, "%d", &block_x) != 1)
	    {
	      fprintf (stderr, "Does not get block_x, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	      exit(EXIT_FAILURE);
	    }
	  break;
	  
	case 'd':	  
	  if (sscanf (optarg, "%"SCNu64"", &n_accumulate) != 1)
	    {
	      fprintf (stderr, "Does not get n_accumulate, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	      exit(EXIT_FAILURE);
	    }
	  fprintf(stdout, "n_accumulate is %"PRIu64"\n",  n_accumulate);
	  break;
	  
	case 'e':	  
	  if (sscanf (optarg, "%d", &nstream) != 1)
	    {
	      fprintf (stderr, "Does not get nstream, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	      exit(EXIT_FAILURE);
	    }
	  fprintf(stdout, "nstream is %d\n",  nstream);
	  break;
	  
	case 'f':	  
	  if (sscanf (optarg, "%f", &scl_ndim) != 1)
	    {
	      fprintf (stderr, "Does not get scl_ndim, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	      exit(EXIT_FAILURE);
	    }
	  fprintf(stdout, "scl_ndim is %f\n",  scl_ndim);
	  break;
	}
    }

  fprintf(stdout, "grid_x is %d, grid_y is %d, block_x is %d, n_accumulate is %"SCNu64", nstream is %d and scl_ndim is %f\n", grid_x, grid_y, block_x, n_accumulate, nstream, scl_ndim);
  
  /* Setup size */
  gridsize_reduce9.x  = grid_x;
  gridsize_reduce9.y  = grid_y;
  gridsize_reduce9.z  = 1;
  blocksize_reduce9.x = block_x;
  blocksize_reduce9.y = 1;
  blocksize_reduce9.z = 1;
  len_out             = grid_x*grid_y;
  len_in              = len_out*n_accumulate;

  /* Create buffer */
  CudaSafeCall(hipHostMalloc((void **)&data, nstream * len_in * sizeof(hipfftComplex)));
  CudaSafeCall(hipMalloc((void **)&g_in,     nstream * len_in * sizeof(hipfftComplex)));
  
  CudaSafeCall(hipHostMalloc((void **)&h_result, len_out * sizeof(hipfftComplex)));
  CudaSafeCall(hipHostMalloc((void **)&g_result, len_out * sizeof(hipfftComplex)));
  CudaSafeCall(hipMalloc((void **)&g_out,        len_out * sizeof(hipfftComplex)));
  
  CudaSafeCall(hipMemset((void *)g_out,  0,      len_out * sizeof(hipfftComplex)));
  CudaSafeCall(hipMemset((void *)h_result, 0,    len_out * sizeof(hipfftComplex)));
  
  /* cauculate on CPU */
  srand(time(NULL));
  for(i = 0; i < len_out; i ++)
    {
      h_result[i].x = 0;
      h_result[i].y= 0;
      for(j = 0; j < n_accumulate; j++)
	{
	  idx = i * n_accumulate + j;
	  for(k = 0; k < nstream; k++)
	    {
	      data[idx+k*len_in].x = (float)rand()/(float)(RAND_MAX/(float)MAX_RAND);
	      data[idx+k*len_in].y = (float)rand()/(float)(RAND_MAX/(float)MAX_RAND);
	  
	      h_result[i].x += (data[idx+k*len_in].x/scl_ndim);
	      h_result[i].y += (data[idx+k*len_in].y/scl_ndim);
	    }
	}
    }
  
  /* Calculate on GPU */
  CudaSafeCall(hipMemcpy(g_in, data, nstream * len_in * sizeof(hipfftComplex), hipMemcpyHostToDevice));
  switch (blocksize_reduce9.x)
    {
    case 1024:
      reduce9_kernel<1024><<<gridsize_reduce9, blocksize_reduce9, blocksize_reduce9.x * NBYTE_RT>>>(g_in, g_out, len_in, n_accumulate, nstream, scl_ndim);
      break;
      
    case 512:
      reduce9_kernel< 512><<<gridsize_reduce9, blocksize_reduce9, blocksize_reduce9.x * NBYTE_RT>>>(g_in, g_out, len_in, n_accumulate, nstream, scl_ndim);
      break;
      
    case 256:
      reduce9_kernel< 256><<<gridsize_reduce9, blocksize_reduce9, blocksize_reduce9.x * NBYTE_RT>>>(g_in, g_out, len_in, n_accumulate, nstream, scl_ndim);
      break;
      
    case 128:
      reduce9_kernel< 128><<<gridsize_reduce9, blocksize_reduce9, blocksize_reduce9.x * NBYTE_RT>>>(g_in, g_out, len_in, n_accumulate, nstream, scl_ndim);
      break;
      
    case 64:
      reduce9_kernel<  64><<<gridsize_reduce9, blocksize_reduce9, blocksize_reduce9.x * NBYTE_RT>>>(g_in, g_out, len_in, n_accumulate, nstream, scl_ndim);
      break;
      
    case 32:
      reduce9_kernel<  32><<<gridsize_reduce9, blocksize_reduce9, blocksize_reduce9.x * NBYTE_RT>>>(g_in, g_out, len_in, n_accumulate, nstream, scl_ndim);
      break;
      
    case 16:
      reduce9_kernel<  16><<<gridsize_reduce9, blocksize_reduce9, blocksize_reduce9.x * NBYTE_RT>>>(g_in, g_out, len_in, n_accumulate, nstream, scl_ndim);
      break;
      
    case 8:
      reduce9_kernel<   8><<<gridsize_reduce9, blocksize_reduce9, blocksize_reduce9.x * NBYTE_RT>>>(g_in, g_out, len_in, n_accumulate, nstream, scl_ndim);
      break;
      
    case 4:
      reduce9_kernel<   4><<<gridsize_reduce9, blocksize_reduce9, blocksize_reduce9.x * NBYTE_RT>>>(g_in, g_out, len_in, n_accumulate, nstream, scl_ndim);
      break;
      
    case 2:
      reduce9_kernel<   2><<<gridsize_reduce9, blocksize_reduce9, blocksize_reduce9.x * NBYTE_RT>>>(g_in, g_out, len_in, n_accumulate, nstream, scl_ndim);
      break;
      
    case 1:
      reduce9_kernel<   1><<<gridsize_reduce9, blocksize_reduce9, blocksize_reduce9.x * NBYTE_RT>>>(g_in, g_out, len_in, n_accumulate, nstream, scl_ndim);
      break;
    }
  CHECK_LAUNCH_ERROR();
  CudaSafeCall(hipMemcpy(g_result, g_out, len_out * sizeof(hipfftComplex), hipMemcpyDeviceToHost));

  /* Check the result */
  for(i = 0; i < len_out; i++)
    {
      h_total += (h_result[i].x + h_result[i].y);
      g_total += (g_result[i].x + g_result[i].y);
    }
  //fprintf(stdout, "%f\t%f\t%E\n", h_total, g_total, (g_total - h_total)/h_total);
  fprintf(stdout, "CPU:\t%f\nGPU:\t%f\n%E\n", h_total, g_total, (g_total - h_total)/h_total);
  
  /* Free buffer */
  CudaSafeCall(hipFree(g_in));
  CudaSafeCall(hipFree(g_out));
  CudaSafeCall(hipHostFree(data));
  CudaSafeCall(hipHostFree(h_result));
  CudaSafeCall(hipHostFree(g_result));
  
  return EXIT_SUCCESS;
}