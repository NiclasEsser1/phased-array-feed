#include "hip/hip_runtime.h"
#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <inttypes.h>
#include "cudautil.cuh"
#include "kernel.cuh"

#include "constants.h"

extern "C" void usage ()
{
  fprintf (stdout,
	   "scale2_test - Test the scale2 kernel \n"
	   "\n"
	   "Usage: scale2_test [options]\n"
	   " -a  Number of channels\n"
	   " -h  show help\n");
}

// ./scale2_test -a 512
int main(int argc, char *argv[])
{
  int i, arg, nchan;
  float *h_result = NULL;
  float temp;
  dim3 gridsize_scale2, blocksize_scale2;
  hipfftComplex *g = NULL, *data = NULL, *g_result = NULL;
  
  /* Read in parameters, the arguments here have the same name  */
  while((arg=getopt(argc,argv,"ha:")) != -1)
    {
      switch(arg)
	{
	case 'h':
	  usage();
	  exit(EXIT_FAILURE);	  

	case 'a':	  
	  if (sscanf (optarg, "%d", &nchan) != 1)
	    {
	      fprintf (stderr, "Does not get nchan, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	      exit(EXIT_FAILURE);
	    }
	  break;
	}
    }
  fprintf(stdout, "nchan is %d\n", nchan);
  
  /* Setup size */
  gridsize_scale2.x = 1;
  gridsize_scale2.y = 1;
  gridsize_scale2.z = 1;
  blocksize_scale2.x = nchan;
  blocksize_scale2.y = 1;
  blocksize_scale2.z = 1;
  fprintf(stdout, "configuration for kernel is (%d, %d, %d) and (%d, %d, %d)", gridsize_scale2.x, gridsize_scale2.y, gridsize_scale2.z, blocksize_scale2.x, blocksize_scale2.y, blocksize_scale2.z);

  /* Create buffer */
  CudaSafeCall(hipHostMalloc((void **)&data, nchan * NBYTE_CUFFT_COMPLEX));
  CudaSafeCall(hipHostMalloc((void **)&h_result,  nchan * NBYTE_FLOAT));
  CudaSafeCall(hipHostMalloc((void **)&g_result, nchan * NBYTE_CUFFT_COMPLEX));
  CudaSafeCall(hipMalloc((void **)&g, nchan * NBYTE_CUFFT_COMPLEX));
  /* prepare data and calculate on CPU*/
  srand(time(NULL));
  for(i = 0; i < nchan; i++)
    {
      data[i].x = rand()*RAND_STD/RAND_MAX;
      temp = rand()*RAND_STD/RAND_MAX;
      data[i].y =  temp * temp;
      while (data[i].y < data[i].x * data[i].x)
	{
	  temp = rand()*RAND_STD/RAND_MAX;
	  data[i].y =  temp * temp;
	}
      h_result[i]  = SCL_NSIG * sqrt(data[i].y - data[i].x * data[i].x)/SCL_UINT8;
    }

  /* Calculate on GPU */
  CudaSafeCall(hipMemcpy(g, data, nchan * NBYTE_CUFFT_COMPLEX, hipMemcpyHostToDevice));
  scale2_kernel<<<gridsize_scale2, blocksize_scale2>>>(g, SCL_NSIG, SCL_UINT8);
  CudaSafeKernelLaunch();
  
  CudaSafeCall(hipMemcpy(g_result, g, nchan * NBYTE_CUFFT_COMPLEX, hipMemcpyDeviceToHost));

  /* Check the result */
  for(i = 0; i < nchan; i++)
    fprintf(stdout, "%E\t%f\t%f\n", g_result[i].y - h_result[i], g_result[i].y, h_result[i]);
  
  /* Free memory */
  CudaSafeCall(hipHostFree(data));
  CudaSafeCall(hipHostFree(h_result));
  CudaSafeCall(hipHostFree(g_result));
  
  CudaSafeCall(hipFree(g));
  
  return EXIT_SUCCESS;
}