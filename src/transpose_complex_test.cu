#include "hip/hip_runtime.h"
#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <inttypes.h>
#include <byteswap.h>

#include "cudautil.cuh"
#include "kernel.cuh"
#include "constants.h"

// ./transpose_complex_test -a 48 -b 1024 -c 64
// ./transpose_complex_test -a 33 -b 1024 -c 64

extern "C" void usage ()
{
  fprintf (stdout,
	   "transpose_complex_test - Test the transpose_complex kernel \n"
	   "\n"
	   "Usage: transpose_complex_test [options]\n"
	   " -a  Number of input frequency chunks\n"
	   " -b  Number of packets of each stream per frequency chunk\n"
	   " -c  Number of FFT points\n"
	   " -h  show help\n");
}

int main(int argc, char *argv[])
{
  int arg;
  int i, j, k;
  int nchk_in, nchan_in;
  int stream_ndf_chk, cufft_nx, cufft_mod;
  int nchan_keep_chan;
  dim3 grid_size, block_size;
  uint64_t nsamp_in, nsamp_out, npol_in, npol_out, idx_in, idx_out;
  hipfftComplex *data = NULL, *h_result = NULL, *g_result = NULL, *g_in = NULL, *g_out = NULL;
  
  /* Read in parameters */
  while((arg=getopt(argc,argv,"a:b:hc:")) != -1)
    {
      switch(arg)
	{
	case 'h':
	  usage();
	  exit(EXIT_FAILURE);
	  
	case 'a':	  
	  if (sscanf (optarg, "%d", &nchk_in) != 1)
	    {
	      fprintf (stderr, "Could not get nchk_in, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	      exit(EXIT_FAILURE);
	    }
	  break;
	  
	case 'b':	  
	  if (sscanf (optarg, "%d", &stream_ndf_chk) != 1)
	    {
	      fprintf (stderr, "Could not get stream_ndf_chk, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	      exit(EXIT_FAILURE);
	    }
	  break;
	  
	case 'c':	  
	  if (sscanf (optarg, "%d", &cufft_nx) != 1)
	    {
	      fprintf (stderr, "Could not get cufft_nx, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	      exit(EXIT_FAILURE);
	    }
	  break;
	  
	}
    }
  fprintf(stdout, "nchk_in is %d, stream_ndf_chk is %d and cufft_nx is %d\n", nchk_in, stream_ndf_chk, cufft_nx);

  /* Setup size */
  nchan_in        = nchk_in * NCHAN_PER_CHUNK;
  nchan_keep_chan = cufft_nx / OVER_SAMP_RATE;
  cufft_mod       = 0.5 * nchan_keep_chan;
  fprintf(stdout, "nchan_in is %d, nchan_keep_chan is %d and cufft_mod is %d\n", nchan_in, nchan_keep_chan, cufft_mod);
    
  grid_size.x = stream_ndf_chk * NSAMP_DF / cufft_nx;
  grid_size.y = nchan_in;
  grid_size.z = 1;  
  block_size.x = nchan_keep_chan;
  block_size.y = 1;
  block_size.z = 1;
  fprintf(stdout, "kernel configuration is (%d, %d, %d) and (%d, %d, %d)\n", grid_size.x, grid_size.y, grid_size.z, block_size.x, block_size.y, block_size.z);

  nsamp_in  = stream_ndf_chk * nchan_in * NSAMP_DF / OVER_SAMP_RATE;
  nsamp_out = nsamp_in;
  npol_in   = nsamp_in * NPOL_BASEBAND;
  npol_out  = nsamp_out * NPOL_BASEBAND;
  
  fprintf(stdout, "%"PRIu64"\t%"PRIu64"\t%"PRIu64"\t%"PRIu64"\n", nsamp_in, nsamp_out, npol_in, npol_out);

  /* Create buffer */
  CudaSafeCall(hipHostMalloc((void **)&data,     npol_in * NBYTE_CUFFT_COMPLEX));
  CudaSafeCall(hipHostMalloc((void **)&h_result, npol_out * NBYTE_CUFFT_COMPLEX));
  CudaSafeCall(hipHostMalloc((void **)&g_result, npol_out * NBYTE_CUFFT_COMPLEX));
  CudaSafeCall(hipMalloc((void **)&g_in,         npol_in * NBYTE_CUFFT_COMPLEX));
  CudaSafeCall(hipMalloc((void **)&g_out,        npol_out * NBYTE_CUFFT_COMPLEX));

  /* Prepare the data */
  srand(time(NULL));
  for(i = 0; i < grid_size.x; i++)
    {
      for(j = 0; j < grid_size.y; j++)
	{
	  for(k = 0; k < block_size.x; k++)
	    {
	      idx_in = i * grid_size.y * block_size.x + j * block_size.x + k;
	      data[idx_in].x = rand() * RAND_STD/RAND_MAX;
	      data[idx_in].y = rand() * RAND_STD/RAND_MAX;
	      data[idx_in+nsamp_in].x = rand() * RAND_STD/RAND_MAX;
	      data[idx_in+nsamp_in].y = rand() * RAND_STD/RAND_MAX;

	      idx_out = j * grid_size.x * block_size.x + i * block_size.x + k;
	      
	      h_result[idx_out].x   = data[idx_in].x;
	      h_result[idx_out].y   = data[idx_in].y;
	      h_result[idx_out+nsamp_out].x = data[idx_in+nsamp_in].x;
	      h_result[idx_out+nsamp_out].y = data[idx_in+nsamp_in].y;
	    }
	}
    }

  /* Calculate on GPU */
  CudaSafeCall(hipMemcpy(g_in, data, npol_in * NBYTE_CUFFT_COMPLEX, hipMemcpyHostToDevice));
  transpose_complex_kernel<<<grid_size, block_size>>>(g_in, nsamp_in, g_out);
  CudaSafeKernelLaunch();
  CudaSafeCall(hipMemcpy(g_result, g_out, npol_out * NBYTE_CUFFT_COMPLEX, hipMemcpyDeviceToHost));

  /* Check the result */
  for(i = 0; i < nsamp_out; i++)
    {      
      if(fabs((h_result[i].x - g_result[i].x)/g_result[i].x) > 1.0E-3 || fabs((h_result[i].y - g_result[i].y)/g_result[i].y) > 1.0E-3)
	fprintf(stdout, "%f\t%f\t%f\t%f\t%f\t%f\n", h_result[i].x, g_result[i].x, h_result[i].x - g_result[i].x, h_result[i].y, g_result[i].y, h_result[i].y - g_result[i].y);
    }

  /* Free buffer */
  CudaSafeCall(hipHostFree(data));
  CudaSafeCall(hipHostFree(h_result));
  CudaSafeCall(hipHostFree(g_result));
  CudaSafeCall(hipFree(g_in));
  CudaSafeCall(hipFree(g_out));
  
  return EXIT_SUCCESS;
}