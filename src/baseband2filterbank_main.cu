#include "hip/hip_runtime.h"
#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <inttypes.h>

#include "baseband2filterbank.cuh"
#include "cudautil.cuh"
#include "log.h"

pthread_mutex_t log_mutex = PTHREAD_MUTEX_INITIALIZER;

void usage ()
{
  fprintf (stdout,
	   "baseband2filterbank_main - Convert BMF 16bits baseband data into 8bits filterbank data \n"
	   "\n"
	   "Usage: baseband2filterbank_main [options]\n"
	   " -a  Hexacdecimal shared memory key for incoming ring buffer\n"
	   " -b  Hexacdecimal shared memory key for outcoming ring buffer\n"
	   " -c  The number of data frame (per frequency chunk) of each incoming ring buffer block\n"
	   " -d  The number of streams \n"
	   " -e  The number of data frame (per frequency chunk) of each stream\n"
	   " -f  The directory to put runtime files\n"
	   " -g  Start of the data or not\n"
	   " -h  show help\n"
	   " -i  Number of chunks of input\n"
	   " -j  FFT length\n"
	   " -k  The number of output channels\n"
	   " -l  Number of channels keep for the band\n");
}

int main(int argc, char *argv[])
{
  int i, arg;
  conf_t conf;
  char log_fname[MSTR_LEN] = {'\0'};
  char command_line[MSTR_LEN] = {'\0'};
  
  /* Initial part */  
  while((arg=getopt(argc,argv,"a:b:c:d:e:f:hg:i:j:k:l:")) != -1)
    {
      switch(arg)
	{
	case 'h':
	  usage();
	  fprintf (stderr, "BASEBAND2FILTERBANK_ERROR:\tno input, which happens at \"%s\", line [%d].\n",  __FILE__, __LINE__);
	  exit(EXIT_FAILURE);
	  
	case 'a':	  
	  if (sscanf (optarg, "%x", &conf.key_in) != 1)
	    {
	      fprintf (stderr, "BASEBAND2FILTERBANK_ERROR:\tCould not parse key from %s, which happens at \"%s\", line [%d].\n", optarg, __FILE__, __LINE__);
	      exit(EXIT_FAILURE);
	    }
	  break;
	  
	case 'b':
	  if (sscanf (optarg, "%x", &conf.key_out) != 1)
	    {
	      fprintf (stderr, "BASEBAND2FILTERBANK_ERROR:\tCould not parse key from %s, which happens at \"%s\", line [%d].\n", optarg, __FILE__, __LINE__);
	      exit(EXIT_FAILURE);
	    }
	  break;
	  	  
	case 'c':
	  sscanf(optarg, "%"SCNu64"", &conf.rbufin_ndf_chk);
	  break;
	  
	case 'd':
	  sscanf(optarg, "%d", &conf.nstream);
	  break;
	  
	case 'e':
	  sscanf(optarg, "%d", &conf.stream_ndf_chk);
	  break;
	  	  
	case 'f':
	  sscanf(optarg, "%s", conf.dir);
	  break;
	  
	case 'g':
	  sscanf(optarg, "%d", &conf.sod);
	  break;
	  
	case 'i':
	  sscanf(optarg, "%d", &conf.nchk_in);
	  break;

	case 'j':
	  sscanf(optarg, "%d", &conf.cufft_nx);
	  break;

	case 'k':
	  sscanf(optarg, "%d", &conf.nchan_out);
	  break;
	  
	case 'l':
	  sscanf(optarg, "%d", &conf.nchan_keep_band);
	  break;
	}
    }

  /* Setup log interface */
  sprintf(log_fname, "%s/baseband2filterbank.log", conf.dir);
  conf.logfile = paf_log_open(log_fname, "ab+");
  if(conf.logfile == NULL)
    {
      fprintf(stderr, "Can not open log file %s\n", log_fname);
      exit(EXIT_FAILURE);
    }
  paf_log_add(conf.logfile, "INFO", 1, log_mutex, "BASEBAND2FILTERBANK START");

  /* Log the input */
  strcpy(command_line, argv[0]);
  for(i = 1; i < argc; i++)
    {
      strcat(command_line, " ");
      strcat(command_line, argv[i]);
    }
  paf_log_add(conf.logfile, "INFO", 1, log_mutex, "The command line is \"%s\"", command_line);
  paf_log_add(conf.logfile, "INFO", 1, log_mutex, "The input ring buffer key is %x", conf.key_in); 
  paf_log_add(conf.logfile, "INFO", 1, log_mutex, "The output ring buffer key is %x", conf.key_out);
  paf_log_add(conf.logfile, "INFO", 1, log_mutex, "Each input ring buffer block has %"PRIu64" packets per frequency chunk", conf.rbufin_ndf_chk);
  paf_log_add(conf.logfile, "INFO", 1, log_mutex, "%d streams run on GPU", conf.nstream);
  paf_log_add(conf.logfile, "INFO", 1, log_mutex, "Each stream process %d packets per frequency chunk", conf.stream_ndf_chk);
  paf_log_add(conf.logfile, "INFO", 1, log_mutex, "The runtime information is %s", conf.dir);
  if(conf.sod)
    paf_log_add(conf.logfile, "INFO", 1, log_mutex, "The filterbank data is enabled at the beginning");
  else
    paf_log_add(conf.logfile, "INFO", 1, log_mutex, "The filterbank data is NOT enabled at the beginning");
  paf_log_add(conf.logfile, "INFO", 1, log_mutex, "%d chunks of input data", conf.nchk_in);
  paf_log_add(conf.logfile, "INFO", 1, log_mutex, "We use %d points FFT", conf.cufft_nx);
  paf_log_add(conf.logfile, "INFO", 1, log_mutex, "We output %d channels", conf.nchan_out);
  paf_log_add(conf.logfile, "INFO", 1, log_mutex, "We keep %d fine channels for the whole band after FFT", conf.nchan_keep_band);
  
  /* init */
  init_baseband2filterbank(&conf);

  /* Play with data */
  baseband2filterbank(conf);

  /* Destroy */
  paf_log_add(conf.logfile, "INFO", 1, log_mutex, "BEFORE destroy");  
  destroy_baseband2filterbank(conf);
  paf_log_add(conf.logfile, "INFO", 1, log_mutex, "END destroy");
  
  /* Destory log interface */  
  paf_log_add(conf.logfile, "INFO", 1, log_mutex, "BASEBAND2FILTERBANK END");  
  paf_log_close(conf.logfile);
  
  return EXIT_SUCCESS;
}