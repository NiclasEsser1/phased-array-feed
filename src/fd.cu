#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>

#define CUDA_ERROR_CHECK
#define CudaSafeCall(err) __cudaSafeCall(err, __FILE__, __LINE__)
#define CudaSynchronizeCall()  __cudaSynchronizeCall(__FILE__, __LINE__)
#define CudaSafeKernelLaunch()  __CudaSafeKernelLaunch(__FILE__, __LINE__)

inline void __cudaSynchronizeCall(const char *file, const int line);
inline void __cudaSafeKernelLaunch(const char *file, const int line);
inline void __cudaSafeCall(hipError_t err, const char *file, const int line);

inline void __cudaSafeCall(hipError_t err, const char *file, const int line){
#ifdef CUDA_ERROR_CHECK
  if (hipSuccess != err) {
    fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n",
	    file, line, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
#endif
  
  return;
}

inline void __cudaSynchronizeCall(const char *file, const int line){
#ifdef CUDA_ERROR_CHECK
  hipError_t err = hipGetLastError();
  err = hipDeviceSynchronize();
  if(hipSuccess != err){
    fprintf(stderr, "cudaSynchronizeCall() with sync failed at %s:%i : %s\n",
	    file, line, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
#endif
  
  return;
}

// Macro to catch CUDA errors in kernel launches
inline void __CudaSafeKernelLaunch(const char *file, const int line){
#ifdef CUDA_ERROR_CHECK
  
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
    fprintf(stderr, "cudaSynchronizeCall() failed at %s:%i : %s\n",
	    file, line, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
#endif
}

#define MATRIX_SIZE 10000 // Size of the one dimensional of the matrix
#define NSMP        (MATRIX_SIZE*MATRIX_SIZE)

// y and z in grid to represent i in the fortran code
// x in block and x in grid to represent j in the fortran code
// "good" number picked here so that i do not need to check the boundary in kernel
#define BLOCK_X     100   // proposed block size x, better to be power 2
#define BLOCK_Y     1     // proposed block size y
#define BLOCK_Z     1     // proposed block size y
#define GRID_X      100    // proposed grid size x, better to be power 2
#define GRID_Y      100    // proposed grid size y, better to be power 2
#define GRID_Z      100    // proposed grid size y, better to be power 2

#define NITERATION  10

__global__ void kernel(double *a, double *b){
#if 0
program fd
    implicit none
 
   double precision, dimension(10000,10000) :: a, b
   double precision :: h
   integer :: i, j, n
 
    a=0.0d0
 
    ! set the static boundary conditions
    do i=1,10000
        a(i,1)=1.0d0
        a(i,10000)=2.0d0
        a(1,i)=3.0d0
        a(10000,i)=4.0d0
    end do
    b=a
 
    ! Run through the matrix solving for the stencil
    do n=1, 10
        do i=2, 9999
            do j=2, 9999
                b(i,j)=a(i,j)/2.0d0+a(i+1,j)/8.0d0+a(i-1,j)/8.0d0+a(i,j+1)/8.0d0+a(i,j-1)/8.0d0
            end do
        end do
 
        a=b
    end do
 
    write(*,*) a(5,5),a(1000,1000),a(9995,9995)
end program
#endif
      
  // y and z in grid to represent i in the fortran code 
  int i = blockIdx.y * gridDim.z +
    blockIdx.z;
  
  // x in block and x in grid to represent j in the fortran code
  int j = blockIdx.x * blockDim.x +
    threadIdx.x;  
  
  // Quick and dirty code here
  // The memory access here may not be fully optimized
  if((0 < i)&&(i < (MATRIX_SIZE-1)) && (0 < j)&&(j < (MATRIX_SIZE-1))){    
    b[i*MATRIX_SIZE+j] = a[i*MATRIX_SIZE+j]/2.0 +
      a[(i+1)*MATRIX_SIZE+j]/8.0 +
      a[(i-1)*MATRIX_SIZE+j]/8.0 +
      a[i*MATRIX_SIZE+j+1]/8.0 +
      a[i*MATRIX_SIZE+j-1]/8.0;
  }
}

int main(int argc, char *argv[]){
  // The size here is not optimized for sure, use this number just to make sure that I do not need to check the boundary inside the kernel
  // It is better to be power of 2, but again quick and dirty code here
  // The threads in one block can not exceed 512 for most platforms, here to be easier, set it to 100
  dim3 block_size = {BLOCK_X, BLOCK_Y, BLOCK_Z};
  
  // The blocks in on grid can be 65536x65536x65536
  // grid_size.y and grid_size.z to represent i in the fortran code
  // grid_size.x and block_size.x to represent j in the fortran code
  dim3 grid_size = {GRID_X, GRID_Y, GRID_Z};
  
  double *a_host = NULL;
  double *b_host = NULL;
  double *a_device = NULL;
  double *b_device = NULL;
  
  // Get required memory both in host and device
  int buffer_size = NSMP*sizeof(double);
  CudaSafeCall(hipHostMalloc((void **)&a_host, buffer_size, hipHostMallocDefault));  // Memory on host for a
  CudaSafeCall(hipHostMalloc((void **)&b_host, buffer_size, hipHostMallocDefault));  // Memory on host for b
  CudaSafeCall(hipMemset((void *)a_host, 0,    buffer_size));  // To be safe, set to zeros
  CudaSafeCall(hipMemset((void *)b_host, 0,    buffer_size));  // To be safe, set to zeros
  
  double *a_test = (double *)malloc(buffer_size);  // Memory on host for test a
  double *b_test = (double *)malloc(buffer_size);  // Memory on host for test b
  memset(a_test, 0x00, buffer_size); // To be safe, set to zeros
  memset(b_test, 0x00, buffer_size); // To be safe, set to zeros
  
  CudaSafeCall(hipMalloc((void **)&a_device, buffer_size));   // Memory on device for a
  CudaSafeCall(hipMalloc((void **)&b_device, buffer_size));   // Memory on device for b
  CudaSafeCall(hipMemset((void *)a_device, 0, buffer_size));  // To be safe, set to zeros
  CudaSafeCall(hipMemset((void *)b_device, 0, buffer_size));  // To be safe, set to zeros

  // Prepare initial input 
  for(int i = 0; i < MATRIX_SIZE; i++){
    a_host[i*MATRIX_SIZE]                 = 1.0;
    a_host[i*MATRIX_SIZE+MATRIX_SIZE-1]   = 2.0;
    a_host[i]                             = 3.0;
    a_host[(MATRIX_SIZE-1)*MATRIX_SIZE+i] = 4.0;
  }
  memcpy(b_host, a_host, buffer_size); // Copy a to b
  memcpy(a_test, a_host, buffer_size); // Copy to test buffer
  memcpy(b_test, b_host, buffer_size); // Copy to test buffer
  
  struct timespec start, stop;
  double elapsed_time;
#if 0
  /*
    First version use host memory to bridge memory copy, which is a bad idea
  */
  clock_gettime(CLOCK_REALTIME, &start);
  // Do the iteration
  for(int i = 0; i < NITERATION; i++){
    // Most strightforward way to do the iteration
    // Qucik and dirty code, can be improved if time premits
    // 1. Copy a and b host data into device
    // 2. Do the calculation
    // 3. Copy the b back to host
    // 4. Copy b host data to a host and return to 1.
    
    CudaSafeCall(cudaMemcpy(a_device, a_host, buffer_size, cudaMemcpyHostToDevice));  // Copy data into device
    CudaSafeCall(cudaMemcpy(b_device, b_host, buffer_size, cudaMemcpyHostToDevice));  // Copy data into device
    
    kernel<<<grid_size, block_size>>>(a_device, b_device);  
    CudaSafeKernelLaunch(); // No stream, not sync required
    
    CudaSafeCall(cudaMemcpy(b_host, b_device, buffer_size, cudaMemcpyDeviceToHost)); // Copy data from device, no need to copy a from device to host here
    memcpy(a_host, b_host, buffer_size); // Copy b to a on host
  }
  
  // Copy a from device to host in the end, now users are free to use the result on host
  CudaSafeCall(cudaMemcpy(a_host, a_device, buffer_size, cudaMemcpyDeviceToHost));
  CudaSafeCall(cudaMemcpy(b_host, b_device, buffer_size, cudaMemcpyDeviceToHost));
  clock_gettime(CLOCK_REALTIME, &stop);
  elapsed_time = (stop.tv_sec - start.tv_sec) + (stop.tv_nsec - start.tv_nsec)/1.0E9L;
  fprintf(stdout, "Elapse time for GPU version is %f seconds\n", elapsed_time);
#endif
  
  /*
    Second version which use direct device to device memcpy
  */
  clock_gettime(CLOCK_REALTIME, &start);  
  CudaSafeCall(hipMemcpy(a_device, a_host, buffer_size, hipMemcpyHostToDevice));  // Copy data into device
  CudaSafeCall(hipMemcpy(b_device, b_host, buffer_size, hipMemcpyHostToDevice));  // Copy data into device

  // Do the iteration
  for(int i = 0; i < NITERATION; i++){
    kernel<<<grid_size, block_size>>>(a_device, b_device);
    CudaSafeKernelLaunch(); // No stream, not sync required
    hipMemcpy(a_device, b_device, buffer_size, hipMemcpyDeviceToDevice);
  }
  
  // Copy a from device to host in the end, now users are free to use the result on host
  CudaSafeCall(hipMemcpy(a_host, a_device, buffer_size, hipMemcpyDeviceToHost));
  CudaSafeCall(hipMemcpy(b_host, b_device, buffer_size, hipMemcpyDeviceToHost));
  clock_gettime(CLOCK_REALTIME, &stop);
  elapsed_time = (stop.tv_sec - start.tv_sec) + (stop.tv_nsec - start.tv_nsec)/1.0E9L;
  fprintf(stdout, "Elapse time for GPU version is %f seconds\n", elapsed_time);
  
  // interesting to see how long the CPU code will take
  clock_gettime(CLOCK_REALTIME, &start);
  for(int k = 0; k < NITERATION; k++){    
    for(int i = 1; i < MATRIX_SIZE-1; i++){
      for(int j = 1; j < MATRIX_SIZE-1; j++){
	b_test[i*MATRIX_SIZE+j] = a_test[i*MATRIX_SIZE+j]/2.0 +
	  a_test[(i+1)*MATRIX_SIZE+j]/8.0 +
	  a_test[(i-1)*MATRIX_SIZE+j]/8.0 +
	  a_test[i*MATRIX_SIZE+j+1]/8.0 +
	  a_test[i*MATRIX_SIZE+j-1]/8.0;
      }
    }
    memcpy(a_test, b_test, buffer_size); // Copy b to a on host
  }
  clock_gettime(CLOCK_REALTIME, &stop);
  elapsed_time = (stop.tv_sec - start.tv_sec) + (stop.tv_nsec - start.tv_nsec)/1.0E9L;
  fprintf(stdout, "Elapse time for CPU version is %f seconds\n", elapsed_time);

  // Check the rest?
  for(int i = 0; i < NSMP; i++){
    if((a_test[i]!=a_host[i])||(b_test[i]!=b_host[i])){
      std::cout<<i<<" "<<a_test[i]<<" "<<a_host[i]<<" "<<b_test[i]<<" "<<b_host[i]<<" "<<std::endl;
    }
  }
  
  // Free the memory space
  CudaSafeCall(hipHostFree(a_host));
  CudaSafeCall(hipHostFree(b_host));
  CudaSafeCall(hipFree(a_device));
  CudaSafeCall(hipFree(b_device));
  free(a_test);
  free(b_test);
  
  return EXIT_SUCCESS;
}
