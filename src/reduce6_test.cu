#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <inttypes.h>
#include "cudautil.cuh"
#include "kernel.cuh"

#include "constants.h"

#define NBYTE_RT   8

extern "C" void usage ()
{
  fprintf (stdout,
	   "reduce6_test - Test the reduce6 kernel \n"
	   "\n"
	   "Usage: reduce6_test [options]\n"
	   " -a  Grid size in X\n"
	   " -b  Grid size in Y\n"
	   " -c  Block size in X\n"
	   " -d  Number of samples to accumulate in each block\n"
	   " -h  show help\n");
}

// ./reduce6_test -a 512 -b 1 -c 512 -d 1024
int main(int argc, char *argv[])
{
  int i, j, arg;
  int grid_x, grid_y, block_x;
  uint64_t n_accumulate;
  uint64_t len_in, len_out, idx;
  dim3 gridsize_reduce6, blocksize_reduce6;
  float h_total = 0, g_total = 0;
  hipfftComplex *h_result = NULL, *g_result = NULL, *data = NULL, *g_in = NULL, *g_out = NULL;
  
  /* Read in parameters, the arguments here have the same name  */
  while((arg=getopt(argc,argv,"a:b:hc:d:")) != -1)
    {
      switch(arg)
	{
	case 'h':
	  usage();
	  exit(EXIT_FAILURE);	  

	case 'a':	  
	  if (sscanf (optarg, "%d", &grid_x) != 1)
	    {
	      fprintf (stderr, "Does not get grid_x, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	      exit(EXIT_FAILURE);
	    }
	  break;
	  
	case 'b':	  
	  if (sscanf (optarg, "%d", &grid_y) != 1)
	    {
	      fprintf (stderr, "Does not get grid_y, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	      exit(EXIT_FAILURE);
	    }
	  break;
	  
	case 'c':	  
	  if (sscanf (optarg, "%d", &block_x) != 1)
	    {
	      fprintf (stderr, "Does not get block_x, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	      exit(EXIT_FAILURE);
	    }
	  break;
	  
	case 'd':	  
	  if (sscanf (optarg, "%"SCNu64"", &n_accumulate) != 1)
	    {
	      fprintf (stderr, "Does not get n_accumulate, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	      exit(EXIT_FAILURE);
	    }
	  fprintf(stdout, "n_accumulate is %"SCNu64"\n",  n_accumulate);
	  break;
	}
    }

  fprintf(stdout, "grid_x is %d, grid_y is %d, block_x is %d and n_accumulate is %"SCNu64"\n", grid_x, grid_y, block_x, n_accumulate);
  
  /* Setup size */
  gridsize_reduce6.x  = grid_x;
  gridsize_reduce6.y  = grid_y;
  gridsize_reduce6.z  = 1;
  blocksize_reduce6.x = block_x;
  blocksize_reduce6.y = 1;
  blocksize_reduce6.z = 1;
  len_out             = grid_x*grid_y;
  len_in              = len_out*n_accumulate;

  /* Create buffer */
  CudaSafeCall(hipHostMalloc((void **)&data,     len_in * sizeof(hipfftComplex)));
  CudaSafeCall(hipHostMalloc((void **)&h_result, len_out * sizeof(hipfftComplex)));
  CudaSafeCall(hipHostMalloc((void **)&g_result, len_out * sizeof(hipfftComplex)));
  CudaSafeCall(hipMalloc((void **)&g_in,         len_in * sizeof(hipfftComplex)));
  CudaSafeCall(hipMalloc((void **)&g_out,        len_out * sizeof(hipfftComplex)));

  /* cauculate on CPU */
  srand(time(NULL));
  for(i = 0; i < len_out; i ++)
    {
      h_result[i].x = 0;
      h_result[i].y = 0;
      for(j = 0; j < n_accumulate; j++)
	{
	  idx = i * n_accumulate + j;
	  data[idx].x = (float)rand()/(float)(RAND_MAX/(float)MAX_RAND);
	  data[idx].y = (float)rand()/(float)(RAND_MAX/(float)MAX_RAND);

	  h_result[i].x += data[idx].x;
	  h_result[i].y += data[idx].y;
	}
    }
  
  /* Calculate on GPU */
  CudaSafeCall(hipMemcpy(g_in, data, len_in * sizeof(hipfftComplex), hipMemcpyHostToDevice));
  switch (blocksize_reduce6.x)
    {
    case 1024:
      reduce6_kernel<1024><<<gridsize_reduce6, blocksize_reduce6, blocksize_reduce6.x * NBYTE_RT>>>(g_in, g_out, n_accumulate);
      break;
      
    case 512:
      reduce6_kernel< 512><<<gridsize_reduce6, blocksize_reduce6, blocksize_reduce6.x * NBYTE_RT>>>(g_in, g_out, n_accumulate);
      break;
      
    case 256:
      reduce6_kernel< 256><<<gridsize_reduce6, blocksize_reduce6, blocksize_reduce6.x * NBYTE_RT>>>(g_in, g_out, n_accumulate);
      break;
      
    case 128:
      reduce6_kernel< 128><<<gridsize_reduce6, blocksize_reduce6, blocksize_reduce6.x * NBYTE_RT>>>(g_in, g_out, n_accumulate);
      break;
      
    case 64:
      reduce6_kernel<  64><<<gridsize_reduce6, blocksize_reduce6, blocksize_reduce6.x * NBYTE_RT>>>(g_in, g_out, n_accumulate);
      break;
      
    case 32:
      reduce6_kernel<  32><<<gridsize_reduce6, blocksize_reduce6, blocksize_reduce6.x * NBYTE_RT>>>(g_in, g_out, n_accumulate);
      break;
      
    case 16:
      reduce6_kernel<  16><<<gridsize_reduce6, blocksize_reduce6, blocksize_reduce6.x * NBYTE_RT>>>(g_in, g_out, n_accumulate);
      break;
      
    case 8:
      reduce6_kernel<   8><<<gridsize_reduce6, blocksize_reduce6, blocksize_reduce6.x * NBYTE_RT>>>(g_in, g_out, n_accumulate);
      break;
      
    case 4:
      reduce6_kernel<   4><<<gridsize_reduce6, blocksize_reduce6, blocksize_reduce6.x * NBYTE_RT>>>(g_in, g_out, n_accumulate);
      break;
      
    case 2:
      reduce6_kernel<   2><<<gridsize_reduce6, blocksize_reduce6, blocksize_reduce6.x * NBYTE_RT>>>(g_in, g_out, n_accumulate);
      break;
      
    case 1:
      reduce6_kernel<   1><<<gridsize_reduce6, blocksize_reduce6, blocksize_reduce6.x * NBYTE_RT>>>(g_in, g_out, n_accumulate);
      break;
    }
  CHECK_LAUNCH_ERROR();
  CudaSafeCall(hipMemcpy(g_result, g_out, len_out * sizeof(hipfftComplex), hipMemcpyDeviceToHost));

  /* Check the result */
  for(i = 0; i < len_out; i++)
    {
      h_total += (h_result[i].x + h_result[i].y);
      g_total += (g_result[i].x + g_result[i].y);
    }
  //fprintf(stdout, "%f\t%f\t%E\n", h_total, g_total, (g_total - h_total)/h_total);
  fprintf(stdout, "CPU:\t%f\nGPU:\t%f\n%E\n", h_total, g_total, (g_total - h_total)/h_total);
  
  /* Free buffer */
  CudaSafeCall(hipHostFree(data));
  CudaSafeCall(hipHostFree(h_result));
  CudaSafeCall(hipHostFree(g_result));
  CudaSafeCall(hipFree(g_in));
  CudaSafeCall(hipFree(g_out));
  
  return EXIT_SUCCESS;
}