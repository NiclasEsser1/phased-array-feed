#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <inttypes.h>
#include "cudautil.cuh"
#include "kernel.cuh"

#include "constants.h"

#define NBYTE_RT   8
#define MAX_RAND       1000

extern "C" void usage ()
{
  fprintf (stdout,
	   "accumulate_test - Test the accumulate kernel \n"
	   "\n"
	   "Usage: accumulate_test [options]\n"
	   " -a  Grid size in X\n"
	   " -b  Grid size in Y\n"
	   " -c  Block size in X\n"
	   " -d  Number of samples to accumulate in each block\n"
	   " -h  show help\n");
}

// ./accumulate_test -a 512 -b 1 -c 512 -d 1024
int main(int argc, char *argv[])
{
  int i, j, arg;
  int grid_x, grid_y, block_x;
  uint64_t n_accumulate;
  uint64_t len_in, len_out, idx;
  dim3 gridsize_accumulate, blocksize_accumulate;
  float h_total = 0, g_total = 0;
  hipfftComplex *h_result = NULL, *g_result = NULL, *data = NULL, *g_in = NULL, *g_out = NULL;
  
  /* Read in parameters, the arguments here have the same name  */
  while((arg=getopt(argc,argv,"a:b:hc:d:")) != -1)
    {
      switch(arg)
	{
	case 'h':
	  usage();
	  exit(EXIT_FAILURE);	  

	case 'a':	  
	  if (sscanf (optarg, "%d", &grid_x) != 1)
	    {
	      fprintf (stderr, "Does not get grid_x, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	      exit(EXIT_FAILURE);
	    }
	  break;
	  
	case 'b':	  
	  if (sscanf (optarg, "%d", &grid_y) != 1)
	    {
	      fprintf (stderr, "Does not get grid_y, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	      exit(EXIT_FAILURE);
	    }
	  break;
	  
	case 'c':	  
	  if (sscanf (optarg, "%d", &block_x) != 1)
	    {
	      fprintf (stderr, "Does not get block_x, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	      exit(EXIT_FAILURE);
	    }
	  break;
	  
	case 'd':	  
	  if (sscanf (optarg, "%"SCNu64"", &n_accumulate) != 1)
	    {
	      fprintf (stderr, "Does not get n_accumulate, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	      exit(EXIT_FAILURE);
	    }
	  fprintf(stdout, "n_accumulate is %"SCNu64"\n",  n_accumulate);
	  break;
	}
    }

  fprintf(stdout, "grid_x is %d, grid_y is %d, block_x is %d and n_accumulate is %"SCNu64"\n", grid_x, grid_y, block_x, n_accumulate);
  
  /* Setup size */
  gridsize_accumulate.x  = grid_x;
  gridsize_accumulate.y  = grid_y;
  gridsize_accumulate.z  = 1;
  blocksize_accumulate.x = block_x;
  blocksize_accumulate.y = 1;
  blocksize_accumulate.z = 1;
  len_out                = grid_x*grid_y;
  len_in                 = len_out*n_accumulate;

  /* Create buffer */
  CudaSafeCall(hipHostMalloc((void **)&data,     len_in * sizeof(hipfftComplex)));
  CudaSafeCall(hipHostMalloc((void **)&h_result, len_out * sizeof(hipfftComplex)));
  CudaSafeCall(hipHostMalloc((void **)&g_result, len_out * sizeof(hipfftComplex)));
  CudaSafeCall(hipMalloc((void **)&g_in,         len_in * sizeof(hipfftComplex)));
  CudaSafeCall(hipMalloc((void **)&g_out,        len_out * sizeof(hipfftComplex)));

  /* cauculate on CPU */
  srand(time(NULL));
  for(i = 0; i < len_out; i ++)
    {
      h_result[i].x = 0;
      h_result[i].y = 0;
      for(j = 0; j < n_accumulate; j++)
	{
	  idx = i * n_accumulate + j;
	  data[idx].x = (float)rand()/(float)(RAND_MAX/(float)MAX_RAND);
	  data[idx].y = (float)rand()/(float)(RAND_MAX/(float)MAX_RAND);

	  h_result[i].x += data[idx].x;
	  h_result[i].y += data[idx].y;
	}
    }
  
  /* Calculate on GPU */
  CudaSafeCall(hipMemcpy(g_in, data, len_in * sizeof(hipfftComplex), hipMemcpyHostToDevice));
  accumulate_kernel<<<gridsize_accumulate, blocksize_accumulate, blocksize_accumulate.x * NBYTE_RT>>>(g_in, g_out);
  
  CHECK_LAUNCH_ERROR();
  CudaSafeCall(hipMemcpy(g_result, g_out, len_out * sizeof(hipfftComplex), hipMemcpyDeviceToHost));

  /* Check the result */
  for(i = 0; i < len_out; i++)
    {
      h_total += (h_result[i].x + h_result[i].y);
      g_total += (g_result[i].x + g_result[i].y);
    }
  //fprintf(stdout, "%f\t%f\t%E\n", h_total, g_total, (g_total - h_total)/h_total);
  fprintf(stdout, "CPU:\t%f\nGPU:\t%f\n%E\n", h_total, g_total, (g_total - h_total)/h_total);
  
  /* Free buffer */
  CudaSafeCall(hipHostFree(data));
  CudaSafeCall(hipHostFree(h_result));
  CudaSafeCall(hipHostFree(g_result));
  CudaSafeCall(hipFree(g_in));
  CudaSafeCall(hipFree(g_out));
  
  return EXIT_SUCCESS;
}