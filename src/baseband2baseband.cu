#include "hip/hip_runtime.h"
#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

#include <time.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>
#include <math.h>
#include <arpa/inet.h>
#include <sys/socket.h>
#include <netinet/in.h>
#include <hip/hip_runtime_api.h>
#include <unistd.h>

#include "log.h"
#include "baseband2baseband.cuh"
#include "cudautil.cuh"
#include "kernel.cuh"
#include "constants.h"

int default_arguments(conf_t *conf)
{
  memset(conf->dir, 0x00, sizeof(conf->dir));
  sprintf(conf->dir, "unset"); // Default with "unset"
  memset(conf->ip, 0x00, sizeof(conf->ip));
  sprintf(conf->ip, "unset"); // Default with "unset"
  
  conf->ndf_per_chunk_rbufin = 0; // Default with an impossible value
  conf->nstream          = -1; // Default with an impossible value
  conf->ndf_per_chunk_stream = 0; // Default with an impossible value
  conf->nchunk = -1;
  conf->cufft_nx = -1;
  conf->sod = -1;
  
  conf->port = -1;
  conf->fits_flag = 0; // default not use FITSWriter interface
  conf->pol_type = -1;
  
  return EXIT_SUCCESS;
}

int initialize_baseband2baseband(conf_t *conf)
{
  int i;
  int iembed1, istride1, idist1, oembed1, ostride1, odist1, batch1, nx1;
  int iembed2, istride2, idist2, oembed2, ostride2, odist2, batch2, nx2;
  uint64_t hdrsz;
  uint64_t naccumulate_pow2;

  conf->nrepeat_per_blk = conf->ndf_per_chunk_rbufin / (conf->ndf_per_chunk_stream * conf->nstream);
  conf->nchan = conf->nchunk * NCHAN_PER_CHUNK;
  conf->nchan_keep_chan = (int)(conf->cufft_nx / OVER_SAMP_RATE);
  conf->cufft_mod = (int)(0.5 * conf->nchan_keep_chan);
  
  log_add(conf->log_file, "INFO", 1,  "We have %d channels input", conf->nchan);
  log_add(conf->log_file, "INFO", 1,  "The mod to reduce oversampling is %d", conf->cufft_mod);
  log_add(conf->log_file, "INFO", 1,  "We will keep %d fine channels for each input channel after FFT", conf->nchan_keep_chan);
  log_add(conf->log_file, "INFO", 1,  "%d run to finish one ring buffer block", conf->nrepeat_per_blk);
  
  conf->fits = NULL;
  if(conf->fits_flag == 1)
    {
      conf->nseg_per_blk = conf->nstream * conf->nrepeat_per_blk;
      conf->neth_per_blk = conf->nseg_per_blk * NDATA_PER_SAMP_FULL;
      conf->fits         = (fits_t *)malloc(conf->neth_per_blk * sizeof(fits_t));
      for(i = 0; i < conf->neth_per_blk; i++)
	{
	  memset(conf->fits[i].data, 0x00, UDP_PAYLOAD_SIZE_MAX);
	  hipHostRegister ((void *) conf->fits[i].data, sizeof(conf->fits[i].data), 0);
	}
      log_add(conf->log_file, "INFO", 1,  "%d network packets are requied for each buffer block", conf->neth_per_blk);
      
      conf->dtsz_network    = NBYTE_FLOAT * conf->nchan;
      conf->pktsz_network   = conf->dtsz_network + 3 * NBYTE_FLOAT + 6 * NBYTE_INT + FITS_TIME_STAMP_LEN;
      log_add(conf->log_file, "INFO", 1,  "Network data size is %d", conf->dtsz_network);
      log_add(conf->log_file, "INFO", 1,  "Network packet size is %d", conf->pktsz_network); 
    }
  
  /* Prepare buffer, stream and fft plan for process */
  conf->ndim_scale = conf->ndf_per_chunk_rbufin * NSAMP_DF * NPOL_BASEBAND * NDIM_BASEBAND / OVER_SAMP_RATE; // Only works when two polarisations has similar power level
  conf->scale_dtsz = NBYTE_FOLD /((double)NBYTE_BASEBAND * OVER_SAMP_RATE);
  log_add(conf->log_file, "INFO", 1,  "ndim_scale %f", conf->ndim_scale);
  log_add(conf->log_file, "INFO", 1,  "The data size rate is %f", conf->scale_dtsz);
  
  conf->nsamp_in  = conf->ndf_per_chunk_stream * conf->nchan * NSAMP_DF;  // For each stream
  conf->npol_in   = conf->nsamp_in * NPOL_BASEBAND;
  conf->ndata_in  = conf->npol_in  * NDIM_BASEBAND;
  log_add(conf->log_file, "INFO", 1,  "nsamp_in %"PRIu64", npol_in %"PRIu64", ndata_in %"PRIu64"", conf->nsamp_in, conf->npol_in, conf->ndata_in);
  
  conf->nsamp_keep  = conf->nsamp_in / OVER_SAMP_RATE;
  conf->npol_keep   = conf->nsamp_keep * NPOL_BASEBAND;
  conf->ndata_keep  = conf->npol_keep  * NDIM_BASEBAND;
  log_add(conf->log_file, "INFO", 1,  "nsamp_keep %"PRIu64", npol_keep %"PRIu64", ndata_keep %"PRIu64"", conf->nsamp_keep, conf->npol_keep, conf->ndata_keep);
  
  conf->nsamp_out      = conf->nchan;
  conf->ndata_out      = conf->nsamp_out  * NDATA_PER_SAMP_RT;
  log_add(conf->log_file, "INFO", 1,  "nsamp_out %"PRIu64", ndata_out %"PRIu64"", conf->nsamp_out, conf->ndata_out);
  
  nx1        = conf->cufft_nx;
  batch1     = conf->npol_in / conf->cufft_nx;
  log_add(conf->log_file, "INFO", 1,  "nx1 %d, batch1 %d", nx1, batch1);
  
  iembed1    = nx1;
  istride1   = 1;
  idist1     = nx1;
  
  oembed1    = nx1;
  ostride1   = 1;
  odist1     = nx1;
  
  nx2        = conf->nchan_keep_chan;
  batch2     = conf->npol_keep / conf->nchan_keep_chan;
  log_add(conf->log_file, "INFO", 1,  "nx2 %d, batch2 %d", nx2, batch2);
  
  iembed2    = nx2;
  istride2   = 1;
  idist2     = nx2;
  
  oembed2    = nx2;
  ostride2   = 1;
  odist2     = nx2;

  conf->streams = NULL;
  conf->fft_plans_forward = NULL;
  conf->fft_plans_backward = NULL;
  conf->streams = (hipStream_t *)malloc(conf->nstream * sizeof(hipStream_t));
  conf->fft_plans_forward = (hipfftHandle *)malloc(conf->nstream * sizeof(hipfftHandle));
  conf->fft_plans_backward = (hipfftHandle *)malloc(conf->nstream * sizeof(hipfftHandle));
  for(i = 0; i < conf->nstream; i ++)
    {
      CudaSafeCall(hipStreamCreate(&conf->streams[i]));
      CufftSafeCall(hipfftPlanMany(&conf->fft_plans_forward[i], CUFFT_RANK, &nx1, &iembed1, istride1, idist1, &oembed1, ostride1, odist1, HIPFFT_C2C, batch1));
      CufftSafeCall(hipfftPlanMany(&conf->fft_plans_backward[i], CUFFT_RANK, &nx2, &iembed2, istride2, idist2, &oembed2, ostride2, odist2, HIPFFT_C2C, batch2));
      
      CufftSafeCall(hipfftSetStream(conf->fft_plans_forward[i], conf->streams[i]));
      CufftSafeCall(hipfftSetStream(conf->fft_plans_backward[i], conf->streams[i]));
    }
  
  conf->sbufin_size    = conf->ndata_in * NBYTE_BASEBAND;
  conf->sbufout_size   = conf->ndata_keep * NBYTE_FOLD;
  conf->sbufout_size_monitor   = conf->ndata_out * NBYTE_FLOAT;
  log_add(conf->log_file, "INFO", 1, 
	  "sbufin_size %"PRIu64", sbufout_size %"PRIu64" and sbufout_size_monitor %"PRIu64"",
	  conf->sbufin_size,
	  conf->sbufout_size,
	  conf->sbufout_size_monitor);
  
  conf->bufin_size     = conf->nstream * conf->sbufin_size;
  conf->bufout_size    = conf->nstream * conf->sbufout_size;
  conf->bufout_size_monitor    = conf->nstream * conf->sbufout_size_monitor;
  log_add(conf->log_file, "INFO", 1, 
	  "bufin_size %"PRIu64", bufout_size %"PRIu64" and bufout_size_monitor %"PRIu64"",
	  conf->bufin_size,
	  conf->bufout_size,
	  conf->bufout_size_monitor);

  conf->sbufrt1_size = conf->npol_in * NBYTE_CUFFT_COMPLEX;
  conf->sbufrt2_size = conf->npol_keep * NBYTE_CUFFT_COMPLEX;
  conf->bufrt1_size  = conf->nstream * conf->sbufrt1_size;
  conf->bufrt2_size  = conf->nstream * conf->sbufrt2_size;
  log_add(conf->log_file, "INFO", 1, 
	  "sbufrt1_size %"PRIu64", sbufrt2_size %"PRIu64", bufrt1_size %"PRIu64" and bufrt2_size %"PRIu64"",
	  conf->sbufrt1_size,
	  conf->sbufrt2_size,
	  conf->bufrt1_size,
	  conf->bufrt2_size);
  
  conf->hbufin_offset = conf->sbufin_size / NBYTE_CHAR;
  conf->dbufin_offset = conf->sbufin_size / (NBYTE_BASEBAND * NPOL_BASEBAND * NDIM_BASEBAND);
  conf->bufrt1_offset = conf->sbufrt1_size / NBYTE_CUFFT_COMPLEX;
  conf->bufrt2_offset = conf->sbufrt2_size / NBYTE_CUFFT_COMPLEX;
  log_add(conf->log_file, "INFO", 1, 
	  "hbufin_offset %"PRIu64", dbufin_offset %"PRIu64", bufrt1_offset %"PRIu64" and bufrt2_offset %"PRIu64"",
	  conf->hbufin_offset,
	  conf->dbufin_offset,
	  conf->bufrt1_offset,
	  conf->bufrt2_offset);
  
  conf->dbufout_offset   = conf->sbufout_size / NBYTE_FOLD;
  conf->hbufout_offset   = conf->sbufout_size;
  conf->dbufout_offset_monitor   = conf->sbufout_size_monitor / NBYTE_FLOAT;
  log_add(conf->log_file, "INFO", 1, 
	  "dbufout_offset %"PRIu64", hbufout_offset %"PRIu64" and dbufout_offset_monitor %"PRIu64"",
	  conf->dbufout_offset,
	  conf->hbufout_offset,
	  conf->dbufout_offset_monitor);

  conf->dbuf_in = NULL;
  conf->dbuf_out = NULL;
  conf->dbuf_out_monitor = NULL;
  conf->buf_rt1 = NULL;
  conf->buf_rt2 = NULL;
  conf->offset_scale_d = NULL;
  conf->offset_scale_h = NULL;
  CudaSafeCall(hipMalloc((void **)&conf->dbuf_in, conf->bufin_size));  
  CudaSafeCall(hipMalloc((void **)&conf->dbuf_out, conf->bufout_size));
  CudaSafeCall(hipMalloc((void **)&conf->dbuf_out_monitor, conf->bufout_size_monitor));
  CudaSafeCall(hipMalloc((void **)&conf->buf_rt1, conf->bufrt1_size));
  CudaSafeCall(hipMalloc((void **)&conf->buf_rt2, conf->bufrt2_size)); 
  CudaSafeCall(hipMalloc((void **)&conf->offset_scale_d, conf->nstream * conf->nchan * NBYTE_CUFFT_COMPLEX));
  CudaSafeCall(hipHostMalloc((void **)&conf->offset_scale_h, conf->nchan * NBYTE_CUFFT_COMPLEX));
  CudaSafeCall(hipMemset((void *)conf->offset_scale_d, 0, sizeof(conf->offset_scale_d)));// We have to clear the memory for this parameter
  CudaSafeCall(hipMemset((void *)conf->dbuf_out_monitor, 0, sizeof(conf->dbuf_out_monitor)));// We have to clear the memory for this parameter
  
  /* Prepare the setup of kernels */
  conf->gridsize_unpack.x = conf->ndf_per_chunk_stream;
  conf->gridsize_unpack.y = conf->nchunk;
  conf->gridsize_unpack.z = 1;
  conf->blocksize_unpack.x = NSAMP_DF; 
  conf->blocksize_unpack.y = NCHAN_PER_CHUNK;
  conf->blocksize_unpack.z = 1;
  log_add(conf->log_file, "INFO", 1,  "The configuration of unpack kernel is (%d, %d, %d) and (%d, %d, %d)",
	  conf->gridsize_unpack.x, conf->gridsize_unpack.y, conf->gridsize_unpack.z,
	  conf->blocksize_unpack.x, conf->blocksize_unpack.y, conf->blocksize_unpack.z);
  
  conf->naccumulate = conf->ndf_per_chunk_stream * NSAMP_DF / OVER_SAMP_RATE * NPOL_BASEBAND;
  naccumulate_pow2  = (uint64_t)pow(2.0, floor(log2((double)conf->naccumulate)));
  conf->gridsize_taccumulate.x = conf->nchan;
  conf->gridsize_taccumulate.y = 1;
  conf->gridsize_taccumulate.z = 1;
  conf->blocksize_taccumulate.x = (naccumulate_pow2<1024)?naccumulate_pow2:1024;
  conf->blocksize_taccumulate.y = 1;
  conf->blocksize_taccumulate.z = 1;
  log_add(conf->log_file, "INFO", 1,  "naccumulate is %"PRIu64" and naccumulate_pow2 is %"PRIu64"", conf->naccumulate, naccumulate_pow2);
  log_add(conf->log_file, "INFO", 1,  "The configuration of taccumulate kernel is (%d, %d, %d) and (%d, %d, %d)",
	  conf->gridsize_taccumulate.x, conf->gridsize_taccumulate.y, conf->gridsize_taccumulate.z,
	  conf->blocksize_taccumulate.x, conf->blocksize_taccumulate.y, conf->blocksize_taccumulate.z);
  
  conf->gridsize_scale.x = 1;
  conf->gridsize_scale.y = 1;
  conf->gridsize_scale.z = 1;
  conf->blocksize_scale.x = conf->nchan;
  conf->blocksize_scale.y = 1;
  conf->blocksize_scale.z = 1;
  log_add(conf->log_file, "INFO", 1,  "The configuration of scale kernel is (%d, %d, %d) and (%d, %d, %d)",
	  conf->gridsize_scale.x, conf->gridsize_scale.y, conf->gridsize_scale.z,
	  conf->blocksize_scale.x, conf->blocksize_scale.y, conf->blocksize_scale.z);
  
  conf->gridsize_swap_select_transpose_swap.x = conf->nchan;
  conf->gridsize_swap_select_transpose_swap.y = conf->ndf_per_chunk_stream * NSAMP_DF / conf->cufft_nx;
  conf->gridsize_swap_select_transpose_swap.z = 1;  
  conf->blocksize_swap_select_transpose_swap.x = conf->cufft_nx;
  conf->blocksize_swap_select_transpose_swap.y = 1;
  conf->blocksize_swap_select_transpose_swap.z = 1;
  log_add(conf->log_file, "INFO", 1,  "The configuration of swap_select_transpose_swap kernel is (%d, %d, %d) and (%d, %d, %d)",
	  conf->gridsize_swap_select_transpose_swap.x, conf->gridsize_swap_select_transpose_swap.y, conf->gridsize_swap_select_transpose_swap.z,
	  conf->blocksize_swap_select_transpose_swap.x, conf->blocksize_swap_select_transpose_swap.y, conf->blocksize_swap_select_transpose_swap.z);
  
  conf->gridsize_transpose_pad.x = conf->ndf_per_chunk_stream * NSAMP_DF / conf->cufft_nx; 
  conf->gridsize_transpose_pad.y = conf->nchan;
  conf->gridsize_transpose_pad.z = 1;
  conf->blocksize_transpose_pad.x = conf->nchan_keep_chan;
  conf->blocksize_transpose_pad.y = 1;
  conf->blocksize_transpose_pad.z = 1;
  log_add(conf->log_file, "INFO", 1,  "The configuration of transpose_pad kernel is (%d, %d, %d) and (%d, %d, %d)",
	  conf->gridsize_transpose_pad.x, conf->gridsize_transpose_pad.y, conf->gridsize_transpose_pad.z,
	  conf->blocksize_transpose_pad.x, conf->blocksize_transpose_pad.y, conf->blocksize_transpose_pad.z);
  
  conf->gridsize_transpose_scale.x = ceil(conf->nchan_keep_chan / (double)TILE_DIM);  
  conf->gridsize_transpose_scale.y = ceil(conf->nchan / (double)TILE_DIM);
  conf->gridsize_transpose_scale.z = conf->ndf_per_chunk_stream * NSAMP_DF / conf->cufft_nx; 
  conf->blocksize_transpose_scale.x = TILE_DIM;
  conf->blocksize_transpose_scale.y = NROWBLOCK_TRANS;
  conf->blocksize_transpose_scale.z = 1;
  log_add(conf->log_file, "INFO", 1,  "The configuration of transpose_scale kernel is (%d, %d, %d) and (%d, %d, %d)",
	  conf->gridsize_transpose_scale.x, conf->gridsize_transpose_scale.y, conf->gridsize_transpose_scale.z,
	  conf->blocksize_transpose_scale.x, conf->blocksize_transpose_scale.y, conf->blocksize_transpose_scale.z);
    
  conf->gridsize_transpose_complex.x = conf->ndf_per_chunk_stream * NSAMP_DF / conf->cufft_nx;
  conf->gridsize_transpose_complex.y = conf->nchan;
  conf->gridsize_transpose_complex.z = 1;  
  conf->blocksize_transpose_complex.x = conf->nchan_keep_chan;
  conf->blocksize_transpose_complex.y = 1;
  conf->blocksize_transpose_complex.z = 1;  
  log_add(conf->log_file, "INFO", 1,  "The configuration of transpose_complex kernel is (%d, %d, %d) and (%d, %d, %d)",
	  conf->gridsize_transpose_complex.x, conf->gridsize_transpose_complex.y, conf->gridsize_transpose_complex.z,
	  conf->blocksize_transpose_complex.x, conf->blocksize_transpose_complex.y, conf->blocksize_transpose_complex.z);

  conf->naccumulate_spectral = conf->ndf_per_chunk_stream * NSAMP_DF / OVER_SAMP_RATE;
  naccumulate_pow2  = (uint64_t)pow(2.0, floor(log2((double)conf->naccumulate_spectral)));
  conf->gridsize_spectral_taccumulate.x = conf->nchan;
  conf->gridsize_spectral_taccumulate.y = 1;
  conf->gridsize_spectral_taccumulate.z = 1;
  conf->blocksize_spectral_taccumulate.x = (naccumulate_pow2<1024)?naccumulate_pow2:1024;
  conf->blocksize_spectral_taccumulate.y = 1;
  conf->blocksize_spectral_taccumulate.z = 1;
  log_add(conf->log_file, "INFO", 1,  "naccumulate_spectral is %"PRIu64" and naccumulate_pow2 is %"PRIu64"", conf->naccumulate_spectral, naccumulate_pow2);
  log_add(conf->log_file, "INFO", 1, 
	  "The configuration of spectral_taccumulate kernel is (%d, %d, %d) and (%d, %d, %d)",
	  conf->gridsize_spectral_taccumulate.x,
	  conf->gridsize_spectral_taccumulate.y,
	  conf->gridsize_spectral_taccumulate.z,
	  conf->blocksize_spectral_taccumulate.x,
	  conf->blocksize_spectral_taccumulate.y,
	  conf->blocksize_spectral_taccumulate.z);
  
  /* attach to input ring buffer */
  conf->hdu_in = dada_hdu_create(NULL);
  dada_hdu_set_key(conf->hdu_in, conf->key_in);
  if(dada_hdu_connect(conf->hdu_in) < 0)
    {
      log_add(conf->log_file, "ERR", 1,  "Can not connect to hdu, which happens at \"%s\", line [%d].", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: Can not connect to hdu, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      destroy_baseband2baseband(*conf);
      exit(EXIT_FAILURE);    
    }  
  conf->db_in = (ipcbuf_t *) conf->hdu_in->data_block;
  conf->rbufin_size = ipcbuf_get_bufsz(conf->db_in);
  log_add(conf->log_file, "INFO", 1,  "Input buffer block size is %"PRIu64".", conf->rbufin_size);
  
  if(conf->rbufin_size != conf->bufin_size*conf->nrepeat_per_blk)  
    {
      log_add(conf->log_file, "ERR", 1,  "Buffer size mismatch, which happens at \"%s\", line [%d].", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      destroy_baseband2baseband(*conf);
      exit(EXIT_FAILURE);    
    }
  struct timespec start, stop;
  double elapsed_time;
  clock_gettime(CLOCK_REALTIME, &start);
  dada_cuda_dbregister(conf->hdu_in);  // registers the existing host memory range for use by CUDA
  clock_gettime(CLOCK_REALTIME, &stop);
  elapsed_time = (stop.tv_sec - start.tv_sec) + (stop.tv_nsec - start.tv_nsec)/1.0E9L;
  fprintf(stdout, "elapsed_time for dbregister of input ring buffer is %f\n", elapsed_time);
  fflush(stdout);

  hdrsz = ipcbuf_get_bufsz(conf->hdu_in->header_block);  
  if(hdrsz != DADA_DEFAULT_HEADER_SIZE)    // This number should match
    {
      log_add(conf->log_file, "ERR", 1,  "Buffer size mismatch, which happens at \"%s\", line [%d].", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      destroy_baseband2baseband(*conf);
      exit(EXIT_FAILURE);    
    }
  if(dada_hdu_lock_read(conf->hdu_in) < 0) // make ourselves the read client 
    {
      log_add(conf->log_file, "ERR", 1,  "Error locking HDU, which happens at \"%s\", line [%d].", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: Error locking HDU, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      destroy_baseband2baseband(*conf);
      exit(EXIT_FAILURE);
    }

  /* Prepare output ring buffer */
  conf->hdu_out = dada_hdu_create(NULL);
  dada_hdu_set_key(conf->hdu_out, conf->key_out);
  if(dada_hdu_connect(conf->hdu_out) < 0)
    {
      log_add(conf->log_file, "ERR", 1,  "Can not connect to hdu, which happens at \"%s\", line [%d].", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: Can not connect to hdu, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      destroy_baseband2baseband(*conf);
      exit(EXIT_FAILURE);    
    }
  conf->db_out = (ipcbuf_t *) conf->hdu_out->data_block;
  conf->rbufout_size = ipcbuf_get_bufsz(conf->db_out);
  log_add(conf->log_file, "INFO", 1,  "Output buffer block size is %"PRIu64".", conf->rbufout_size);
  
  if(conf->rbufout_size != (conf->bufout_size * conf->nrepeat_per_blk))  
    {
      log_add(conf->log_file, "ERR", 1,  "Buffer size mismatch, which happens at \"%s\", line [%d].", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      destroy_baseband2baseband(*conf);
      exit(EXIT_FAILURE);    
    }  
  hdrsz = ipcbuf_get_bufsz(conf->hdu_out->header_block);  
  if(hdrsz != DADA_DEFAULT_HEADER_SIZE)    // This number should match
    {
      log_add(conf->log_file, "ERR", 1,  "Buffer size mismatch, which happens at \"%s\", line [%d].", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      destroy_baseband2baseband(*conf);
      exit(EXIT_FAILURE);    
    }  
  if(dada_hdu_lock_write(conf->hdu_out) < 0)   // make ourselves the write client 
    {
      log_add(conf->log_file, "ERR", 1,  "Error locking HDU, which happens at \"%s\", line [%d].", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: Error locking HDU, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      destroy_baseband2baseband(*conf);
      exit(EXIT_FAILURE);
    }
  clock_gettime(CLOCK_REALTIME, &start);
  dada_cuda_dbregister(conf->hdu_out);  // registers the existing host memory range for use by CUDA
  clock_gettime(CLOCK_REALTIME, &stop);
  elapsed_time = (stop.tv_sec - start.tv_sec) + (stop.tv_nsec - start.tv_nsec)/1.0E9L;
  fprintf(stdout, "elapsed_time for dbregister of output ring buffer is %f\n", elapsed_time);
  fflush(stdout);
  
  if(conf->sod == 0)
    {
      if(ipcbuf_disable_sod(conf->db_out) < 0)
	{
	  log_add(conf->log_file, "ERR", 1,  "Can not write data before start, which happens at \"%s\", line [%d], has to abort.", __FILE__, __LINE__);
	  fprintf(stderr, "BASEBAND2BASEBAND_ERROR: Can not write data before start, which happens at \"%s\", line [%d], has to abort.\n", __FILE__, __LINE__);
	  
	  destroy_baseband2baseband(*conf);
	  fclose(conf->log_file);
	  exit(EXIT_FAILURE);
	}
    }
  
  return EXIT_SUCCESS;
}

int baseband2baseband(conf_t conf)
{
  /*
    The whole procedure for fold mode is :
    1. Unpack the data and reorder it from TFTFP to PFT order, prepare for the forward FFT;
    2. Forward FFT the PFT data to get finer channelzation and the data is in PFTF order after FFT;
    3. Swap the FFT output to put the frequency centre on the right place, drop frequency channel edge and band edge and put the data into PTF order, swap the data and put the centre frequency at bin 0 for each FFT block, prepare for inverse FFT;
    4. Inverse FFT the data to get PTFT order data;
    5. Transpose the data to get TFP data and scale it;    
  */
  uint64_t i, j, k;
  uint64_t hbufin_offset, dbufin_offset, bufrt1_offset, bufrt2_offset, hbufout_offset, dbufout_offset, dbufout_offset_monitor;
  dim3 gridsize_unpack, blocksize_unpack;
  dim3 gridsize_swap_select_transpose_swap, blocksize_swap_select_transpose_swap;
  dim3 gridsize_transpose_scale, blocksize_transpose_scale;
  dim3 gridsize_transpose_complex, blocksize_transpose_complex;
  dim3 gridsize_spectral_taccumulate, blocksize_spectral_taccumulate;
  uint64_t cbufsz;
  int first = 1;
  double time_res_blk, time_offset = 0;  
  double chan_width; 
  double time_res_stream;
  int eth_index;
  struct tm tm_stamp;
  char time_stamp[MSTR_LEN];
  double time_stamp_f;
  time_t time_stamp_i;
  int sock_udp, enable = 1;
  struct sockaddr_in sa_udp;
  socklen_t tolen = sizeof(sa_udp);
  
  gridsize_unpack                      = conf.gridsize_unpack;
  blocksize_unpack                     = conf.blocksize_unpack;
  gridsize_swap_select_transpose_swap  = conf.gridsize_swap_select_transpose_swap;   
  blocksize_swap_select_transpose_swap = conf.blocksize_swap_select_transpose_swap;  
  gridsize_transpose_scale             = conf.gridsize_transpose_scale;
  blocksize_transpose_scale            = conf.blocksize_transpose_scale;
  gridsize_transpose_complex           = conf.gridsize_transpose_complex;
  blocksize_transpose_complex          = conf.blocksize_transpose_complex;
  gridsize_spectral_taccumulate        = conf.gridsize_spectral_taccumulate;
  blocksize_spectral_taccumulate       = conf.blocksize_spectral_taccumulate;
    
  read_dada_header(&conf); 
  time_res_blk = conf.tsamp * conf.ndf_per_chunk_rbufin * NSAMP_DF / 1.0E6; // This has to be after read_register_header, in seconds
  if(conf.fits_flag == 1)
    {
      time_res_stream = conf.tsamp * conf.ndf_per_chunk_stream * NSAMP_DF / 1.0E6; // This has to be after read_register_header, in seconds
      strptime(conf.utc_start, DADA_TIMESTR, &tm_stamp);
      time_stamp_f = mktime(&tm_stamp) + conf.picoseconds / 1.0E12 + 0.5 * time_res_stream;
      chan_width = 1.0;
    }
  if(conf.sod == 1)
    register_dada_header(&conf); 
  
  /* Create socket */
  if(conf.fits_flag == 1)
    {
      /* For both multicast and unicast */
      if((sock_udp = socket(AF_INET, SOCK_DGRAM, IPPROTO_UDP)) == -1)
      	{
      	  fprintf(stderr, "BASEBAND2BASEBAND_ERROR: socket creation failed, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      	  log_add(conf.log_file, "ERR", 1,  "socket creation failed, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      	  
      	  destroy_baseband2baseband(conf);
      	  fclose(conf.log_file);
      	  exit(EXIT_FAILURE);
      	}
      memset((char *) &sa_udp, 0, sizeof(sa_udp));
      sa_udp.sin_family      = AF_INET;
      sa_udp.sin_port        = htons(conf.port);
      sa_udp.sin_addr.s_addr = inet_addr(conf.ip);
      setsockopt(sock_udp, SOL_SOCKET, SO_REUSEADDR, &enable, sizeof(enable));
    }
  
  /* Do the real job */  
  while(!ipcbuf_eod(conf.db_in))
    {
      conf.cbuf_in  = ipcbuf_get_next_read(conf.db_in, &cbufsz);
      conf.cbuf_out = ipcbuf_get_next_write(conf.db_out);
      
      /* Get scale of data */
      if(first)
      	{
      	  first = 0;
      	  offset_scale(conf);
      	}
      for(i = 0; i < conf.nrepeat_per_blk; i ++)
	{
	  for(j = 0; j < conf.nstream; j++)
	    {
	      hbufin_offset = (i * conf.nstream + j) * conf.hbufin_offset;// + i * conf.bufin_size;
	      dbufin_offset = j * conf.dbufin_offset; 
	      bufrt1_offset = j * conf.bufrt1_offset;
	      bufrt2_offset = j * conf.bufrt2_offset;

	      dbufout_offset = j * conf.dbufout_offset;
	      dbufout_offset_monitor = j * conf.dbufout_offset_monitor;
	      hbufout_offset = (i * conf.nstream + j) * conf.hbufout_offset;// + i * conf.bufout_size;
	      
	      CudaSafeCall(hipMemcpyAsync(&conf.dbuf_in[dbufin_offset], &conf.cbuf_in[hbufin_offset], conf.sbufin_size, hipMemcpyHostToDevice, conf.streams[j]));

	      /* Unpack raw data into hipfftComplex array */
	      unpack_kernel<<<gridsize_unpack, blocksize_unpack, 0, conf.streams[j]>>>(&conf.dbuf_in[dbufin_offset], &conf.buf_rt1[bufrt1_offset], conf.nsamp_in);
	      CudaSafeKernelLaunch();
	      
	      /* Do forward FFT */
	      CufftSafeCall(hipfftExecC2C(conf.fft_plans_forward[j], &conf.buf_rt1[bufrt1_offset], &conf.buf_rt1[bufrt1_offset], HIPFFT_FORWARD));

	      /* Prepare for inverse FFT */
	      swap_select_transpose_swap_kernel<<<gridsize_swap_select_transpose_swap, blocksize_swap_select_transpose_swap, 0, conf.streams[j]>>>(&conf.buf_rt1[bufrt1_offset], &conf.buf_rt2[bufrt2_offset], conf.nsamp_in, conf.nsamp_keep, conf.cufft_nx, conf.cufft_mod, conf.nchan_keep_chan);
	      CudaSafeKernelLaunch();
	      
	      /* Do inverse FFT */
	      CufftSafeCall(hipfftExecC2C(conf.fft_plans_backward[j], &conf.buf_rt2[bufrt2_offset], &conf.buf_rt2[bufrt2_offset], HIPFFT_BACKWARD));
	      
	      if(conf.fits_flag == 1)
		{
		  /* Tranpose from PTFT to PFT order */
		  transpose_complex_kernel<<<gridsize_transpose_complex, blocksize_transpose_complex, 0, conf.streams[j]>>>(&conf.buf_rt2[bufrt2_offset], conf.nsamp_keep, &conf.buf_rt1[bufrt1_offset]);
		  CudaSafeKernelLaunch();
		  
		  switch(blocksize_spectral_taccumulate.x)
		    {
		    case 1024:
		      spectral_taccumulate_fold_kernel
			<1024>
			<<<gridsize_spectral_taccumulate,
			blocksize_spectral_taccumulate,
			blocksize_spectral_taccumulate.x * NDATA_PER_SAMP_RT * NBYTE_SPECTRAL,
			conf.streams[j]>>>
			(&conf.buf_rt1[bufrt1_offset],
			 &conf.dbuf_out_monitor[dbufout_offset_monitor],
			 conf.nsamp_keep,
			 conf.nsamp_out,
			 conf.naccumulate_spectral);
		      break;
		      
		    case 512:
		      spectral_taccumulate_fold_kernel
			< 512>
			<<<gridsize_spectral_taccumulate,
			blocksize_spectral_taccumulate,
			blocksize_spectral_taccumulate.x * NDATA_PER_SAMP_RT * NBYTE_SPECTRAL,
			conf.streams[j]>>>
			(&conf.buf_rt1[bufrt1_offset],
			 &conf.dbuf_out_monitor[dbufout_offset_monitor],
			 conf.nsamp_keep,
			 conf.nsamp_out,
			 conf.naccumulate_spectral);
		      break;
		      
		    case 256:
		      spectral_taccumulate_fold_kernel
			< 256>
			<<<gridsize_spectral_taccumulate,
			blocksize_spectral_taccumulate,
			blocksize_spectral_taccumulate.x * NDATA_PER_SAMP_RT * NBYTE_SPECTRAL,
			conf.streams[j]>>>
			(&conf.buf_rt1[bufrt1_offset],
			 &conf.dbuf_out_monitor[dbufout_offset_monitor],
			 conf.nsamp_keep,
			 conf.nsamp_out,
			 conf.naccumulate_spectral);
		      break;
		      
		    case 128:
		      spectral_taccumulate_fold_kernel
			< 128>
			<<<gridsize_spectral_taccumulate,
			blocksize_spectral_taccumulate,
			blocksize_spectral_taccumulate.x * NDATA_PER_SAMP_RT * NBYTE_SPECTRAL,
			conf.streams[j]>>>
			(&conf.buf_rt1[bufrt1_offset],
			 &conf.dbuf_out_monitor[dbufout_offset_monitor],
			 conf.nsamp_keep,
			 conf.nsamp_out,
			 conf.naccumulate_spectral);
		      break;
		      
		    case  64:
		      spectral_taccumulate_fold_kernel
			<  64>
			<<<gridsize_spectral_taccumulate,
			blocksize_spectral_taccumulate,
			blocksize_spectral_taccumulate.x * NDATA_PER_SAMP_RT * NBYTE_SPECTRAL,
			conf.streams[j]>>>
			(&conf.buf_rt1[bufrt1_offset],
			 &conf.dbuf_out_monitor[dbufout_offset_monitor],
			 conf.nsamp_keep,
			 conf.nsamp_out,
			 conf.naccumulate_spectral);
		      break;
		      
		    case  32:
		      spectral_taccumulate_fold_kernel
			<  32>
			<<<gridsize_spectral_taccumulate,
			blocksize_spectral_taccumulate,
			blocksize_spectral_taccumulate.x * NDATA_PER_SAMP_RT * NBYTE_SPECTRAL,
			conf.streams[j]>>>
			(&conf.buf_rt1[bufrt1_offset],
			 &conf.dbuf_out_monitor[dbufout_offset_monitor],
			 conf.nsamp_keep,
			 conf.nsamp_out,
			 conf.naccumulate_spectral);
		      break;
		      
		    case  16:
		      spectral_taccumulate_fold_kernel
			<  16>		    
			<<<gridsize_spectral_taccumulate,
			blocksize_spectral_taccumulate,
			blocksize_spectral_taccumulate.x * NDATA_PER_SAMP_RT * NBYTE_SPECTRAL,
			conf.streams[j]>>>
			(&conf.buf_rt1[bufrt1_offset],
			 &conf.dbuf_out_monitor[dbufout_offset_monitor],
			 conf.nsamp_keep,
			 conf.nsamp_out,
			 conf.naccumulate_spectral);
		      break;
		      
		    case  8:
		      spectral_taccumulate_fold_kernel
			<   8>		    
			<<<gridsize_spectral_taccumulate,
			blocksize_spectral_taccumulate,
			blocksize_spectral_taccumulate.x * NDATA_PER_SAMP_RT * NBYTE_SPECTRAL,
			conf.streams[j]>>>
			(&conf.buf_rt1[bufrt1_offset],
			 &conf.dbuf_out_monitor[dbufout_offset_monitor],
			 conf.nsamp_keep,
			 conf.nsamp_out,
			 conf.naccumulate_spectral);
		      break;
		      
		    case  4:
		      spectral_taccumulate_fold_kernel
			<   4>		    		    
			<<<gridsize_spectral_taccumulate,
			blocksize_spectral_taccumulate,
			blocksize_spectral_taccumulate.x * NDATA_PER_SAMP_RT * NBYTE_SPECTRAL,
			conf.streams[j]>>>
			(&conf.buf_rt1[bufrt1_offset],
			 &conf.dbuf_out_monitor[dbufout_offset_monitor],
			 conf.nsamp_keep,
			 conf.nsamp_out,
			 conf.naccumulate_spectral);
		      break;
		      
		    case  2:
		      spectral_taccumulate_fold_kernel
			<   2>		    		    		    
			<<<gridsize_spectral_taccumulate,
			blocksize_spectral_taccumulate,
			blocksize_spectral_taccumulate.x * NDATA_PER_SAMP_RT * NBYTE_SPECTRAL,
			conf.streams[j]>>>
			(&conf.buf_rt1[bufrt1_offset],
			 &conf.dbuf_out_monitor[dbufout_offset_monitor],
			 conf.nsamp_keep,
			 conf.nsamp_out,
			 conf.naccumulate_spectral);
		      break;
		      
		    case  1:
		      spectral_taccumulate_fold_kernel
			<   1>		    		    		    
			<<<gridsize_spectral_taccumulate,
			blocksize_spectral_taccumulate,
			blocksize_spectral_taccumulate.x * NDATA_PER_SAMP_RT * NBYTE_SPECTRAL,
			conf.streams[j]>>>
			(&conf.buf_rt1[bufrt1_offset],
			 &conf.dbuf_out_monitor[dbufout_offset_monitor],
			 conf.nsamp_keep,
			 conf.nsamp_out,
			 conf.naccumulate_spectral);
		      break;
		    }
		  CudaSafeKernelLaunch();

		  /* Setup ethernet packets */
		  time_stamp_i = (time_t)time_stamp_f;
		  strftime(time_stamp, FITS_TIME_STAMP_LEN, FITS_TIMESTR, gmtime(&time_stamp_i)); 
		  sprintf(time_stamp, "%s.%04dUTC ", time_stamp, (int)((time_stamp_f - time_stamp_i) * 1E4 + 0.5));
		  for(k = 0; k < NDATA_PER_SAMP_FULL; k++)
		    {		  
		      eth_index = i * conf.nstream * NDATA_PER_SAMP_FULL + j * NDATA_PER_SAMP_FULL + k;
		      
		      strncpy(conf.fits[eth_index].time_stamp, time_stamp, FITS_TIME_STAMP_LEN);		  
		      conf.fits[eth_index].tsamp = time_res_stream;
		      conf.fits[eth_index].nchan = conf.nchan;
		      conf.fits[eth_index].chan_width = chan_width;
		      conf.fits[eth_index].pol_type = conf.pol_type;
		      conf.fits[eth_index].pol_index = k;
		      conf.fits[eth_index].beam_index  = conf.beam_index;
		      conf.fits[eth_index].center_freq = conf.center_freq;
		      conf.fits[eth_index].nchunk = 1;
		      conf.fits[eth_index].chunk_index = 0;

		      if(k < conf.pol_type)
			{
			  if(conf.pol_type == 2)
			    {
			      CudaSafeCall(hipMemcpyAsync(conf.fits[eth_index].data,
							   &conf.dbuf_out_monitor[dbufout_offset_monitor +
									   conf.nchan  *
									   (NDATA_PER_SAMP_FULL + k)],
							   conf.dtsz_network,
							   hipMemcpyDeviceToHost,
							   conf.streams[j]));
			    }
			  else
			    CudaSafeCall(hipMemcpyAsync(conf.fits[eth_index].data,
							 &conf.dbuf_out_monitor[dbufout_offset_monitor +
										k * conf.nchan],
							 conf.dtsz_network,
							 hipMemcpyDeviceToHost,
							 conf.streams[j]));
			}
		    }
		  time_stamp_f += time_res_stream;
		}
	      
	      /* Get baseband output */
	      transpose_scale_kernel<<<gridsize_transpose_scale, blocksize_transpose_scale, 0, conf.streams[j]>>>(&conf.buf_rt2[bufrt2_offset], &conf.dbuf_out[dbufout_offset], conf.nchan_keep_chan, conf.nchan, conf.nsamp_keep, conf.offset_scale_d);
	      CudaSafeKernelLaunch();
	      
	      /* Copy the final output to host */
	      CudaSafeCall(hipMemcpyAsync(&conf.cbuf_out[hbufout_offset], &conf.dbuf_out[dbufout_offset], conf.sbufout_size, hipMemcpyDeviceToHost, conf.streams[j]));
	    }
	}
      CudaSynchronizeCall(); // Sync here is for multiple streams

      /* Send all packets from the previous buffer block with one go */
      if(conf.fits_flag == 1)
	{
	  for(i = 0; i < conf.neth_per_blk; i++)
	    {
	      if(sendto(sock_udp, (void *)&conf.fits[i], conf.pktsz_network, 0, (struct sockaddr *)&sa_udp, tolen) == -1)
		{
		  fprintf(stderr, "BASEBAND2BASEBAND_ERROR: sendto() failed, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
		  log_add(conf.log_file, "ERR", 1,  "sendto() failed, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
		  
		  destroy_baseband2baseband(conf);
		  fclose(conf.log_file);
		  exit(EXIT_FAILURE);
		}
	      usleep(1);
	    }
	}
      
      /* Close current buffer */
      ipcbuf_mark_filled(conf.db_out, (uint64_t)(cbufsz * conf.scale_dtsz));
      //ipcbuf_mark_filled(conf.db_out, conf.bufout_size * conf.nrepeat_per_blk);
      //ipcbuf_mark_filled(conf.db_out, conf.rbufout_size);
      fprintf(stdout, "%"PRIu64"\n", conf.rbufout_size);
      fflush(stdout);
      ipcbuf_mark_cleared(conf.db_in);

      time_offset += time_res_blk;
      fprintf(stdout, "BASEBAND2BASEBAND, finished %f seconds data\n", time_offset);
      log_add(conf.log_file, "INFO", 1,  "finished %f seconds data", time_offset);
      fflush(stdout);
    }
  return EXIT_SUCCESS;
}

int destroy_baseband2baseband(conf_t conf)
{
  int i;
  
  for (i = 0; i < conf.nstream; i++)
    {
      if(conf.fft_plans_forward[i])
	CufftSafeCall(hipfftDestroy(conf.fft_plans_forward[i]));
      if(conf.fft_plans_backward[i])
	CufftSafeCall(hipfftDestroy(conf.fft_plans_backward[i]));
    }

  if(conf.fft_plans_forward)
    free(conf.fft_plans_forward);
  if(conf.fft_plans_backward)
    free(conf.fft_plans_backward);

  if(conf.dbuf_in)
    hipFree(conf.dbuf_in);
  if(conf.buf_rt1)
    hipFree(conf.buf_rt1);
  if(conf.buf_rt2)
    hipFree(conf.buf_rt2);

  if(conf.dbuf_out)
    hipFree(conf.dbuf_out);
  if(conf.dbuf_out_monitor)
    hipFree(conf.dbuf_out_monitor);
  if(conf.offset_scale_h)
    hipHostFree(conf.offset_scale_h);
  if(conf.offset_scale_d)
    hipFree(conf.offset_scale_d);

  if(conf.db_out)
    {
      dada_cuda_dbunregister(conf.hdu_out);  
      dada_hdu_unlock_write(conf.hdu_out);
      dada_hdu_destroy(conf.hdu_out);
    }

  if(conf.db_in)
    {
      dada_cuda_dbunregister(conf.hdu_in);  
      dada_hdu_unlock_read(conf.hdu_in);
      dada_hdu_destroy(conf.hdu_in);
    }
  
  for(i = 0; i < conf.nstream; i++)
    {
      if(conf.streams[i])
	CudaSafeCall(hipStreamDestroy(conf.streams[i]));
    }
  if(conf.streams)
    free(conf.streams);    

  CudaSafeCall(hipProfilerStop());
  CudaSafeCall(hipDeviceReset());
  
  return EXIT_SUCCESS;
}

int offset_scale(conf_t conf)
{
  /*
    The procedure for fold mode is:
    1. Get PTFT data as we did at process;
    2. Pad the data;
    3. Add the padded data in time;
    4. Get the mean of the added data;
    5. Get the scale with the mean;

    The procedure for search mode is:
    1. Get PTF data as we did at process;
    2. Add the data in frequency to get NCHAN_SEARCH channels, detect the added data and pad it;
    3. Add the padded data in time;    
    4. Get the mean of the added data;
    5. Get the scale with the mean;
  */
  size_t i, j;
  dim3 gridsize_unpack, blocksize_unpack;
  dim3 gridsize_swap_select_transpose_swap, blocksize_swap_select_transpose_swap;
  dim3 gridsize_scale, blocksize_scale; 
  dim3 gridsize_transpose_pad, blocksize_transpose_pad;
  dim3 gridsize_taccumulate, blocksize_taccumulate;
  
  size_t hbufin_offset, dbufin_offset, bufrt1_offset, bufrt2_offset;
    
  gridsize_unpack                      = conf.gridsize_unpack;
  blocksize_unpack                     = conf.blocksize_unpack;
  gridsize_swap_select_transpose_swap  = conf.gridsize_swap_select_transpose_swap;   
  blocksize_swap_select_transpose_swap = conf.blocksize_swap_select_transpose_swap; 
  gridsize_transpose_pad               = conf.gridsize_transpose_pad;
  blocksize_transpose_pad              = conf.blocksize_transpose_pad;
  	         	               	
  gridsize_scale        = conf.gridsize_scale;	       
  blocksize_scale       = conf.blocksize_scale;
  gridsize_taccumulate  = conf.gridsize_taccumulate;
  blocksize_taccumulate = conf.blocksize_taccumulate;

  for(i = 0; i < conf.nrepeat_per_blk; i ++)
    {
      for (j = 0; j < conf.nstream; j++)
	{
	  hbufin_offset = (i * conf.nstream + j) * conf.hbufin_offset;
	  dbufin_offset = j * conf.dbufin_offset; 
	  bufrt1_offset = j * conf.bufrt1_offset;
	  bufrt2_offset = j * conf.bufrt2_offset;
	  
	  /* Copy data into device */
	  CudaSafeCall(hipMemcpyAsync(&conf.dbuf_in[dbufin_offset], &conf.cbuf_in[hbufin_offset], conf.sbufin_size, hipMemcpyHostToDevice, conf.streams[j]));

	  /* Unpack raw data into hipfftComplex array */
	  unpack_kernel<<<gridsize_unpack, blocksize_unpack, 0, conf.streams[j]>>>(&conf.dbuf_in[dbufin_offset], &conf.buf_rt1[bufrt1_offset], conf.nsamp_in);
	  CudaSafeKernelLaunch();
	  
	  /* Do forward FFT */
	  CufftSafeCall(hipfftExecC2C(conf.fft_plans_forward[j], &conf.buf_rt1[bufrt1_offset], &conf.buf_rt1[bufrt1_offset], HIPFFT_FORWARD));

	  /* Prepare for inverse FFT */
	  swap_select_transpose_swap_kernel<<<gridsize_swap_select_transpose_swap, blocksize_swap_select_transpose_swap, 0, conf.streams[j]>>>(&conf.buf_rt1[bufrt1_offset], &conf.buf_rt2[bufrt2_offset], conf.nsamp_in, conf.nsamp_keep, conf.cufft_nx, conf.cufft_mod, conf.nchan_keep_chan);
	  CudaSafeKernelLaunch();
	  
	  /* Do inverse FFT */
	  CufftSafeCall(hipfftExecC2C(conf.fft_plans_backward[j], &conf.buf_rt2[bufrt2_offset], &conf.buf_rt2[bufrt2_offset], HIPFFT_BACKWARD));
	  
	  /* Transpose the data from PTFT to FTP for later calculation */
	  transpose_pad_kernel<<<gridsize_transpose_pad, blocksize_transpose_pad, 0, conf.streams[j]>>>(&conf.buf_rt2[bufrt2_offset], conf.nsamp_keep, &conf.buf_rt1[bufrt1_offset]);
	  CudaSafeKernelLaunch();
	  
	  switch (blocksize_taccumulate.x)
	    {
	    case 1024:
	      reduce10_kernel<1024><<<gridsize_taccumulate, blocksize_taccumulate, blocksize_taccumulate.x * NBYTE_CUFFT_COMPLEX, conf.streams[j]>>>(&conf.buf_rt1[bufrt1_offset], &conf.offset_scale_d[j*conf.nchan], conf.naccumulate, conf.ndim_scale);
	      break;
	    case 512:
	      reduce10_kernel< 512><<<gridsize_taccumulate, blocksize_taccumulate, blocksize_taccumulate.x * NBYTE_CUFFT_COMPLEX, conf.streams[j]>>>(&conf.buf_rt1[bufrt1_offset], &conf.offset_scale_d[j*conf.nchan], conf.naccumulate, conf.ndim_scale);
	      break;
	    case 256:
	      reduce10_kernel< 256><<<gridsize_taccumulate, blocksize_taccumulate, blocksize_taccumulate.x * NBYTE_CUFFT_COMPLEX, conf.streams[j]>>>(&conf.buf_rt1[bufrt1_offset], &conf.offset_scale_d[j*conf.nchan], conf.naccumulate, conf.ndim_scale);
	      break;
	    case 128:
	      reduce10_kernel< 128><<<gridsize_taccumulate, blocksize_taccumulate, blocksize_taccumulate.x * NBYTE_CUFFT_COMPLEX, conf.streams[j]>>>(&conf.buf_rt1[bufrt1_offset], &conf.offset_scale_d[j*conf.nchan], conf.naccumulate, conf.ndim_scale);
	      break;
	    case 64:
	      reduce10_kernel<  64><<<gridsize_taccumulate, blocksize_taccumulate, blocksize_taccumulate.x * NBYTE_CUFFT_COMPLEX, conf.streams[j]>>>(&conf.buf_rt1[bufrt1_offset], &conf.offset_scale_d[j*conf.nchan], conf.naccumulate, conf.ndim_scale);
	      break;
	    case 32:
	      reduce10_kernel<  32><<<gridsize_taccumulate, blocksize_taccumulate, blocksize_taccumulate.x * NBYTE_CUFFT_COMPLEX, conf.streams[j]>>>(&conf.buf_rt1[bufrt1_offset], &conf.offset_scale_d[j*conf.nchan], conf.naccumulate, conf.ndim_scale);
	      break;
	    case 16:
	      reduce10_kernel<  16><<<gridsize_taccumulate, blocksize_taccumulate, blocksize_taccumulate.x * NBYTE_CUFFT_COMPLEX, conf.streams[j]>>>(&conf.buf_rt1[bufrt1_offset], &conf.offset_scale_d[j*conf.nchan], conf.naccumulate, conf.ndim_scale);
	      break;
	    case 8:
	      reduce10_kernel<   8><<<gridsize_taccumulate, blocksize_taccumulate, blocksize_taccumulate.x * NBYTE_CUFFT_COMPLEX, conf.streams[j]>>>(&conf.buf_rt1[bufrt1_offset], &conf.offset_scale_d[j*conf.nchan], conf.naccumulate, conf.ndim_scale);
	      break;
	    case 4:
	      reduce10_kernel<   4><<<gridsize_taccumulate, blocksize_taccumulate, blocksize_taccumulate.x * NBYTE_CUFFT_COMPLEX, conf.streams[j]>>>(&conf.buf_rt1[bufrt1_offset], &conf.offset_scale_d[j*conf.nchan], conf.naccumulate, conf.ndim_scale);
	      break;
	    case 2:
	      reduce10_kernel<   2><<<gridsize_taccumulate, blocksize_taccumulate, blocksize_taccumulate.x * NBYTE_CUFFT_COMPLEX, conf.streams[j]>>>(&conf.buf_rt1[bufrt1_offset], &conf.offset_scale_d[j*conf.nchan], conf.naccumulate, conf.ndim_scale);
	      break;
	    case 1:
	      reduce10_kernel<   1><<<gridsize_taccumulate, blocksize_taccumulate, blocksize_taccumulate.x * NBYTE_CUFFT_COMPLEX, conf.streams[j]>>>(&conf.buf_rt1[bufrt1_offset], &conf.offset_scale_d[j*conf.nchan], conf.naccumulate, conf.ndim_scale);
	      break;
	    }
	  CudaSafeKernelLaunch();
	}
    }
  CudaSynchronizeCall();
  
  /* Get the scale of each chanel */
  scale3_kernel<<<gridsize_scale, blocksize_scale>>>(conf.offset_scale_d, conf.nchan, conf.nstream, SCL_NSIG, SCL_INT8);
  CudaSafeKernelLaunch();
  CudaSynchronizeCall();
  
  CudaSafeCall(hipMemcpy(conf.offset_scale_h, conf.offset_scale_d, NBYTE_CUFFT_COMPLEX * conf.nchan, hipMemcpyDeviceToHost));
  CudaSynchronizeCall();
  
  /* Record scale into file */
  for (i = 0; i< conf.nchan; i++)
    {
      fprintf(stdout, "%E\t%E\n", conf.offset_scale_h[i].x, conf.offset_scale_h[i].y);
      fflush(stdout);
    }
  char fname[MSTR_LEN];
  FILE *fp=NULL;
  sprintf(fname, "%s/%s_baseband2baseband.scl", conf.dir, conf.utc_start);
  fp = fopen(fname, "w");
  if(fp == NULL)
    {
      log_add(conf.log_file, "ERR", 1,  "Can not open scale file, which happens at \"%s\", line [%d].", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: Can not open scale file, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      destroy_baseband2baseband(conf);
      exit(EXIT_FAILURE);
    }
  for (i = 0; i< conf.nchan; i++)
    fprintf(fp, "%E\t%E\n", conf.offset_scale_h[i].x, conf.offset_scale_h[i].y);

  fclose(fp);
  return EXIT_SUCCESS;
}


int examine_record_arguments(conf_t conf, char **argv, int argc)
{
  int i;
  char command_line[MSTR_LEN] = {'\0'};
  
  /* Log the input */
  strcpy(command_line, argv[0]);
  for(i = 1; i < argc; i++)
    {
      strcat(command_line, " ");
      strcat(command_line, argv[i]);
    }
  log_add(conf.log_file, "INFO", 1,  "The command line is \"%s\"", command_line);
  log_add(conf.log_file, "INFO", 1,  "The input ring buffer key is %x", conf.key_in); 
  log_add(conf.log_file, "INFO", 1,  "The output ring buffer key is %x", conf.key_out);

  if(conf.ndf_per_chunk_rbufin == 0)
    {
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: ndf_per_chunk_rbuf shoule be a positive number, but it is %"PRIu64", which happens at \"%s\", line [%d], has to abort\n", conf.ndf_per_chunk_rbufin, __FILE__, __LINE__);
      log_add(conf.log_file, "ERR", 1,  "ndf_per_chunk_rbuf shoule be a positive number, but it is %"PRIu64", which happens at \"%s\", line [%d], has to abort", conf.ndf_per_chunk_rbufin, __FILE__, __LINE__);
      
      log_close(conf.log_file);
      exit(EXIT_FAILURE);
    }
  log_add(conf.log_file, "INFO", 1,  "Each input ring buffer block has %"PRIu64" packets per frequency chunk", conf.ndf_per_chunk_rbufin); 

  if(conf.nstream <= 0)
    {
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: nstream shoule be a positive number, but it is %d, which happens at \"%s\", line [%d], has to abort\n", conf.nstream, __FILE__, __LINE__);
      log_add(conf.log_file, "ERR", 1,  "nstream shoule be a positive number, but it is %d, which happens at \"%s\", line [%d], has to abort", conf.nstream, __FILE__, __LINE__);
      
      log_close(conf.log_file);
      exit(EXIT_FAILURE);
    }
  log_add(conf.log_file, "INFO", 1,  "%d streams run on GPU", conf.nstream);
  
  if(conf.ndf_per_chunk_stream == 0)
    {
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: ndf_per_chunk_stream shoule be a positive number, but it is %d, which happens at \"%s\", line [%d], has to abort\n", conf.ndf_per_chunk_stream, __FILE__, __LINE__);
      log_add(conf.log_file, "ERR", 1,  "ndf_per_chunk_stream shoule be a positive number, but it is %d, which happens at \"%s\", line [%d], has to abort", conf.ndf_per_chunk_stream, __FILE__, __LINE__);
      
      log_close(conf.log_file);
      exit(EXIT_FAILURE);
    }
  log_add(conf.log_file, "INFO", 1,  "Each stream process %d packets per frequency chunk", conf.ndf_per_chunk_stream);
  log_add(conf.log_file, "INFO", 1,  "The runtime information is %s", conf.dir);  // Checked already
  
  if(conf.nchunk<=0 || conf.nchunk>NCHUNK_FULL_BEAM)    
    {
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: nchunk shoule be in (0 %d], but it is %d, which happens at \"%s\", line [%d], has to abort\n", NCHUNK_FULL_BEAM, conf.nchunk, __FILE__, __LINE__);
      log_add(conf.log_file, "ERR", 1,  "nchunk shoule be in (0 %d], but it is %d, which happens at \"%s\", line [%d], has to abort", NCHUNK_FULL_BEAM, conf.nchunk, __FILE__, __LINE__);
      
      log_close(conf.log_file);
      exit(EXIT_FAILURE);
    }  
  log_add(conf.log_file, "INFO", 1,  "%d chunks of input data", conf.nchunk);

  if(conf.cufft_nx<=0)    
    {
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: cufft_nx shoule be a positive number, but it is %d, which happens at \"%s\", line [%d], has to abort\n", conf.cufft_nx, __FILE__, __LINE__);
      log_add(conf.log_file, "ERR", 1,  "cufft_nx shoule be a positive number, but it is %d, which happens at \"%s\", line [%d], has to abort", conf.cufft_nx, __FILE__, __LINE__);
      
      log_close(conf.log_file);
      exit(EXIT_FAILURE);
    }
  log_add(conf.log_file, "INFO", 1,  "We use %d points FFT", conf.cufft_nx);
  
  if(conf.sod == 1)
    log_add(conf.log_file, "INFO", 1,  "The baseband data is enabled at the beginning");
  else if(conf.sod == 0)
    log_add(conf.log_file, "INFO", 1,  "The baseband data is NOT enabled at the beginning");
  else
    {      
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: The SOD is not set, which happens at \"%s\", line [%d], has to abort\n", __FILE__, __LINE__);
      log_add(conf.log_file, "ERR", 1,  "The SOD is not set, which happens at \"%s\", line [%d], has to abort", __FILE__, __LINE__);
      
      log_close(conf.log_file);
      exit(EXIT_FAILURE);
    }
  
  if(conf.fits_flag == 1)
    {      
      if(!((conf.pol_type == 1) || (conf.pol_type == 2) || (conf.pol_type == 4)))
	{
	  fprintf(stderr, "BASEBAND2BASEBAND_ERROR: pol_type should be 1, 2 or 4, but it is %d, which happens at \"%s\", line [%d], has to abort\n", conf.pol_type, __FILE__, __LINE__);
	  log_add(conf.log_file, "ERR", 1,  "pol_type should be 1, 2 or 4, but it is %d, which happens at \"%s\", line [%d], has to abort", conf.pol_type, __FILE__, __LINE__);
      
	  log_close(conf.log_file);
	  exit(EXIT_FAILURE);
	}
      else
	log_add(conf.log_file, "INFO", 1,  "pol_type is %d", conf.pol_type);
            
      if(conf.port == -1)
	{
	  fprintf(stderr, "BASEBAND2BASEBAND_ERROR: port shoule be a positive number, but it is %d, which happens at \"%s\", line [%d], has to abort\n", conf.port, __FILE__, __LINE__);
	  log_add(conf.log_file, "ERR", 1,  "port shoule be a positive number, but it is %d, which happens at \"%s\", line [%d], has to abort", conf.port, __FILE__, __LINE__);
	  
	  log_close(conf.log_file);
	  exit(EXIT_FAILURE);
	}
      
      if(strstr(conf.ip, "unset"))
	{
	  fprintf(stderr, "BASEBAND2BASEBAND_ERROR: ip is unset, which happens at \"%s\", line [%d], has to abort\n", __FILE__, __LINE__);
	  log_add(conf.log_file, "ERR", 1,  "ip is unset, which happens at \"%s\", line [%d], has to abort", __FILE__, __LINE__);
	  
	  log_close(conf.log_file);
	  exit(EXIT_FAILURE);
	}
      log_add(conf.log_file, "INFO", 1,  "We will send data to %s:%d", conf.ip, conf.port); 
    }
  else
    log_add(conf.log_file, "INFO", 1,  "We will not send data to FITSwriter interface");
  
  return EXIT_SUCCESS;
}

int read_dada_header(conf_t *conf)
{  
  uint64_t hdrsz;
  
  conf->hdrbuf_in  = ipcbuf_get_next_read(conf->hdu_in->header_block, &hdrsz);  
  if (!conf->hdrbuf_in)
    {
      log_add(conf->log_file, "ERR", 1,  "Error getting header_buf, which happens at \"%s\", line [%d].", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: Error getting header_buf, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      
      destroy_baseband2baseband(*conf);
      fclose(conf->log_file);
      exit(EXIT_FAILURE);
    }
  if(hdrsz != DADA_DEFAULT_HEADER_SIZE)
    {
      log_add(conf->log_file, "ERR", 1,  "Header size mismatch, which happens at \"%s\", line [%d].", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: Header size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);

      destroy_baseband2baseband(*conf);
      fclose(conf->log_file);
      exit(EXIT_FAILURE);
    }
  
  if (ascii_header_get(conf->hdrbuf_in, "FILE_SIZE", "%"SCNu64"", &conf->file_size_in) < 0)  
    {
      log_add(conf->log_file, "ERR", 1,  "Error getting FILE_SIZE, which happens at \"%s\", line [%d].", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: Error getting FILE_SIZE, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);

      destroy_baseband2baseband(*conf);
      fclose(conf->log_file);
      exit(EXIT_FAILURE);
    }   
  log_add(conf->log_file, "INFO", 1,  "FILE_SIZE from DADA header is %"PRIu64"", conf->file_size_in);
  
  if (ascii_header_get(conf->hdrbuf_in, "BYTES_PER_SECOND", "%"SCNu64"", &conf->bytes_per_second_in) < 0)  
    {
      log_add(conf->log_file, "ERR", 1,  "Error getting BYTES_PER_SECOND, which happens at \"%s\", line [%d].", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: Error getting BYTES_PER_SECOND, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);

      destroy_baseband2baseband(*conf);
      fclose(conf->log_file);
      exit(EXIT_FAILURE);
    }
  log_add(conf->log_file, "INFO", 1,  "BYTES_PER_SECOND from DADA header is %"PRIu64"", conf->bytes_per_second_in);
  
  if (ascii_header_get(conf->hdrbuf_in, "TSAMP", "%lf", &conf->tsamp) < 0)  
    {
      log_add(conf->log_file, "ERR", 1,  "Error getting TSAMP, which happens at \"%s\", line [%d].", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: Error getting TSAMP, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);

      destroy_baseband2baseband(*conf);
      fclose(conf->log_file);
      exit(EXIT_FAILURE);
    }
  log_add(conf->log_file, "INFO", 1,  "TSAMP from DADA header is %f", conf->tsamp);
  
  /* Get utc_start from hdrin */
  if (ascii_header_get(conf->hdrbuf_in, "UTC_START", "%s", conf->utc_start) < 0)  
    {
      log_add(conf->log_file, "ERR", 1,  "Error getting UTC_START, which happens at \"%s\", line [%d].", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: Error getting UTC_START, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);      
      destroy_baseband2baseband(*conf);
      fclose(conf->log_file);
      exit(EXIT_FAILURE);
    }
  log_add(conf->log_file, "INFO", 1,  "UTC_START from DADA header is %s", conf->utc_start);
    
  if(ascii_header_get(conf->hdrbuf_in, "PICOSECONDS", "%"SCNu64"", &(conf->picoseconds)) < 0)
    {
      log_add(conf->log_file, "ERR", 1,  "Error getting PICOSECONDS, which happens at \"%s\", line [%d], has to abort", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: Error getting PICOSECONDS, which happens at \"%s\", line [%d], has to abort.\n", __FILE__, __LINE__);

      destroy_baseband2baseband(*conf);
      log_close(conf->log_file);
      exit(EXIT_FAILURE);
    }
  log_add(conf->log_file, "INFO", 1,  "PICOSECONDS from DADA header is %"PRIu64"", conf->picoseconds);
  
  if (ascii_header_get(conf->hdrbuf_in, "RECEIVER", "%d", &conf->beam_index) < 0)  
    {
      log_add(conf->log_file, "ERR", 1,  "Error getting RECEIVER, which happens at \"%s\", line [%d].", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2SPECTRAL_ERROR: Error getting RECEIVER, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);

      destroy_baseband2baseband(*conf);
      fclose(conf->log_file);
      exit(EXIT_FAILURE);
    }
  log_add(conf->log_file, "INFO", 1,  "RECEIVER from DADA header is %d", conf->beam_index);
  
  if(ascii_header_get(conf->hdrbuf_in, "FREQ", "%lf", &(conf->center_freq)) < 0)
    {
      log_add(conf->log_file, "ERR", 1,  "Error egtting FREQ, which happens at \"%s\", line [%d], has to abort", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2SPECTRAL_ERROR: Error getting FREQ, which happens at \"%s\", line [%d], has to abort.\n", __FILE__, __LINE__);

      destroy_baseband2baseband(*conf);
      log_close(conf->log_file);
      exit(EXIT_FAILURE);
    }
  log_add(conf->log_file, "INFO", 1,  "FREQ from DADA header is %f", conf->center_freq);
  
  if(ipcbuf_mark_cleared (conf->hdu_in->header_block))  // We are the only one reader, so that we can clear it after read;
    {
      log_add(conf->log_file, "ERR", 1,  "Error header_clear, which happens at \"%s\", line [%d].", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: Error header_clear, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);

      destroy_baseband2baseband(*conf);
      fclose(conf->log_file);
      exit(EXIT_FAILURE);
    }
  return EXIT_SUCCESS;
}

int register_dada_header(conf_t *conf)
{
  char *hdrbuf_out = NULL;
  uint64_t file_size, bytes_per_second;
  
  hdrbuf_out = ipcbuf_get_next_write(conf->hdu_out->header_block);
  if (!hdrbuf_out)
    {
      log_add(conf->log_file, "ERR", 1,  "Error getting header_buf, which happens at \"%s\", line [%d].", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: Error getting header_buf, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);

      destroy_baseband2baseband(*conf);
      fclose(conf->log_file);
      exit(EXIT_FAILURE);
    }  
  memcpy(hdrbuf_out, conf->hdrbuf_in, DADA_DEFAULT_HEADER_SIZE); // Pass the header
  
  file_size = (uint64_t)(conf->file_size_in * conf->scale_dtsz);
  bytes_per_second = (uint64_t)(conf->bytes_per_second_in * conf->scale_dtsz);
  
  if (ascii_header_set(hdrbuf_out, "TSAMP", "%f", conf->tsamp * OVER_SAMP_RATE) < 0)  
    {
      log_add(conf->log_file, "ERR", 1,  "Error setting TSAMP, which happens at \"%s\", line [%d].", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: Error setting TSAMP, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);

      destroy_baseband2baseband(*conf);
      fclose(conf->log_file);
      exit(EXIT_FAILURE);
    }
  log_add(conf->log_file, "INFO", 1,  "TSAMP to DADA header is %f", conf->tsamp * OVER_SAMP_RATE);

  if (ascii_header_set(hdrbuf_out, "NBIT", "%d", NBIT_FOLD) < 0)  
    {
      log_add(conf->log_file, "ERR", 1,  "Can not connect to hdu, which happens at \"%s\", line [%d].", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: Error setting NBIT, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);

      destroy_baseband2baseband(*conf);
      fclose(conf->log_file);
      exit(EXIT_FAILURE);
    }
  log_add(conf->log_file, "INFO", 1,  "NBIT to DADA header is %d", NBIT_BASEBAND);
  
  if (ascii_header_set(hdrbuf_out, "FILE_SIZE", "%"PRIu64"", file_size) < 0)  
    {
      log_add(conf->log_file, "ERR", 1,  "Can not connect to hdu, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: BASEBAND2BASEBAND_ERROR:\tError setting FILE_SIZE, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);

      destroy_baseband2baseband(*conf);
      fclose(conf->log_file);
      exit(EXIT_FAILURE);
    }
  log_add(conf->log_file, "INFO", 1,  "FILE_SIZE to DADA header is %"PRIu64"", file_size);
  
  if (ascii_header_set(hdrbuf_out, "BYTES_PER_SECOND", "%"PRIu64"", bytes_per_second) < 0)  
    {
      log_add(conf->log_file, "ERR", 1,  "Error setting BYTES_PER_SECOND, which happens at \"%s\", line [%d].", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: Error setting BYTES_PER_SECOND, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      
      destroy_baseband2baseband(*conf);
      fclose(conf->log_file);
      exit(EXIT_FAILURE);
    }
  log_add(conf->log_file, "INFO", 1,  "BYTES_PER_SECOND to DADA header is %"PRIu64"", bytes_per_second);
  
  /* donot set header parameters anymore */
  if (ipcbuf_mark_filled (conf->hdu_out->header_block, DADA_DEFAULT_HEADER_SIZE) < 0)
    {
      log_add(conf->log_file, "ERR", 1,  "Error header_fill, which happens at \"%s\", line [%d].", __FILE__, __LINE__);
      fprintf(stderr, "BASEBAND2BASEBAND_ERROR: Error header_fill, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);

      destroy_baseband2baseband(*conf);
      fclose(conf->log_file);
      exit(EXIT_FAILURE);
    }

  return EXIT_SUCCESS;
}
