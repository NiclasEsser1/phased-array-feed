#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

#include <time.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>
#include <math.h>

#include "multilog.h"
#include "baseband2baseband.cuh"
#include "cudautil.cuh"
#include "kernel.cuh"

extern multilog_t *runtime_log;

int init_baseband2baseband(conf_t *conf)
{
  CudaSafeCall(hipSetDevice(conf->device_id));
  
  int i;
  int iembed1, istride1, idist1, oembed1, ostride1, odist1, batch1, nx1;
  int iembed2, istride2, idist2, oembed2, ostride2, odist2, batch2, nx2;
  ipcbuf_t *db = NULL;
  
  /* Prepare buffer, stream and fft plan for process */
  conf->sclndim = conf->rbufin_ndf * NSAMP_DF * NPOL_SAMP * NDIM_POL; // Only works when two polarisations has similar power level
  conf->nsamp1       = conf->stream_ndf * NCHK_CAPTURE * NCHAN_CHK * NSAMP_DF;
  conf->npol1        = conf->nsamp1 * NPOL_SAMP;
  conf->ndata1       = conf->npol1  * NDIM_POL;
		     
  conf->nsamp2       = conf->nsamp1 * OSAMP_RATEI / NCHAN_RATEI;
  conf->npol2        = conf->nsamp2 * NPOL_SAMP;
  conf->ndata2       = conf->npol2  * NDIM_POL;

  nx1        = CUFFT_NX1;
  batch1     = conf->npol1 / CUFFT_NX1;
  
  iembed1    = nx1;
  istride1   = 1;
  idist1     = nx1;
  
  oembed1    = nx1;
  ostride1   = 1;
  odist1     = nx1;
  
  nx2        = CUFFT_NX2;
  batch2     = conf->npol2 / CUFFT_NX2;
  
  iembed2    = nx2;
  istride2   = 1;
  idist2     = nx2;
  
  oembed2    = nx2;
  ostride2   = 1;
  odist2     = nx2;

  conf->streams = (hipStream_t *)malloc(conf->nstream * sizeof(hipStream_t));
  conf->fft_plans1 = (hipfftHandle *)malloc(conf->nstream * sizeof(hipfftHandle));
  conf->fft_plans2 = (hipfftHandle *)malloc(conf->nstream * sizeof(hipfftHandle));
  for(i = 0; i < conf->nstream; i ++)
    {
      CudaSafeCall(hipStreamCreate(&conf->streams[i]));
      CufftSafeCall(hipfftPlanMany(&conf->fft_plans1[i], CUFFT_RANK1, &nx1, &iembed1, istride1, idist1, &oembed1, ostride1, odist1, HIPFFT_C2C, batch1));
      CufftSafeCall(hipfftPlanMany(&conf->fft_plans2[i], CUFFT_RANK2, &nx2, &iembed2, istride2, idist2, &oembed2, ostride2, odist2, HIPFFT_C2C, batch2));
      
      CufftSafeCall(hipfftSetStream(conf->fft_plans1[i], conf->streams[i]));
      CufftSafeCall(hipfftSetStream(conf->fft_plans2[i], conf->streams[i]));
    }
  
  conf->sbufin_size    = conf->ndata1 * NBYTE_IN;
  conf->sbufout_size   = conf->ndata2 * NBYTE_OUT;
  
  conf->bufin_size     = conf->nstream * conf->sbufin_size;
  conf->bufout_size    = conf->nstream * conf->sbufout_size;
  
  conf->sbufrt1_size = conf->npol1 * sizeof(hipfftComplex);
  conf->sbufrt2_size = conf->npol2 * sizeof(hipfftComplex);
  conf->bufrt1_size  = conf->nstream * conf->sbufrt1_size;
  conf->bufrt2_size  = conf->nstream * conf->sbufrt2_size;
    
  //conf->hbufin_offset = conf->sbufin_size / sizeof(char);
  conf->hbufin_offset = conf->sbufin_size;
  conf->dbufin_offset = conf->sbufin_size / sizeof(int64_t);
  conf->bufrt1_offset = conf->sbufrt1_size / sizeof(hipfftComplex);
  conf->bufrt2_offset = conf->sbufrt2_size / sizeof(hipfftComplex);
  
  conf->dbufout_offset   = conf->sbufout_size / NBYTE_OUT;
  //conf->hbufout_offset   = conf->sbufout_size / sizeof(char);
  conf->hbufout_offset   = conf->sbufout_size;

  CudaSafeCall(hipMalloc((void **)&conf->dbuf_in, conf->bufin_size));
  
  CudaSafeCall(hipMalloc((void **)&conf->dbuf_out, conf->bufout_size));       
  CudaSafeCall(hipMalloc((void **)&conf->ddat_offs, NCHAN * sizeof(float)));
  CudaSafeCall(hipMalloc((void **)&conf->dsquare_mean, NCHAN * sizeof(float)));
  CudaSafeCall(hipMalloc((void **)&conf->ddat_scl, NCHAN * sizeof(float)));
      
  CudaSafeCall(hipMemset((void *)conf->ddat_offs, 0, NCHAN * sizeof(float)));   // We have to clear the memory for this parameter
  CudaSafeCall(hipMemset((void *)conf->dsquare_mean, 0, NCHAN * sizeof(float)));// We have to clear the memory for this parameter
  
  CudaSafeCall(hipHostMalloc((void **)&conf->hdat_scl, NCHAN * sizeof(float)));   // Malloc host memory to receive data from device
  CudaSafeCall(hipHostMalloc((void **)&conf->hdat_offs, NCHAN * sizeof(float)));   // Malloc host memory to receive data from device
  CudaSafeCall(hipHostMalloc((void **)&conf->hsquare_mean, NCHAN * sizeof(float)));   // Malloc host memory to receive data from device
  
  CudaSafeCall(hipMalloc((void **)&conf->buf_rt1, conf->bufrt1_size));
  CudaSafeCall(hipMalloc((void **)&conf->buf_rt2, conf->bufrt2_size)); 

  /* Prepare the setup of kernels */
  conf->gridsize_unpack.x = conf->stream_ndf;
  conf->gridsize_unpack.y = NCHK_CAPTURE;
  conf->gridsize_unpack.z = 1;
  conf->blocksize_unpack.x = NSAMP_DF; 
  conf->blocksize_unpack.y = NCHAN_CHK;
  conf->blocksize_unpack.z = 1;
  
  conf->gridsize_swap_select_transpose_swap.x = NCHK_CAPTURE * NCHAN_CHK;
  conf->gridsize_swap_select_transpose_swap.y = conf->stream_ndf * NSAMP_DF / CUFFT_NX1;
  conf->gridsize_swap_select_transpose_swap.z = 1;  
  conf->blocksize_swap_select_transpose_swap.x = CUFFT_NX1;
  conf->blocksize_swap_select_transpose_swap.y = 1;
  conf->blocksize_swap_select_transpose_swap.z = 1;
  
  conf->gridsize_mean.x = 1; 
  conf->gridsize_mean.y = 1; 
  conf->gridsize_mean.z = 1;
  conf->blocksize_mean.x = NCHAN; 
  conf->blocksize_mean.y = 1;
  conf->blocksize_mean.z = 1;
  
  conf->gridsize_scale.x = 1;
  conf->gridsize_scale.y = 1;
  conf->gridsize_scale.z = 1;
  conf->blocksize_scale.x = NCHAN;
  conf->blocksize_scale.y = 1;
  conf->blocksize_scale.z = 1;
  
  conf->gridsize_transpose_pad.x = conf->stream_ndf * NSAMP_DF / CUFFT_NX1; 
  conf->gridsize_transpose_pad.y = NCHAN;
  conf->gridsize_transpose_pad.z = 1;
  conf->blocksize_transpose_pad.x = CUFFT_NX2;
  conf->blocksize_transpose_pad.y = 1;
  conf->blocksize_transpose_pad.z = 1;

  conf->gridsize_sum1.x = NCHAN;
  conf->gridsize_sum1.y = conf->stream_ndf * NPOL_SAMP;
  conf->gridsize_sum1.z = 1;
  conf->blocksize_sum1.x = NSAMP_DF * CUFFT_NX2 / (2 * CUFFT_NX1);  // This is the right setup if CUFFT_NX2 is not equal to CUFFT_NX1
  conf->blocksize_sum1.y = 1;
  conf->blocksize_sum1.z = 1;
  
  conf->gridsize_sum2.x = NCHAN;
  conf->gridsize_sum2.y = 1;
  conf->gridsize_sum2.z = 1;
  conf->blocksize_sum2.x = conf->stream_ndf * NPOL_SAMP / 2;
  conf->blocksize_sum2.y = 1;
  conf->blocksize_sum2.z = 1;
  
  conf->gridsize_transpose_scale.x = conf->stream_ndf * NSAMP_DF / CUFFT_NX1; 
  conf->gridsize_transpose_scale.y = NCHAN / TILE_DIM;
  conf->gridsize_transpose_scale.z = 1;
  conf->blocksize_transpose_scale.x = TILE_DIM;
  conf->blocksize_transpose_scale.y = NROWBLOCK_TRANS;
  conf->blocksize_transpose_scale.z = 1;
  
  conf->gridsize_transpose_float.x = conf->stream_ndf * NSAMP_DF / CUFFT_NX1; 
  conf->gridsize_transpose_float.y = NCHAN / TILE_DIM;
  conf->gridsize_transpose_float.z = 1;
  conf->blocksize_transpose_float.x = TILE_DIM;
  conf->blocksize_transpose_float.y = NROWBLOCK_TRANS;
  conf->blocksize_transpose_float.z = 1;
  
  /* attach to input ring buffer */
  conf->hdu_in = dada_hdu_create(runtime_log);
  dada_hdu_set_key(conf->hdu_in, conf->key_in);
  if(dada_hdu_connect(conf->hdu_in) < 0)
    {
      multilog(runtime_log, LOG_ERR, "could not connect to hdu\n");
      fprintf(stderr, "Can not connect to hdu, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }  
  db = (ipcbuf_t *) conf->hdu_in->data_block;
  conf->rbufin_size = ipcbuf_get_bufsz(db);  
  if(conf->rbufin_size % conf->bufin_size != 0)  
    {
      multilog(runtime_log, LOG_ERR, "data buffer size mismatch\n");
      fprintf(stderr, "Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }
  
  /* registers the existing host memory range for use by CUDA */
  dada_cuda_dbregister(conf->hdu_in);
        
  conf->hdrsz = ipcbuf_get_bufsz(conf->hdu_in->header_block);  
  if(conf->hdrsz != DADA_HDR_SIZE)    // This number should match
    {
      multilog(runtime_log, LOG_ERR, "data buffer size mismatch\n");
      fprintf(stderr, "Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }
  
  /* make ourselves the read client */
  if(dada_hdu_lock_read(conf->hdu_in) < 0)
    {
      multilog(runtime_log, LOG_ERR, "open_hdu: could not lock write\n");
      fprintf(stderr, "Error locking HDU, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }

  /* Prepare output ring buffer */
  conf->hdu_out = dada_hdu_create(runtime_log);
  dada_hdu_set_key(conf->hdu_out, conf->key_out);
  if(dada_hdu_connect(conf->hdu_out) < 0)
    {
      multilog(runtime_log, LOG_ERR, "could not connect to hdu\n");
      fprintf(stderr, "Can not connect to hdu, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }
  db = (ipcbuf_t *) conf->hdu_out->data_block;
  conf->rbufout_size = ipcbuf_get_bufsz(db);
  if(conf->rbufout_size % conf->bufout_size != 0)  
    {
      multilog(runtime_log, LOG_ERR, "data buffer size mismatch\n");
      fprintf(stderr, "Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }
  
  conf->hdrsz = ipcbuf_get_bufsz(conf->hdu_out->header_block);  
  if(conf->hdrsz != DADA_HDR_SIZE)    // This number should match
    {
      multilog(runtime_log, LOG_ERR, "data buffer size mismatch\n");
      fprintf(stderr, "Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }  
  /* make ourselves the write client */
  if(dada_hdu_lock_write(conf->hdu_out) < 0)
    {
      multilog(runtime_log, LOG_ERR, "open_hdu: could not lock write\n");
      fprintf(stderr, "Error locking HDU, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  
  if(conf->sod)
    {      
      if(ipcbuf_enable_sod(db, 0, 0) < 0)  // We start at the beginning
  	{
	  multilog(runtime_log, LOG_ERR, "Can not write data before start, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
  	  fprintf(stderr, "Can not write data before start, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
  	  return EXIT_FAILURE;
  	}
    }
  else
    {
      if(ipcbuf_disable_sod(db) < 0)
  	{
	  multilog(runtime_log, LOG_ERR, "Can not write data before start, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
  	  fprintf(stderr, "Can not write data before start, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
  	  return EXIT_FAILURE;
  	}
    }
      
  /* Register header */
  if(register_header(conf))
    {
      multilog(runtime_log, LOG_ERR, "header register failed, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      fprintf(stderr, "header register failed, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  
  return EXIT_SUCCESS;
}

int do_baseband2baseband(conf_t conf)
{
  CudaSafeCall(hipSetDevice(conf.device_id));
  
  /*
    The whole procedure for fold mode is :
    1. Unpack the data and reorder it from TFTFP to PFT order, prepare for the forward FFT;
    2. Forward FFT the PFT data to get finer channelzation and the data is in PFTF order after FFT;
    3. Swap the FFT output to put the frequency centre on the right place, drop frequency channel edge and band edge and put the data into PTF order, swap the data and put the centre frequency at bin 0 for each FFT block, prepare for inverse FFT;
    4. Inverse FFT the data to get PTFT order data;
    5. Transpose the data to get TFP data and scale it;    
  */
  size_t i, j;
  size_t hbufin_offset, dbufin_offset, bufrt1_offset, bufrt2_offset, hbufout_offset, dbufout_offset;
  dim3 gridsize_unpack, blocksize_unpack;
  dim3 gridsize_swap_select_transpose_swap, blocksize_swap_select_transpose_swap;
  dim3 gridsize_transpose_scale, blocksize_transpose_scale;
  dim3 gridsize_transpose_float, blocksize_transpose_float;
  uint64_t block_id = 0;
  uint64_t curbufsz;
  
  gridsize_unpack                      = conf.gridsize_unpack;
  blocksize_unpack                     = conf.blocksize_unpack;
  gridsize_swap_select_transpose_swap  = conf.gridsize_swap_select_transpose_swap;   
  blocksize_swap_select_transpose_swap = conf.blocksize_swap_select_transpose_swap;  
  gridsize_transpose_scale             = conf.gridsize_transpose_scale;
  blocksize_transpose_scale            = conf.blocksize_transpose_scale;
  gridsize_transpose_float             = conf.gridsize_transpose_float;
  blocksize_transpose_float            = conf.blocksize_transpose_float;
  
  /* Get scale of data */
  dat_offs_scl(conf);
  
  /* Do the real job */
  conf.hdu_out->data_block->curbuf = ipcio_open_block_write(conf.hdu_out->data_block, &block_id);   /* Open buffer to write */
  
  while(conf.hdu_in->data_block->curbufsz == conf.rbufin_size)
    // The first time we open a block at the scale calculation, we need to make sure that the input ring buffer block is bigger than the block needed for scale calculation
    // Otherwise we have to open couple of blocks to calculate scales and these blocks will dropped after that
    {
      //for(i = 0; i < conf.rbufin_size; i += conf.bufin_size)
      for(i = 0; i < conf.nrun_blk; i ++)
	{
	  //fprintf(stdout, "REPEAT HERE\n\n");
	  for(j = 0; j < conf.nstream; j++)
	    {
	      //fprintf(stdout, "STREAM HERE 1\t");
	      hbufin_offset = j * conf.hbufin_offset + i * conf.bufin_size;
	      dbufin_offset = j * conf.dbufin_offset; 
	      bufrt1_offset = j * conf.bufrt1_offset;
	      bufrt2_offset = j * conf.bufrt2_offset;

	      dbufout_offset = j * conf.dbufout_offset;
	      hbufout_offset = j * conf.hbufout_offset + i * conf.bufout_size;
	      
	      CudaSafeCall(hipMemcpyAsync(&conf.dbuf_in[dbufin_offset], &conf.hdu_in->data_block->curbuf[hbufin_offset], conf.sbufin_size, hipMemcpyHostToDevice, conf.streams[j]));
	      	      
	      /* Unpack raw data into hipfftComplex array */
	      unpack_kernel<<<gridsize_unpack, blocksize_unpack, 0, conf.streams[j]>>>(&conf.dbuf_in[dbufin_offset], &conf.buf_rt1[bufrt1_offset], conf.nsamp1);
	      
	      /* Do forward FFT */
	      CufftSafeCall(hipfftExecC2C(conf.fft_plans1[j], &conf.buf_rt1[bufrt1_offset], &conf.buf_rt1[bufrt1_offset], HIPFFT_FORWARD));

	      /* Prepare for inverse FFT */
	      swap_select_transpose_swap_kernel<<<gridsize_swap_select_transpose_swap, blocksize_swap_select_transpose_swap, 0, conf.streams[j]>>>(&conf.buf_rt1[bufrt1_offset], &conf.buf_rt2[bufrt2_offset], conf.nsamp1, conf.nsamp2); 
	      /* Do inverse FFT */
	      CufftSafeCall(hipfftExecC2C(conf.fft_plans2[j], &conf.buf_rt2[bufrt2_offset], &conf.buf_rt2[bufrt2_offset], HIPFFT_BACKWARD));
	      /* Get final output */
	      transpose_scale_kernel<<<gridsize_transpose_scale, blocksize_transpose_scale, 0, conf.streams[j]>>>(&conf.buf_rt2[bufrt2_offset], &conf.dbuf_out[dbufout_offset], conf.nsamp2, conf.ddat_offs, conf.ddat_scl);   
	      /* Copy the final output to host */
	      CudaSafeCall(hipMemcpyAsync(&conf.hdu_out->data_block->curbuf[hbufout_offset], &conf.dbuf_out[dbufout_offset], conf.sbufout_size, hipMemcpyDeviceToHost, conf.streams[j]));
	    }
	  CudaSynchronizeCall(); // Sync here is for multiple streams
	}
      	  
      /* Close current buffer */
      if(ipcio_close_block_write(conf.hdu_out->data_block, conf.rbufout_size) < 0)
	{
	  multilog (runtime_log, LOG_ERR, "close_buffer: ipcio_close_block_write failed\n");
	  fprintf(stderr, "close_buffer: ipcio_close_block_write failed, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	  return EXIT_FAILURE;
	}
      conf.hdu_out->data_block->curbuf = ipcio_open_block_write(conf.hdu_out->data_block, &block_id);   /* Open buffer to write */
      
      ipcio_close_block_read(conf.hdu_in->data_block, conf.hdu_in->data_block->curbufsz);
      conf.hdu_in->data_block->curbuf = ipcio_open_block_read(conf.hdu_in->data_block, &curbufsz, &block_id);
    }

  ipcio_close_block_read(conf.hdu_in->data_block, conf.hdu_in->data_block->curbufsz);
  if (ipcio_close_block_write(conf.hdu_out->data_block, conf.rbufout_size) < 0)
    {
      multilog (runtime_log, LOG_ERR, "close_buffer: ipcio_close_block_write failed\n");
      fprintf(stderr, "close_buffer: ipcio_close_block_write failed, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }

  return EXIT_SUCCESS;
}

int dat_offs_scl(conf_t conf)
{
  CudaSafeCall(hipSetDevice(conf.device_id));

  /*
    The procedure for fold mode is:
    1. Get PTFT data as we did at process;
    2. Pad the data;
    3. Add the padded data in time;
    4. Get the mean of the added data;
    5. Get the scale with the mean;
  */
  size_t i, j;
  dim3 gridsize_unpack, blocksize_unpack;
  dim3 gridsize_swap_select_transpose_swap, blocksize_swap_select_transpose_swap;
  dim3 gridsize_mean, blocksize_mean;
  dim3 gridsize_sum1, blocksize_sum1;
  dim3 gridsize_sum2, blocksize_sum2;
  dim3 gridsize_scale, blocksize_scale;
  dim3 gridsize_transpose_pad, blocksize_transpose_pad;
  uint64_t hbufin_offset, dbufin_offset, bufrt1_offset, bufrt2_offset;
  uint64_t curbufsz, block_id;
  
  char fname[MSTR_LEN];
  FILE *fp=NULL;
    
  gridsize_unpack                      = conf.gridsize_unpack;
  blocksize_unpack                     = conf.blocksize_unpack;
  gridsize_swap_select_transpose_swap  = conf.gridsize_swap_select_transpose_swap;   
  blocksize_swap_select_transpose_swap = conf.blocksize_swap_select_transpose_swap; 
  gridsize_transpose_pad               = conf.gridsize_transpose_pad;
  blocksize_transpose_pad              = conf.blocksize_transpose_pad;
  	         	               						       
  gridsize_sum1              = conf.gridsize_sum1;	       
  blocksize_sum1             = conf.blocksize_sum1;
  gridsize_sum2              = conf.gridsize_sum2;	       
  blocksize_sum2             = conf.blocksize_sum2;
  gridsize_mean              = conf.gridsize_mean;	       
  blocksize_mean             = conf.blocksize_mean;
  gridsize_scale              = conf.gridsize_scale;	       
  blocksize_scale             = conf.blocksize_scale;
  
  conf.hdu_in->data_block->curbuf = ipcio_open_block_read(conf.hdu_in->data_block, &curbufsz, &block_id);
  if(conf.hdu_in->data_block->curbuf == NULL)
    {
      multilog (runtime_log, LOG_ERR, "Can not get buffer block from input ring buffer, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      fprintf(stderr, "Can not get buffer block from input ring buffer, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
    
  for(i = 0; i < conf.rbufin_size; i += conf.bufin_size)
    {
      for (j = 0; j < conf.nstream; j++)
	{
	  hbufin_offset = j * conf.hbufin_offset + i;
	  dbufin_offset = j * conf.dbufin_offset; 
	  bufrt1_offset = j * conf.bufrt1_offset;
	  bufrt2_offset = j * conf.bufrt2_offset;
	  
	  /* Copy data into device */
	  CudaSafeCall(hipMemcpyAsync(&conf.dbuf_in[dbufin_offset], &conf.hdu_in->data_block->curbuf[hbufin_offset], conf.sbufin_size, hipMemcpyHostToDevice, conf.streams[j]));

	  /* Unpack raw data into hipfftComplex array */
	  unpack_kernel<<<gridsize_unpack, blocksize_unpack, 0, conf.streams[j]>>>(&conf.dbuf_in[dbufin_offset], &conf.buf_rt1[bufrt1_offset], conf.nsamp1);

	  /* Do forward FFT */
	  CufftSafeCall(hipfftExecC2C(conf.fft_plans1[j], &conf.buf_rt1[bufrt1_offset], &conf.buf_rt1[bufrt1_offset], HIPFFT_FORWARD));

	  /* Prepare for inverse FFT */
	  swap_select_transpose_swap_kernel<<<gridsize_swap_select_transpose_swap, blocksize_swap_select_transpose_swap, 0, conf.streams[j]>>>(&conf.buf_rt1[bufrt1_offset], &conf.buf_rt2[bufrt2_offset], conf.nsamp1, conf.nsamp2); 
	  
	  /* Do inverse FFT */
	  CufftSafeCall(hipfftExecC2C(conf.fft_plans2[j], &conf.buf_rt2[bufrt2_offset], &conf.buf_rt2[bufrt2_offset], HIPFFT_BACKWARD));
	  
	  /* Transpose the data from PTFT to FTP for later calculation */
	  transpose_pad_kernel<<<gridsize_transpose_pad, blocksize_transpose_pad, 0, conf.streams[j]>>>(&conf.buf_rt2[bufrt2_offset], conf.nsamp2, &conf.buf_rt1[bufrt1_offset]);
	  
	  /* Get the sum of samples and square of samples */
	  sum_kernel<<<gridsize_sum1, blocksize_sum1, blocksize_sum1.x * sizeof(hipfftComplex), conf.streams[j]>>>(&conf.buf_rt1[bufrt1_offset], &conf.buf_rt2[bufrt2_offset]);
	  sum_kernel<<<gridsize_sum2, blocksize_sum2, blocksize_sum2.x * sizeof(hipfftComplex), conf.streams[j]>>>(&conf.buf_rt2[bufrt2_offset], &conf.buf_rt1[bufrt1_offset]);
	}
      CudaSynchronizeCall(); // Sync here is for multiple streams

      mean_kernel<<<gridsize_mean, blocksize_mean>>>(conf.buf_rt1, conf.bufrt1_offset, conf.ddat_offs, conf.dsquare_mean, conf.nstream, conf.sclndim);
    }
  /* Get the scale of each chanel */
  scale_kernel<<<gridsize_scale, blocksize_scale>>>(conf.ddat_offs, conf.dsquare_mean, conf.ddat_scl);
  CudaSynchronizeCall();
  
  CudaSafeCall(hipMemcpy(conf.hdat_offs, conf.ddat_offs, sizeof(float) * NCHAN, hipMemcpyDeviceToHost));
  CudaSafeCall(hipMemcpy(conf.hdat_scl, conf.ddat_scl, sizeof(float) * NCHAN, hipMemcpyDeviceToHost));
  CudaSafeCall(hipMemcpy(conf.hsquare_mean, conf.dsquare_mean, sizeof(float) * NCHAN, hipMemcpyDeviceToHost));
 
  for (i = 0; i< NCHAN; i++)
    fprintf(stdout, "DAT_OFFS:\t%E\tDAT_SCL:\t%E\n", conf.hdat_offs[i], conf.hdat_scl[i]);
  /* Record scale into file */
  sprintf(fname, "%s/%s_scale.txt", conf.dir, conf.utc_start);
  fp = fopen(fname, "w");
  if(fp == NULL)
    {
      multilog (runtime_log, LOG_ERR, "Can not open scale file, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      fprintf(stderr, "Can not open scale file, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }

  for (i = 0; i< NCHAN; i++)
    fprintf(fp, "%E\t%E\n", conf.hdat_offs[i], conf.hdat_scl[i]);
  fclose(fp);
  return EXIT_SUCCESS;
}

int destroy_baseband2baseband(conf_t conf)
{
  int i;
  CudaSafeCall(hipSetDevice(conf.device_id));
  
  for (i = 0; i < conf.nstream; i++)
    {
      CudaSafeCall(hipStreamDestroy(conf.streams[i]));
      CufftSafeCall(hipfftDestroy(conf.fft_plans1[i]));
      CufftSafeCall(hipfftDestroy(conf.fft_plans2[i]));
    }
  
  hipFree(conf.dbuf_in);

  hipFree(conf.dbuf_out);
  hipHostFree(conf.hdat_offs);
  hipHostFree(conf.hsquare_mean);
  hipHostFree(conf.hdat_scl);
  hipFree(conf.ddat_offs);
  hipFree(conf.dsquare_mean);
  hipFree(conf.ddat_scl);
  
  dada_hdu_unlock_write(conf.hdu_out);
  dada_hdu_disconnect(conf.hdu_out);
  dada_hdu_destroy(conf.hdu_out);

  hipFree(conf.buf_rt1);
  hipFree(conf.buf_rt2);

  dada_cuda_dbunregister(conf.hdu_in);
  
  dada_hdu_unlock_read(conf.hdu_in);
  dada_hdu_disconnect(conf.hdu_in);
  dada_hdu_destroy(conf.hdu_in);

  free(conf.streams);
  free(conf.fft_plans1);
  free(conf.fft_plans2);
  
  return EXIT_SUCCESS;
}

int register_header(conf_t *conf)
{
  size_t hdrsz;
  
  conf->hdrbuf_in  = ipcbuf_get_next_read(conf->hdu_in->header_block, &hdrsz);  
  if (!conf->hdrbuf_in)
    {
      multilog(runtime_log, LOG_ERR, "get next header block error.\n");
      fprintf(stderr, "Error getting header_buf, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  if(hdrsz != DADA_HDR_SIZE)
    {
      multilog(runtime_log, LOG_ERR, "get next header block error.\n");
      fprintf(stderr, "Header size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }

  conf->hdrbuf_out = ipcbuf_get_next_write(conf->hdu_out->header_block);
  if (!conf->hdrbuf_out)
    {
      multilog(runtime_log, LOG_ERR, "get next header block error.\n");
      fprintf(stderr, "Error getting header_buf, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	  return EXIT_FAILURE;
    }  

//  if(conf->stream)
//    {      
//      /* Get utc_start from hdrin */
//      if (ascii_header_get(conf->hdrbuf_in, "UTC_START", "%s", conf->utc_start) < 0)  
//	{
//	  multilog(runtime_log, LOG_ERR, "failed ascii_header_get UTC_START\n");
//	  fprintf(stderr, "Error getting UTC_START, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
//	  return EXIT_FAILURE;
//	}
//      fprintf(stdout, "\nGet UTC_START at process stage:\t\t%s\n", conf->utc_start);
//      
//      /* Get picoseconds from hdrin */
//      if (ascii_header_get(conf->hdrbuf_in, "PICOSECONDS", "%"PRIu64, &(conf->picoseconds)) < 0)  
//	{
//	  multilog(runtime_log, LOG_ERR, "failed ascii_header_get PICOSECONDS\n");
//	  fprintf(stderr, "Error getting PICOSECONDS, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
//	  return EXIT_FAILURE;
//	}
//      fprintf(stdout, "Get PICOSECONDS at process stage:\t%"PRIu64"\n", conf->picoseconds);
//      
//      /* Get frequency from hdrin */
//      if (ascii_header_get(conf->hdrbuf_in, "FREQ", "%lf", &freq) < 0)   // RA and DEC also need to pass from hdrin to hdrout
//	{
//	  multilog(runtime_log, LOG_ERR, "failed ascii_header_get FREQ\n");
//	  fprintf(stderr, "Error getting FREQ, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
//	  return EXIT_FAILURE;
//	}
//      if (fileread(conf->hfname, conf->hdrbuf_out, DADA_HDR_SIZE) < 0)
//	{
//	  multilog(runtime_log, LOG_ERR, "cannot read header from %s\n", conf->hfname);
//	  fprintf(stderr, "Error reading header file, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
//	  return EXIT_FAILURE;
//	}
//      
//      /* Pass utc_start */
//      if (ascii_header_set(conf->hdrbuf_out, "UTC_START", "%s", conf->utc_start) < 0)  
//	{
//	  multilog(runtime_log, LOG_ERR, "failed ascii_header_set UTC_START\n");
//	  fprintf(stderr, "Error setting UTC_START, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
//	  return EXIT_FAILURE;
//	}	  
//      fprintf(stdout, "Set UTC_START at process stage:\t\t%s\n", conf->utc_start);
//      multilog(runtime_log, LOG_INFO, "UTC_START:\t%s\n", conf->utc_start);
//      
//      /* Pass picoseconds */
//      if (ascii_header_set(conf->hdrbuf_out, "PICOSECONDS", "%"PRIu64, conf->picoseconds) < 0)  
//	{
//	  multilog(runtime_log, LOG_ERR, "failed ascii_header_set PICOSECONDS\n");
//	  fprintf(stderr, "Error setting PICOSECONDS, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
//	  return EXIT_FAILURE;
//	}	  
//      fprintf(stdout, "Set PICOSECONDS at process stage:\t%"PRIu64"\n\n", conf->picoseconds);
//      multilog(runtime_log, LOG_INFO, "PICOSECONDS:\t%"PRIu64"\n", conf->picoseconds);
//      
//      /* Pass frequency */
//      if (ascii_header_set(conf->hdrbuf_out, "FREQ", "%.1lf", freq) < 0)  
//	{
//	  multilog(runtime_log, LOG_ERR, "failed ascii_header_set FREQ\n");
//	  fprintf(stderr, "Error setting FREQ, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
//	  return EXIT_FAILURE;
//	}
//    }
//  else
//    {      
//      if (ascii_header_get(conf->hdrbuf_in, "UTC_START", "%s", conf->utc_start) < 0)  
//	{
//	  multilog(runtime_log, LOG_ERR, "failed ascii_header_get UTC_START\n");
//	  fprintf(stderr, "Error getting UTC_START, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
//	  return EXIT_FAILURE;
//	}
//      memcpy(conf->hdrbuf_out, conf->hdrbuf_in, DADA_HDR_SIZE);
//    }
//  if(ipcbuf_mark_cleared (conf->hdu_in->header_block))  // We are the only one reader, so that we can clear it after read;
//    {
//      multilog(runtime_log, LOG_ERR, "Could not clear header block\n");
//      fprintf(stderr, "Error header_clear, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
//      return EXIT_FAILURE;
//    }
//
  /* donot set header parameters anymore - acqn. doesn't start */
  if (ipcbuf_mark_filled (conf->hdu_out->header_block, conf->hdrsz) < 0)
    {
      multilog(runtime_log, LOG_ERR, "Could not mark filled header block\n");
      fprintf(stderr, "Error header_fill, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  return EXIT_SUCCESS;
}