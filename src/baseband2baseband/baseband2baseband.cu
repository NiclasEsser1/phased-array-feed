#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

#include <time.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>
#include <math.h>

#include "multilog.h"
#include "baseband2baseband.cuh"
#include "cudautil.cuh"
#include "kernel.cuh"

extern multilog_t *runtime_log;

int init_baseband2baseband(conf_t *conf)
{
  int i;
  int iembed1, istride1, idist1, oembed1, ostride1, odist1, batch1, nx1;
  int iembed2, istride2, idist2, oembed2, ostride2, odist2, batch2, nx2;
  uint64_t hdrsz;
  
  /* Prepare buffer, stream and fft plan for process */
  conf->sclndim = conf->rbufin_ndf_chk * NSAMP_DF * NPOL_SAMP * NDIM_POL; // Only works when two polarisations has similar power level
  conf->nsamp1  = conf->stream_ndf_chk * NCHK_BEAM * NCHAN_CHK * NSAMP_DF;
  conf->npol1   = conf->nsamp1 * NPOL_SAMP;
  conf->ndata1  = conf->npol1  * NDIM_POL;
		
  conf->nsamp2  = conf->nsamp1 * OSAMP_RATEI / NCHAN_RATEI;
  conf->npol2   = conf->nsamp2 * NPOL_SAMP;
  conf->ndata2  = conf->npol2  * NDIM_POL;

  nx1        = CUFFT_NX1;
  batch1     = conf->npol1 / CUFFT_NX1;
  
  iembed1    = nx1;
  istride1   = 1;
  idist1     = nx1;
  
  oembed1    = nx1;
  ostride1   = 1;
  odist1     = nx1;
  
  nx2        = CUFFT_NX2;
  batch2     = conf->npol2 / CUFFT_NX2;
  
  iembed2    = nx2;
  istride2   = 1;
  idist2     = nx2;
  
  oembed2    = nx2;
  ostride2   = 1;
  odist2     = nx2;

  conf->streams = (hipStream_t *)malloc(conf->nstream * sizeof(hipStream_t));
  conf->fft_plans1 = (hipfftHandle *)malloc(conf->nstream * sizeof(hipfftHandle));
  conf->fft_plans2 = (hipfftHandle *)malloc(conf->nstream * sizeof(hipfftHandle));
  for(i = 0; i < conf->nstream; i ++)
    {
      CudaSafeCall(hipStreamCreate(&conf->streams[i]));
      CufftSafeCall(hipfftPlanMany(&conf->fft_plans1[i], CUFFT_RANK1, &nx1, &iembed1, istride1, idist1, &oembed1, ostride1, odist1, HIPFFT_C2C, batch1));
      CufftSafeCall(hipfftPlanMany(&conf->fft_plans2[i], CUFFT_RANK2, &nx2, &iembed2, istride2, idist2, &oembed2, ostride2, odist2, HIPFFT_C2C, batch2));
      
      CufftSafeCall(hipfftSetStream(conf->fft_plans1[i], conf->streams[i]));
      CufftSafeCall(hipfftSetStream(conf->fft_plans2[i], conf->streams[i]));
    }
  
  conf->sbufin_size    = conf->ndata1 * NBYTE_IN;
  conf->sbufout_size   = conf->ndata2 * NBYTE_OUT;
  
  conf->bufin_size     = conf->nstream * conf->sbufin_size;
  conf->bufout_size    = conf->nstream * conf->sbufout_size;
  
  conf->sbufrt1_size = conf->npol1 * sizeof(hipfftComplex);
  conf->sbufrt2_size = conf->npol2 * sizeof(hipfftComplex);
  conf->bufrt1_size  = conf->nstream * conf->sbufrt1_size;
  conf->bufrt2_size  = conf->nstream * conf->sbufrt2_size;
    
  //conf->hbufin_offset = conf->sbufin_size / sizeof(char);
  conf->hbufin_offset = conf->sbufin_size;
  conf->dbufin_offset = conf->sbufin_size / sizeof(int64_t);
  conf->bufrt1_offset = conf->sbufrt1_size / sizeof(hipfftComplex);
  conf->bufrt2_offset = conf->sbufrt2_size / sizeof(hipfftComplex);
  
  conf->dbufout_offset   = conf->sbufout_size / NBYTE_OUT;
  //conf->hbufout_offset   = conf->sbufout_size / sizeof(char);
  conf->hbufout_offset   = conf->sbufout_size;

  CudaSafeCall(hipMalloc((void **)&conf->dbuf_in, conf->bufin_size));
  
  CudaSafeCall(hipMalloc((void **)&conf->dbuf_out, conf->bufout_size));       
  CudaSafeCall(hipMalloc((void **)&conf->ddat_offs, NCHAN_OUT * sizeof(float)));
  CudaSafeCall(hipMalloc((void **)&conf->dsquare_mean, NCHAN_OUT * sizeof(float)));
  CudaSafeCall(hipMalloc((void **)&conf->ddat_scl, NCHAN_OUT * sizeof(float)));
      
  CudaSafeCall(hipMemset((void *)conf->ddat_offs, 0, NCHAN_OUT * sizeof(float)));   // We have to clear the memory for this parameter
  CudaSafeCall(hipMemset((void *)conf->dsquare_mean, 0, NCHAN_OUT * sizeof(float)));// We have to clear the memory for this parameter
  
  CudaSafeCall(hipHostMalloc((void **)&conf->hdat_scl, NCHAN_OUT * sizeof(float)));   // Malloc host memory to receive data from device
  CudaSafeCall(hipHostMalloc((void **)&conf->hdat_offs, NCHAN_OUT * sizeof(float)));   // Malloc host memory to receive data from device
  CudaSafeCall(hipHostMalloc((void **)&conf->hsquare_mean, NCHAN_OUT * sizeof(float)));   // Malloc host memory to receive data from device
  
  CudaSafeCall(hipMalloc((void **)&conf->buf_rt1, conf->bufrt1_size));
  CudaSafeCall(hipMalloc((void **)&conf->buf_rt2, conf->bufrt2_size)); 

  /* Prepare the setup of kernels */
  conf->gridsize_unpack.x = conf->stream_ndf_chk;
  conf->gridsize_unpack.y = NCHK_BEAM;
  conf->gridsize_unpack.z = 1;
  conf->blocksize_unpack.x = NSAMP_DF; 
  conf->blocksize_unpack.y = NCHAN_CHK;
  conf->blocksize_unpack.z = 1;
  
  conf->gridsize_swap_select_transpose_swap.x = NCHK_BEAM * NCHAN_CHK;
  conf->gridsize_swap_select_transpose_swap.y = conf->stream_ndf_chk * NSAMP_DF / CUFFT_NX1;
  conf->gridsize_swap_select_transpose_swap.z = 1;  
  conf->blocksize_swap_select_transpose_swap.x = CUFFT_NX1;
  conf->blocksize_swap_select_transpose_swap.y = 1;
  conf->blocksize_swap_select_transpose_swap.z = 1;
  
  conf->gridsize_mean.x = 1; 
  conf->gridsize_mean.y = 1; 
  conf->gridsize_mean.z = 1;
  conf->blocksize_mean.x = NCHAN_OUT; 
  conf->blocksize_mean.y = 1;
  conf->blocksize_mean.z = 1;
  
  conf->gridsize_scale.x = 1;
  conf->gridsize_scale.y = 1;
  conf->gridsize_scale.z = 1;
  conf->blocksize_scale.x = NCHAN_OUT;
  conf->blocksize_scale.y = 1;
  conf->blocksize_scale.z = 1;
  
  conf->gridsize_transpose_pad.x = conf->stream_ndf_chk * NSAMP_DF / CUFFT_NX1; 
  conf->gridsize_transpose_pad.y = NCHAN_OUT;
  conf->gridsize_transpose_pad.z = 1;
  conf->blocksize_transpose_pad.x = CUFFT_NX2;
  conf->blocksize_transpose_pad.y = 1;
  conf->blocksize_transpose_pad.z = 1;

  conf->gridsize_sum1.x = NCHAN_OUT;
  conf->gridsize_sum1.y = conf->stream_ndf_chk * NPOL_SAMP;
  conf->gridsize_sum1.z = 1;
  conf->blocksize_sum1.x = NSAMP_DF * CUFFT_NX2 / (2 * CUFFT_NX1);  // This is the right setup if CUFFT_NX2 is not equal to CUFFT_NX1
  conf->blocksize_sum1.y = 1;
  conf->blocksize_sum1.z = 1;
  
  conf->gridsize_sum2.x = NCHAN_OUT;
  conf->gridsize_sum2.y = 1;
  conf->gridsize_sum2.z = 1;
  conf->blocksize_sum2.x = conf->stream_ndf_chk * NPOL_SAMP / 2;
  conf->blocksize_sum2.y = 1;
  conf->blocksize_sum2.z = 1;
  
  conf->gridsize_transpose_scale.x = conf->stream_ndf_chk * NSAMP_DF / CUFFT_NX1; 
  conf->gridsize_transpose_scale.y = NCHAN_OUT / TILE_DIM;
  conf->gridsize_transpose_scale.z = 1;
  conf->blocksize_transpose_scale.x = TILE_DIM;
  conf->blocksize_transpose_scale.y = NROWBLOCK_TRANS;
  conf->blocksize_transpose_scale.z = 1;
  
  conf->gridsize_transpose_float.x = conf->stream_ndf_chk * NSAMP_DF / CUFFT_NX1; 
  conf->gridsize_transpose_float.y = NCHAN_OUT / TILE_DIM;
  conf->gridsize_transpose_float.z = 1;
  conf->blocksize_transpose_float.x = TILE_DIM;
  conf->blocksize_transpose_float.y = NROWBLOCK_TRANS;
  conf->blocksize_transpose_float.z = 1;
  
  /* attach to input ring buffer */
  conf->hdu_in = dada_hdu_create(runtime_log);
  dada_hdu_set_key(conf->hdu_in, conf->key_in);
  if(dada_hdu_connect(conf->hdu_in) < 0)
    {
      multilog(runtime_log, LOG_ERR, "could not connect to hdu\n");
      fprintf(stderr, "Can not connect to hdu, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }  
  conf->db_in = (ipcbuf_t *) conf->hdu_in->data_block;
  conf->rbufin_size = ipcbuf_get_bufsz(conf->db_in);
  //fprintf(stdout, "%"PRIu64"\t%"PRIu64"\n", conf->rbufin_size, conf->bufin_size);
  
  if(conf->rbufin_size % conf->bufin_size != 0)  
    {
      multilog(runtime_log, LOG_ERR, "data buffer size mismatch\n");
      fprintf(stderr, "Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }
  dada_cuda_dbregister(conf->hdu_in);  // registers the existing host memory range for use by CUDA   
  hdrsz = ipcbuf_get_bufsz(conf->hdu_in->header_block);  
  if(hdrsz != DADA_HDRSZ)    // This number should match
    {
      multilog(runtime_log, LOG_ERR, "data buffer size mismatch\n");
      fprintf(stderr, "Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }
  if(dada_hdu_lock_read(conf->hdu_in) < 0) // make ourselves the read client 
    {
      multilog(runtime_log, LOG_ERR, "open_hdu: could not lock write\n");
      fprintf(stderr, "Error locking HDU, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }

  /* Prepare output ring buffer */
  conf->hdu_out = dada_hdu_create(runtime_log);
  dada_hdu_set_key(conf->hdu_out, conf->key_out);
  if(dada_hdu_connect(conf->hdu_out) < 0)
    {
      multilog(runtime_log, LOG_ERR, "could not connect to hdu\n");
      fprintf(stderr, "Can not connect to hdu, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }
  conf->db_out = (ipcbuf_t *) conf->hdu_out->data_block;
  conf->rbufout_size = ipcbuf_get_bufsz(conf->db_out);
  if(conf->rbufout_size % conf->bufout_size != 0)  
    {
      multilog(runtime_log, LOG_ERR, "data buffer size mismatch\n");
      fprintf(stderr, "Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }  
  hdrsz = ipcbuf_get_bufsz(conf->hdu_out->header_block);  
  if(hdrsz != DADA_HDRSZ)    // This number should match
    {
      multilog(runtime_log, LOG_ERR, "data buffer size mismatch\n");
      fprintf(stderr, "Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }  
  if(dada_hdu_lock_write(conf->hdu_out) < 0)   // make ourselves the write client 
    {
      multilog(runtime_log, LOG_ERR, "open_hdu: could not lock write\n");
      fprintf(stderr, "Error locking HDU, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }

  //ipcbuf_disable_sod(conf->db_out);
  
  return EXIT_SUCCESS;
}

int baseband2baseband(conf_t conf)
{
  /*
    The whole procedure for fold mode is :
    1. Unpack the data and reorder it from TFTFP to PFT order, prepare for the forward FFT;
    2. Forward FFT the PFT data to get finer channelzation and the data is in PFTF order after FFT;
    3. Swap the FFT output to put the frequency centre on the right place, drop frequency channel edge and band edge and put the data into PTF order, swap the data and put the centre frequency at bin 0 for each FFT block, prepare for inverse FFT;
    4. Inverse FFT the data to get PTFT order data;
    5. Transpose the data to get TFP data and scale it;    
  */
  uint64_t i, j;
  uint64_t hbufin_offset, dbufin_offset, bufrt1_offset, bufrt2_offset, hbufout_offset, dbufout_offset;
  dim3 gridsize_unpack, blocksize_unpack;
  dim3 gridsize_swap_select_transpose_swap, blocksize_swap_select_transpose_swap;
  dim3 gridsize_transpose_scale, blocksize_transpose_scale;
  dim3 gridsize_transpose_float, blocksize_transpose_float;
  uint64_t curbufsz;
  
  gridsize_unpack                      = conf.gridsize_unpack;
  blocksize_unpack                     = conf.blocksize_unpack;
  gridsize_swap_select_transpose_swap  = conf.gridsize_swap_select_transpose_swap;   
  blocksize_swap_select_transpose_swap = conf.blocksize_swap_select_transpose_swap;  
  gridsize_transpose_scale             = conf.gridsize_transpose_scale;
  blocksize_transpose_scale            = conf.blocksize_transpose_scale;
  gridsize_transpose_float             = conf.gridsize_transpose_float;
  blocksize_transpose_float            = conf.blocksize_transpose_float;

  register_header(&conf); // To register header, pass here means the start-of-data is enabled from capture software;
  
  /* Do the real job */  
  while(!ipcbuf_eod(conf.db_in))
    // The first time we open a block at the scale calculation, we need to make sure that the input ring buffer block is bigger than the block needed for scale calculation
    // Otherwise we have to open couple of blocks to calculate scales and these blocks will dropped after that
    {
      fprintf(stdout, "EOD:\t%d\n", ipcbuf_eod(conf.db_in));
      conf.curbuf_in  = ipcbuf_get_next_read(conf.db_in, &curbufsz);
      conf.curbuf_out = ipcbuf_get_next_write(conf.db_out);
      
      for(i = 0; i < conf.nrun_blk; i ++)
	{
	  for(j = 0; j < conf.nstream; j++)
	    {
	      hbufin_offset = j * conf.hbufin_offset + i * conf.bufin_size;
	      dbufin_offset = j * conf.dbufin_offset; 
	      bufrt1_offset = j * conf.bufrt1_offset;
	      bufrt2_offset = j * conf.bufrt2_offset;

	      dbufout_offset = j * conf.dbufout_offset;
	      hbufout_offset = j * conf.hbufout_offset + i * conf.bufout_size;
	      
	      CudaSafeCall(hipMemcpyAsync(&conf.dbuf_in[dbufin_offset], &conf.curbuf_in[hbufin_offset], conf.sbufin_size, hipMemcpyHostToDevice, conf.streams[j]));
	      
	      /* Unpack raw data into hipfftComplex array */
	      unpack_kernel<<<gridsize_unpack, blocksize_unpack, 0, conf.streams[j]>>>(&conf.dbuf_in[dbufin_offset], &conf.buf_rt1[bufrt1_offset], conf.nsamp1);
	      
	      /* Do forward FFT */
	      CufftSafeCall(hipfftExecC2C(conf.fft_plans1[j], &conf.buf_rt1[bufrt1_offset], &conf.buf_rt1[bufrt1_offset], HIPFFT_FORWARD));

	      /* Prepare for inverse FFT */
	      swap_select_transpose_swap_kernel<<<gridsize_swap_select_transpose_swap, blocksize_swap_select_transpose_swap, 0, conf.streams[j]>>>(&conf.buf_rt1[bufrt1_offset], &conf.buf_rt2[bufrt2_offset], conf.nsamp1, conf.nsamp2); 
	      /* Do inverse FFT */
	      CufftSafeCall(hipfftExecC2C(conf.fft_plans2[j], &conf.buf_rt2[bufrt2_offset], &conf.buf_rt2[bufrt2_offset], HIPFFT_BACKWARD));
	      /* Get final output */
	      transpose_scale_kernel<<<gridsize_transpose_scale, blocksize_transpose_scale, 0, conf.streams[j]>>>(&conf.buf_rt2[bufrt2_offset], &conf.dbuf_out[dbufout_offset], conf.nsamp2, conf.ddat_offs, conf.ddat_scl);   
	      /* Copy the final output to host */
	      CudaSafeCall(hipMemcpyAsync(&conf.curbuf_out[hbufout_offset], &conf.dbuf_out[dbufout_offset], conf.sbufout_size, hipMemcpyDeviceToHost, conf.streams[j]));
	    }
	  CudaSynchronizeCall(); // Sync here is for multiple streams
	}
      	  
      /* Close current buffer */
      ipcbuf_mark_filled(conf.db_out, curbufsz);
      ipcbuf_mark_cleared(conf.db_in);
      
    }
  
  return EXIT_SUCCESS;
}

int destroy_baseband2baseband(conf_t conf)
{
  int i;
  
  for (i = 0; i < conf.nstream; i++)
    {
      CudaSafeCall(hipStreamDestroy(conf.streams[i]));
      CufftSafeCall(hipfftDestroy(conf.fft_plans1[i]));
      CufftSafeCall(hipfftDestroy(conf.fft_plans2[i]));
    }
  
  hipFree(conf.dbuf_in);
  hipFree(conf.buf_rt1);
  hipFree(conf.buf_rt2);

  hipFree(conf.dbuf_out);
  hipHostFree(conf.hdat_offs);
  hipHostFree(conf.hsquare_mean);
  hipHostFree(conf.hdat_scl);
  hipFree(conf.ddat_offs);
  hipFree(conf.dsquare_mean);
  hipFree(conf.ddat_scl);
  
  dada_hdu_unlock_write(conf.hdu_out);
  dada_hdu_disconnect(conf.hdu_out);
  dada_hdu_destroy(conf.hdu_out);

  dada_cuda_dbunregister(conf.hdu_in);  
  dada_hdu_unlock_read(conf.hdu_in);
  dada_hdu_disconnect(conf.hdu_in);
  dada_hdu_destroy(conf.hdu_in);

  free(conf.streams);
  free(conf.fft_plans1);
  free(conf.fft_plans2);
  
  return EXIT_SUCCESS;
}

int register_header(conf_t *conf)
{
  uint64_t hdrsz;
  char *hdrbuf_in, *hdrbuf_out;
  uint64_t file_size, bytes_per_seconds;
  double scale;
  
  hdrbuf_in  = ipcbuf_get_next_read(conf->hdu_in->header_block, &hdrsz);  
  if (!hdrbuf_in)
    {
      multilog(runtime_log, LOG_ERR, "get next header block error.\n");
      fprintf(stderr, "Error getting header_buf, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  if(hdrsz != DADA_HDRSZ)
    {
      multilog(runtime_log, LOG_ERR, "get next header block error.\n");
      fprintf(stderr, "Header size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  hdrbuf_out = ipcbuf_get_next_write(conf->hdu_out->header_block);
  if (!hdrbuf_out)
    {
      multilog(runtime_log, LOG_ERR, "get next header block error.\n");
      fprintf(stderr, "Error getting header_buf, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }  
  if (ascii_header_get(hdrbuf_in, "FILE_SIZE", "%"PRIu64"", &file_size) < 0)  
    {
      multilog(runtime_log, LOG_ERR, "failed ascii_header_get FILE_SIZE\n");
      fprintf(stderr, "Error getting FILE_SIZE, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }   
  if (ascii_header_get(hdrbuf_in, "BYTES_PER_SECOND", "%"PRIu64"", &bytes_per_seconds) < 0)  
    {
      multilog(runtime_log, LOG_ERR, "failed ascii_header_get BYTES_PER_SECOND\n");
      fprintf(stderr, "Error getting BYTES_PER_SECOND, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  memcpy(hdrbuf_out, hdrbuf_in, DADA_HDRSZ); // Pass the header 
  scale =  OSAMP_RATEI * (double)NBYTE_OUT/ (NCHAN_RATEI * (double)NBYTE_IN);
  file_size = (uint64_t)(file_size * scale);
  bytes_per_seconds = (uint64_t)(bytes_per_seconds * scale);
  
  if (ascii_header_set(hdrbuf_out, "NCHAN", "%d", NCHAN_OUT) < 0)  
    {
      multilog(runtime_log, LOG_ERR, "failed ascii_header_set NCHAN\n");
      fprintf(stderr, "Error setting NCHAN, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  if (ascii_header_set(hdrbuf_out, "BW", "%d", NCHAN_OUT) < 0)  
    {
      multilog(runtime_log, LOG_ERR, "failed ascii_header_set BW\n");
      fprintf(stderr, "Error setting BW, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  if (ascii_header_set(hdrbuf_out, "TSAMP", "%lf", TSAMP) < 0)  
    {
      multilog(runtime_log, LOG_ERR, "failed ascii_header_set TSAMP\n");
      fprintf(stderr, "Error setting TSAMP, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  if (ascii_header_set(hdrbuf_out, "NBIT", "%d", NBIT) < 0)  
    {
      multilog(runtime_log, LOG_ERR, "failed ascii_header_set NBIT\n");
      fprintf(stderr, "Error setting TSAMP, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  if (ascii_header_set(hdrbuf_out, "FILE_SIZE", "%"PRIu64"", file_size) < 0)  
    {
      multilog(runtime_log, LOG_ERR, "failed ascii_header_set NBIT\n");
      fprintf(stderr, "Error setting TSAMP, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  if (ascii_header_set(hdrbuf_out, "BYTES_PER_SECOND", "%"PRIu64"", bytes_per_seconds) < 0)  
    {
      multilog(runtime_log, LOG_ERR, "failed ascii_header_set BYTES_PER_SECOND\n");
      fprintf(stderr, "Error setting BYTES_PER_SECOND, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  
  if(ipcbuf_mark_filled(conf->hdu_out->header_block, DADA_HDRSZ) < 0)      
    {
      multilog(runtime_log, LOG_ERR, "Could not close header block\n");
      fprintf(stderr, "Error mark_filled, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  if(ipcbuf_mark_cleared(conf->hdu_in->header_block))  
    {
      multilog(runtime_log, LOG_ERR, "Could not clear header block\n");
      fprintf(stderr, "Error header_clear, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }    

  return EXIT_SUCCESS;
}