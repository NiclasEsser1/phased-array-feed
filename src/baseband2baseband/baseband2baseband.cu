#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

#include <time.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>
#include <math.h>

#include "multilog.h"
#include "baseband2baseband.cuh"
#include "cudautil.cuh"
#include "kernel.cuh"

extern multilog_t *runtime_log;

int init_baseband2baseband(conf_t *conf)
{
  int i;
  int iembed1, istride1, idist1, oembed1, ostride1, odist1, batch1, nx1;
  int iembed2, istride2, idist2, oembed2, ostride2, odist2, batch2, nx2;
  ipcbuf_t *db = NULL;
  uint64_t hdrsz;
  
  /* Prepare buffer, stream and fft plan for process */
  conf->sclndim = conf->rbufin_ndf_chk * NSAMP_DF * NPOL_SAMP * NDIM_POL; // Only works when two polarisations has similar power level
  conf->nsamp1  = conf->stream_ndf_chk * NCHK_CAPTURE * NCHAN_CHK * NSAMP_DF;
  conf->npol1   = conf->nsamp1 * NPOL_SAMP;
  conf->ndata1  = conf->npol1  * NDIM_POL;
		
  conf->nsamp2  = conf->nsamp1 * OSAMP_RATEI / NCHAN_RATEI;
  conf->npol2   = conf->nsamp2 * NPOL_SAMP;
  conf->ndata2  = conf->npol2  * NDIM_POL;

  nx1        = CUFFT_NX1;
  batch1     = conf->npol1 / CUFFT_NX1;
  
  iembed1    = nx1;
  istride1   = 1;
  idist1     = nx1;
  
  oembed1    = nx1;
  ostride1   = 1;
  odist1     = nx1;
  
  nx2        = CUFFT_NX2;
  batch2     = conf->npol2 / CUFFT_NX2;
  
  iembed2    = nx2;
  istride2   = 1;
  idist2     = nx2;
  
  oembed2    = nx2;
  ostride2   = 1;
  odist2     = nx2;

  conf->streams = (hipStream_t *)malloc(conf->nstream * sizeof(hipStream_t));
  conf->fft_plans1 = (hipfftHandle *)malloc(conf->nstream * sizeof(hipfftHandle));
  conf->fft_plans2 = (hipfftHandle *)malloc(conf->nstream * sizeof(hipfftHandle));
  for(i = 0; i < conf->nstream; i ++)
    {
      CudaSafeCall(hipStreamCreate(&conf->streams[i]));
      CufftSafeCall(hipfftPlanMany(&conf->fft_plans1[i], CUFFT_RANK1, &nx1, &iembed1, istride1, idist1, &oembed1, ostride1, odist1, HIPFFT_C2C, batch1));
      CufftSafeCall(hipfftPlanMany(&conf->fft_plans2[i], CUFFT_RANK2, &nx2, &iembed2, istride2, idist2, &oembed2, ostride2, odist2, HIPFFT_C2C, batch2));
      
      CufftSafeCall(hipfftSetStream(conf->fft_plans1[i], conf->streams[i]));
      CufftSafeCall(hipfftSetStream(conf->fft_plans2[i], conf->streams[i]));
    }
  
  conf->sbufin_size    = conf->ndata1 * NBYTE_IN;
  conf->sbufout_size   = conf->ndata2 * NBYTE_OUT;
  
  conf->bufin_size     = conf->nstream * conf->sbufin_size;
  conf->bufout_size    = conf->nstream * conf->sbufout_size;
  
  conf->sbufrt1_size = conf->npol1 * sizeof(hipfftComplex);
  conf->sbufrt2_size = conf->npol2 * sizeof(hipfftComplex);
  conf->bufrt1_size  = conf->nstream * conf->sbufrt1_size;
  conf->bufrt2_size  = conf->nstream * conf->sbufrt2_size;
    
  //conf->hbufin_offset = conf->sbufin_size / sizeof(char);
  conf->hbufin_offset = conf->sbufin_size;
  conf->dbufin_offset = conf->sbufin_size / sizeof(int64_t);
  conf->bufrt1_offset = conf->sbufrt1_size / sizeof(hipfftComplex);
  conf->bufrt2_offset = conf->sbufrt2_size / sizeof(hipfftComplex);
  
  conf->dbufout_offset   = conf->sbufout_size / NBYTE_OUT;
  //conf->hbufout_offset   = conf->sbufout_size / sizeof(char);
  conf->hbufout_offset   = conf->sbufout_size;

  CudaSafeCall(hipMalloc((void **)&conf->dbuf_in, conf->bufin_size));
  
  CudaSafeCall(hipMalloc((void **)&conf->dbuf_out, conf->bufout_size));       
  CudaSafeCall(hipMalloc((void **)&conf->ddat_offs, NCHAN * sizeof(float)));
  CudaSafeCall(hipMalloc((void **)&conf->dsquare_mean, NCHAN * sizeof(float)));
  CudaSafeCall(hipMalloc((void **)&conf->ddat_scl, NCHAN * sizeof(float)));
      
  CudaSafeCall(hipMemset((void *)conf->ddat_offs, 0, NCHAN * sizeof(float)));   // We have to clear the memory for this parameter
  CudaSafeCall(hipMemset((void *)conf->dsquare_mean, 0, NCHAN * sizeof(float)));// We have to clear the memory for this parameter
  
  CudaSafeCall(hipHostMalloc((void **)&conf->hdat_scl, NCHAN * sizeof(float)));   // Malloc host memory to receive data from device
  CudaSafeCall(hipHostMalloc((void **)&conf->hdat_offs, NCHAN * sizeof(float)));   // Malloc host memory to receive data from device
  CudaSafeCall(hipHostMalloc((void **)&conf->hsquare_mean, NCHAN * sizeof(float)));   // Malloc host memory to receive data from device
  
  CudaSafeCall(hipMalloc((void **)&conf->buf_rt1, conf->bufrt1_size));
  CudaSafeCall(hipMalloc((void **)&conf->buf_rt2, conf->bufrt2_size)); 

  /* Prepare the setup of kernels */
  conf->gridsize_unpack.x = conf->stream_ndf_chk;
  conf->gridsize_unpack.y = NCHK_CAPTURE;
  conf->gridsize_unpack.z = 1;
  conf->blocksize_unpack.x = NSAMP_DF; 
  conf->blocksize_unpack.y = NCHAN_CHK;
  conf->blocksize_unpack.z = 1;
  
  conf->gridsize_swap_select_transpose_swap.x = NCHK_CAPTURE * NCHAN_CHK;
  conf->gridsize_swap_select_transpose_swap.y = conf->stream_ndf_chk * NSAMP_DF / CUFFT_NX1;
  conf->gridsize_swap_select_transpose_swap.z = 1;  
  conf->blocksize_swap_select_transpose_swap.x = CUFFT_NX1;
  conf->blocksize_swap_select_transpose_swap.y = 1;
  conf->blocksize_swap_select_transpose_swap.z = 1;
  
  conf->gridsize_mean.x = 1; 
  conf->gridsize_mean.y = 1; 
  conf->gridsize_mean.z = 1;
  conf->blocksize_mean.x = NCHAN; 
  conf->blocksize_mean.y = 1;
  conf->blocksize_mean.z = 1;
  
  conf->gridsize_scale.x = 1;
  conf->gridsize_scale.y = 1;
  conf->gridsize_scale.z = 1;
  conf->blocksize_scale.x = NCHAN;
  conf->blocksize_scale.y = 1;
  conf->blocksize_scale.z = 1;
  
  conf->gridsize_transpose_pad.x = conf->stream_ndf_chk * NSAMP_DF / CUFFT_NX1; 
  conf->gridsize_transpose_pad.y = NCHAN;
  conf->gridsize_transpose_pad.z = 1;
  conf->blocksize_transpose_pad.x = CUFFT_NX2;
  conf->blocksize_transpose_pad.y = 1;
  conf->blocksize_transpose_pad.z = 1;

  conf->gridsize_sum1.x = NCHAN;
  conf->gridsize_sum1.y = conf->stream_ndf_chk * NPOL_SAMP;
  conf->gridsize_sum1.z = 1;
  conf->blocksize_sum1.x = NSAMP_DF * CUFFT_NX2 / (2 * CUFFT_NX1);  // This is the right setup if CUFFT_NX2 is not equal to CUFFT_NX1
  conf->blocksize_sum1.y = 1;
  conf->blocksize_sum1.z = 1;
  
  conf->gridsize_sum2.x = NCHAN;
  conf->gridsize_sum2.y = 1;
  conf->gridsize_sum2.z = 1;
  conf->blocksize_sum2.x = conf->stream_ndf_chk * NPOL_SAMP / 2;
  conf->blocksize_sum2.y = 1;
  conf->blocksize_sum2.z = 1;
  
  conf->gridsize_transpose_scale.x = conf->stream_ndf_chk * NSAMP_DF / CUFFT_NX1; 
  conf->gridsize_transpose_scale.y = NCHAN / TILE_DIM;
  conf->gridsize_transpose_scale.z = 1;
  conf->blocksize_transpose_scale.x = TILE_DIM;
  conf->blocksize_transpose_scale.y = NROWBLOCK_TRANS;
  conf->blocksize_transpose_scale.z = 1;
  
  conf->gridsize_transpose_float.x = conf->stream_ndf_chk * NSAMP_DF / CUFFT_NX1; 
  conf->gridsize_transpose_float.y = NCHAN / TILE_DIM;
  conf->gridsize_transpose_float.z = 1;
  conf->blocksize_transpose_float.x = TILE_DIM;
  conf->blocksize_transpose_float.y = NROWBLOCK_TRANS;
  conf->blocksize_transpose_float.z = 1;
  
  /* attach to input ring buffer */
  conf->hdu_in = dada_hdu_create(runtime_log);
  dada_hdu_set_key(conf->hdu_in, conf->key_in);
  if(dada_hdu_connect(conf->hdu_in) < 0)
    {
      multilog(runtime_log, LOG_ERR, "could not connect to hdu\n");
      fprintf(stderr, "Can not connect to hdu, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }  
  db = (ipcbuf_t *) conf->hdu_in->data_block;
  conf->rbufin_size = ipcbuf_get_bufsz(db);  
  if(conf->rbufin_size % conf->bufin_size != 0)  
    {
      multilog(runtime_log, LOG_ERR, "data buffer size mismatch\n");
      fprintf(stderr, "Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }
  dada_cuda_dbregister(conf->hdu_in);  // registers the existing host memory range for use by CUDA   
  hdrsz = ipcbuf_get_bufsz(conf->hdu_in->header_block);  
  if(hdrsz != DADA_HDRSZ)    // This number should match
    {
      multilog(runtime_log, LOG_ERR, "data buffer size mismatch\n");
      fprintf(stderr, "Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }
  if(dada_hdu_lock_read(conf->hdu_in) < 0) // make ourselves the read client 
    {
      multilog(runtime_log, LOG_ERR, "open_hdu: could not lock write\n");
      fprintf(stderr, "Error locking HDU, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }

  /* Prepare output ring buffer */
  conf->hdu_out = dada_hdu_create(runtime_log);
  dada_hdu_set_key(conf->hdu_out, conf->key_out);
  if(dada_hdu_connect(conf->hdu_out) < 0)
    {
      multilog(runtime_log, LOG_ERR, "could not connect to hdu\n");
      fprintf(stderr, "Can not connect to hdu, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }
  db = (ipcbuf_t *) conf->hdu_out->data_block;
  conf->rbufout_size = ipcbuf_get_bufsz(db);
  if(conf->rbufout_size % conf->bufout_size != 0)  
    {
      multilog(runtime_log, LOG_ERR, "data buffer size mismatch\n");
      fprintf(stderr, "Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }  
  hdrsz = ipcbuf_get_bufsz(conf->hdu_out->header_block);  
  if(hdrsz != DADA_HDRSZ)    // This number should match
    {
      multilog(runtime_log, LOG_ERR, "data buffer size mismatch\n");
      fprintf(stderr, "Buffer size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;    
    }  
  if(dada_hdu_lock_write(conf->hdu_out) < 0)   // make ourselves the write client 
    {
      multilog(runtime_log, LOG_ERR, "open_hdu: could not lock write\n");
      fprintf(stderr, "Error locking HDU, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  if(ipcbuf_disable_sod(db) < 0)
    {
      multilog(runtime_log, LOG_ERR, "Can not write data before start, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      fprintf(stderr, "Can not write data before start, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  
  return EXIT_SUCCESS;
}

int do_baseband2baseband(conf_t conf)
{
  /*
    The whole procedure for fold mode is :
    1. Unpack the data and reorder it from TFTFP to PFT order, prepare for the forward FFT;
    2. Forward FFT the PFT data to get finer channelzation and the data is in PFTF order after FFT;
    3. Swap the FFT output to put the frequency centre on the right place, drop frequency channel edge and band edge and put the data into PTF order, swap the data and put the centre frequency at bin 0 for each FFT block, prepare for inverse FFT;
    4. Inverse FFT the data to get PTFT order data;
    5. Transpose the data to get TFP data and scale it;    
  */
  uint64_t i, j;
  uint64_t hbufin_offset, dbufin_offset, bufrt1_offset, bufrt2_offset, hbufout_offset, dbufout_offset;
  dim3 gridsize_unpack, blocksize_unpack;
  dim3 gridsize_swap_select_transpose_swap, blocksize_swap_select_transpose_swap;
  dim3 gridsize_transpose_scale, blocksize_transpose_scale;
  dim3 gridsize_transpose_float, blocksize_transpose_float;
  uint64_t read_blkid, write_blkid;
  uint64_t curbufsz;
  ipcbuf_t *db_in = NULL, *db_out = NULL;
  db_in = (ipcbuf_t *)conf.hdu_in->data_block;
  db_out = (ipcbuf_t *)conf.hdu_out->data_block;
  
  gridsize_unpack                      = conf.gridsize_unpack;
  blocksize_unpack                     = conf.blocksize_unpack;
  gridsize_swap_select_transpose_swap  = conf.gridsize_swap_select_transpose_swap;   
  blocksize_swap_select_transpose_swap = conf.blocksize_swap_select_transpose_swap;  
  gridsize_transpose_scale             = conf.gridsize_transpose_scale;
  blocksize_transpose_scale            = conf.blocksize_transpose_scale;
  gridsize_transpose_float             = conf.gridsize_transpose_float;
  blocksize_transpose_float            = conf.blocksize_transpose_float;
  
  /* Register header */
  if(register_header(&conf))
    {
      multilog(runtime_log, LOG_ERR, "header register failed, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      fprintf(stderr, "header register failed, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }

  /* Start the first */
  conf.hdu_in->data_block->curbuf = ipcio_open_block_read(conf.hdu_in->data_block, &curbufsz, &read_blkid);
  conf.hdu_out->data_block->curbuf = ipcio_open_block_write(conf.hdu_out->data_block, &write_blkid);   /* Open buffer to write */
  //ipcio_open_block_read(conf.hdu_in->data_block, &curbufsz, &read_blkid);
  //ipcio_open_block_write(conf.hdu_out->data_block, &write_blkid);   /* Open buffer to write */
    
  /* Get scale of data */
  dat_offs_scl(conf);
  
  /* Do the real job */  
  while(true)
    // The first time we open a block at the scale calculation, we need to make sure that the input ring buffer block is bigger than the block needed for scale calculation
    // Otherwise we have to open couple of blocks to calculate scales and these blocks will dropped after that
    {
      for(i = 0; i < conf.nrun_blk; i ++)
	{
	  for(j = 0; j < conf.nstream; j++)
	    {
	      hbufin_offset = j * conf.hbufin_offset + i * conf.bufin_size;
	      dbufin_offset = j * conf.dbufin_offset; 
	      bufrt1_offset = j * conf.bufrt1_offset;
	      bufrt2_offset = j * conf.bufrt2_offset;

	      dbufout_offset = j * conf.dbufout_offset;
	      hbufout_offset = j * conf.hbufout_offset + i * conf.bufout_size;
	      
	      CudaSafeCall(hipMemcpyAsync(&conf.dbuf_in[dbufin_offset], &conf.hdu_in->data_block->curbuf[hbufin_offset], conf.sbufin_size, hipMemcpyHostToDevice, conf.streams[j]));
	      
	      /* Unpack raw data into hipfftComplex array */
	      unpack_kernel<<<gridsize_unpack, blocksize_unpack, 0, conf.streams[j]>>>(&conf.dbuf_in[dbufin_offset], &conf.buf_rt1[bufrt1_offset], conf.nsamp1);
	      
	      /* Do forward FFT */
	      CufftSafeCall(hipfftExecC2C(conf.fft_plans1[j], &conf.buf_rt1[bufrt1_offset], &conf.buf_rt1[bufrt1_offset], HIPFFT_FORWARD));

	      /* Prepare for inverse FFT */
	      swap_select_transpose_swap_kernel<<<gridsize_swap_select_transpose_swap, blocksize_swap_select_transpose_swap, 0, conf.streams[j]>>>(&conf.buf_rt1[bufrt1_offset], &conf.buf_rt2[bufrt2_offset], conf.nsamp1, conf.nsamp2); 
	      /* Do inverse FFT */
	      CufftSafeCall(hipfftExecC2C(conf.fft_plans2[j], &conf.buf_rt2[bufrt2_offset], &conf.buf_rt2[bufrt2_offset], HIPFFT_BACKWARD));
	      /* Get final output */
	      transpose_scale_kernel<<<gridsize_transpose_scale, blocksize_transpose_scale, 0, conf.streams[j]>>>(&conf.buf_rt2[bufrt2_offset], &conf.dbuf_out[dbufout_offset], conf.nsamp2, conf.ddat_offs, conf.ddat_scl);   
	      /* Copy the final output to host */
	      CudaSafeCall(hipMemcpyAsync(&conf.hdu_out->data_block->curbuf[hbufout_offset], &conf.dbuf_out[dbufout_offset], conf.sbufout_size, hipMemcpyDeviceToHost, conf.streams[j]));
	    }
	  CudaSynchronizeCall(); // Sync here is for multiple streams
	}
      	  
      /* Close current buffer */
      ipcio_close_block_write(conf.hdu_out->data_block, conf.rbufout_size);
      ipcio_close_block_read(conf.hdu_in->data_block, conf.hdu_in->data_block->curbufsz);

      if(ipcbuf_eod(db_in) > 0)
	{
	  ipcbuf_enable_eod(db_out);

	  if(register_header(&conf))
	    {
	      multilog(runtime_log, LOG_ERR, "header register failed, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	      fprintf(stderr, "header register failed, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
	      return EXIT_FAILURE;
	    }
	  
	  conf.hdu_out->data_block->curbuf = ipcio_open_block_write(conf.hdu_out->data_block, &write_blkid);   /* Open buffer to write */
	  conf.hdu_in->data_block->curbuf = ipcio_open_block_read(conf.hdu_in->data_block, &curbufsz, &read_blkid);

	  //ipcio_open_block_write(conf.hdu_out->data_block, &write_blkid);   /* Open buffer to write */
	  //ipcio_open_block_read(conf.hdu_in->data_block, &curbufsz, &read_blkid);
	  //ipcbuf_enable_sod((ipcbuf_t *)conf.hdu_out->data_block, write_blkid, 0);
	  //dat_offs_scl(conf);
	}
      else
	{
	  conf.hdu_out->data_block->curbuf = ipcio_open_block_write(conf.hdu_out->data_block, &write_blkid);   /* Open buffer to write */
	  conf.hdu_in->data_block->curbuf = ipcio_open_block_read(conf.hdu_in->data_block, &curbufsz, &read_blkid);
	  //ipcio_open_block_write(conf.hdu_out->data_block, &write_blkid);   /* Open buffer to write */
	  //ipcio_open_block_read(conf.hdu_in->data_block, &curbufsz, &read_blkid);
	}
    }

  ipcio_close_block_write(conf.hdu_out->data_block, conf.rbufout_size);
  ipcio_close_block_read(conf.hdu_in->data_block, conf.hdu_in->data_block->curbufsz);
  
  return EXIT_SUCCESS;
}

int dat_offs_scl(conf_t conf)
{
  /*
    The procedure for fold mode is:
    1. Get PTFT data as we did at process;
    2. Pad the data;
    3. Add the padded data in time;
    4. Get the mean of the added data;
    5. Get the scale with the mean;
  */
  uint64_t i, j;
  dim3 gridsize_unpack, blocksize_unpack;
  dim3 gridsize_swap_select_transpose_swap, blocksize_swap_select_transpose_swap;
  dim3 gridsize_mean, blocksize_mean;
  dim3 gridsize_sum1, blocksize_sum1;
  dim3 gridsize_sum2, blocksize_sum2;
  dim3 gridsize_scale, blocksize_scale;
  dim3 gridsize_transpose_pad, blocksize_transpose_pad;
  uint64_t hbufin_offset, dbufin_offset, bufrt1_offset, bufrt2_offset;
    
  char fname[MSTR_LEN];
  FILE *fp=NULL;
  
  gridsize_unpack                      = conf.gridsize_unpack;
  blocksize_unpack                     = conf.blocksize_unpack;
  gridsize_swap_select_transpose_swap  = conf.gridsize_swap_select_transpose_swap;   
  blocksize_swap_select_transpose_swap = conf.blocksize_swap_select_transpose_swap; 
  gridsize_transpose_pad               = conf.gridsize_transpose_pad;
  blocksize_transpose_pad              = conf.blocksize_transpose_pad;
  	         	               						       
  gridsize_sum1              = conf.gridsize_sum1;	       
  blocksize_sum1             = conf.blocksize_sum1;
  gridsize_sum2              = conf.gridsize_sum2;	       
  blocksize_sum2             = conf.blocksize_sum2;
  gridsize_mean              = conf.gridsize_mean;	       
  blocksize_mean             = conf.blocksize_mean;
  gridsize_scale              = conf.gridsize_scale;	       
  blocksize_scale             = conf.blocksize_scale;
  
  if(conf.hdu_in->data_block->curbuf == NULL)
    {
      multilog (runtime_log, LOG_ERR, "Can not get buffer block from input ring buffer, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      fprintf(stderr, "Can not get buffer block from input ring buffer, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
    
  for(i = 0; i < conf.rbufin_size; i += conf.bufin_size)
    {
      for (j = 0; j < conf.nstream; j++)
	{
	  hbufin_offset = j * conf.hbufin_offset + i;
	  dbufin_offset = j * conf.dbufin_offset; 
	  bufrt1_offset = j * conf.bufrt1_offset;
	  bufrt2_offset = j * conf.bufrt2_offset;
	  
	  /* Copy data into device */
	  CudaSafeCall(hipMemcpyAsync(&conf.dbuf_in[dbufin_offset], &conf.hdu_in->data_block->curbuf[hbufin_offset], conf.sbufin_size, hipMemcpyHostToDevice, conf.streams[j]));

	  /* Unpack raw data into hipfftComplex array */
	  unpack_kernel<<<gridsize_unpack, blocksize_unpack, 0, conf.streams[j]>>>(&conf.dbuf_in[dbufin_offset], &conf.buf_rt1[bufrt1_offset], conf.nsamp1);

	  /* Do forward FFT */
	  CufftSafeCall(hipfftExecC2C(conf.fft_plans1[j], &conf.buf_rt1[bufrt1_offset], &conf.buf_rt1[bufrt1_offset], HIPFFT_FORWARD));

	  /* Prepare for inverse FFT */
	  swap_select_transpose_swap_kernel<<<gridsize_swap_select_transpose_swap, blocksize_swap_select_transpose_swap, 0, conf.streams[j]>>>(&conf.buf_rt1[bufrt1_offset], &conf.buf_rt2[bufrt2_offset], conf.nsamp1, conf.nsamp2); 
	  
	  /* Do inverse FFT */
	  CufftSafeCall(hipfftExecC2C(conf.fft_plans2[j], &conf.buf_rt2[bufrt2_offset], &conf.buf_rt2[bufrt2_offset], HIPFFT_BACKWARD));
	  
	  /* Transpose the data from PTFT to FTP for later calculation */
	  transpose_pad_kernel<<<gridsize_transpose_pad, blocksize_transpose_pad, 0, conf.streams[j]>>>(&conf.buf_rt2[bufrt2_offset], conf.nsamp2, &conf.buf_rt1[bufrt1_offset]);
	  
	  /* Get the sum of samples and square of samples */
	  sum_kernel<<<gridsize_sum1, blocksize_sum1, blocksize_sum1.x * sizeof(hipfftComplex), conf.streams[j]>>>(&conf.buf_rt1[bufrt1_offset], &conf.buf_rt2[bufrt2_offset]);
	  sum_kernel<<<gridsize_sum2, blocksize_sum2, blocksize_sum2.x * sizeof(hipfftComplex), conf.streams[j]>>>(&conf.buf_rt2[bufrt2_offset], &conf.buf_rt1[bufrt1_offset]);
	}
      CudaSynchronizeCall(); // Sync here is for multiple streams

      mean_kernel<<<gridsize_mean, blocksize_mean>>>(conf.buf_rt1, conf.bufrt1_offset, conf.ddat_offs, conf.dsquare_mean, conf.nstream, conf.sclndim);
    }
  /* Get the scale of each chanel */
  scale_kernel<<<gridsize_scale, blocksize_scale>>>(conf.ddat_offs, conf.dsquare_mean, conf.ddat_scl);
  CudaSynchronizeCall();
  
  CudaSafeCall(hipMemcpy(conf.hdat_offs, conf.ddat_offs, sizeof(float) * NCHAN, hipMemcpyDeviceToHost));
  CudaSafeCall(hipMemcpy(conf.hdat_scl, conf.ddat_scl, sizeof(float) * NCHAN, hipMemcpyDeviceToHost));
  CudaSafeCall(hipMemcpy(conf.hsquare_mean, conf.dsquare_mean, sizeof(float) * NCHAN, hipMemcpyDeviceToHost));
 
  for (i = 0; i< NCHAN; i++)
    fprintf(stdout, "DAT_OFFS:\t%E\tDAT_SCL:\t%E\n", conf.hdat_offs[i], conf.hdat_scl[i]);
  /* Record scale into file */
  sprintf(fname, "%s/%s_scale.txt", conf.dir, conf.utc_start);
  fp = fopen(fname, "w");
  if(fp == NULL)
    {
      multilog (runtime_log, LOG_ERR, "Can not open scale file, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      fprintf(stderr, "Can not open scale file, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }

  for (i = 0; i< NCHAN; i++)
    fprintf(fp, "%E\t%E\n", conf.hdat_offs[i], conf.hdat_scl[i]);
  fclose(fp);
  return EXIT_SUCCESS;
}

int destroy_baseband2baseband(conf_t conf)
{
  int i;
  
  for (i = 0; i < conf.nstream; i++)
    {
      CudaSafeCall(hipStreamDestroy(conf.streams[i]));
      CufftSafeCall(hipfftDestroy(conf.fft_plans1[i]));
      CufftSafeCall(hipfftDestroy(conf.fft_plans2[i]));
    }
  
  hipFree(conf.dbuf_in);

  hipFree(conf.dbuf_out);
  hipHostFree(conf.hdat_offs);
  hipHostFree(conf.hsquare_mean);
  hipHostFree(conf.hdat_scl);
  hipFree(conf.ddat_offs);
  hipFree(conf.dsquare_mean);
  hipFree(conf.ddat_scl);
  
  dada_hdu_unlock_write(conf.hdu_out);
  dada_hdu_disconnect(conf.hdu_out);
  dada_hdu_destroy(conf.hdu_out);

  hipFree(conf.buf_rt1);
  hipFree(conf.buf_rt2);

  dada_cuda_dbunregister(conf.hdu_in);
  
  dada_hdu_unlock_read(conf.hdu_in);
  dada_hdu_disconnect(conf.hdu_in);
  dada_hdu_destroy(conf.hdu_in);

  free(conf.streams);
  free(conf.fft_plans1);
  free(conf.fft_plans2);
  
  return EXIT_SUCCESS;
}

int register_header(conf_t *conf)
{
  uint64_t hdrsz;
  char *hdrbuf_in, *hdrbuf_out;
  
  hdrbuf_in  = ipcbuf_get_next_read(conf->hdu_in->header_block, &hdrsz);  
  if (!hdrbuf_in)
    {
      multilog(runtime_log, LOG_ERR, "get next header block error.\n");
      fprintf(stderr, "Error getting header_buf, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  if(hdrsz != DADA_HDRSZ)
    {
      multilog(runtime_log, LOG_ERR, "get next header block error.\n");
      fprintf(stderr, "Header size mismatch, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  hdrbuf_out = ipcbuf_get_next_write(conf->hdu_out->header_block);
  if (!hdrbuf_out)
    {
      multilog(runtime_log, LOG_ERR, "get next header block error.\n");
      fprintf(stderr, "Error getting header_buf, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  memcpy(hdrbuf_out, hdrbuf_in, DADA_HDRSZ); // Pass the header 
  
  if (ascii_header_set(hdrbuf_out, "NCHAN", "%d", NCHAN) < 0)  
    {
      multilog(runtime_log, LOG_ERR, "failed ascii_header_set NCHAN\n");
      fprintf(stderr, "Error setting NCHAN, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  if (ascii_header_get(hdrbuf_out, "UTC_START", "%s", conf->utc_start) < 0)  
    {
      multilog(runtime_log, LOG_ERR, "failed ascii_header_set NCHAN\n");
      fprintf(stderr, "Error setting NCHAN, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  if (ascii_header_set(hdrbuf_out, "BW", "%d", NCHAN) < 0)  
    {
      multilog(runtime_log, LOG_ERR, "failed ascii_header_set BW\n");
      fprintf(stderr, "Error setting BW, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  if (ascii_header_set(hdrbuf_out, "TSAMP", "1.0") < 0)  
    {
      multilog(runtime_log, LOG_ERR, "failed ascii_header_set TSAMP\n");
      fprintf(stderr, "Error setting TSAMP, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  
  if(ipcbuf_mark_filled(conf->hdu_in->header_block, DADA_HDRSZ) < 0)      
    {
      multilog(runtime_log, LOG_ERR, "Could not close header block\n");
      fprintf(stderr, "Error mark_filled, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }
  if(ipcbuf_mark_cleared(conf->hdu_in->header_block))  
    {
      multilog(runtime_log, LOG_ERR, "Could not clear header block\n");
      fprintf(stderr, "Error header_clear, which happens at \"%s\", line [%d].\n", __FILE__, __LINE__);
      return EXIT_FAILURE;
    }    

  return EXIT_SUCCESS;
}